#include "hip/hip_runtime.h"
#include "CudaHelper.h"

#define _FFT_MAXDIMDIV 32
#define _FFT_MAXDIMDIVFACTORCOUNT 127
#define _FFT_MAXDIMDIVFACTOR 1021
#define _FFT_MAXDIM 4

#define _FFT_TESTVECTOR 10
#define _FFT_TESTMATRIXX 3
#define _FFT_TESTMATRIXY 4
#define _FFT_TESTMATRIXZ 4
#define _FFT_TESTMATRIXW 3
#define _FFT_TESTMATRIXV (_FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * _FFT_TESTMATRIXW)
#define _FFT_TESTMATRIXV3D (_FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ)

#define _pi_ (3.141592653589f)

#pragma region divice functions

/**
 * DFT of the array
 * diviceSource[iStartS + n * iStrideS]
 * Write to diviceArrayRes[iStartT + n * iStrideT]
 * Tested
 */
__device__ void _deviceSmallDFT(
    bool bInverse, 
    hipComplex* diviceArrayRes,
    const hipComplex* __restrict__ diviceSource,
    int iN, 
    int iStartT, 
    int iStrideT, 
    int iStartS, 
    int iStrideS)
{
    for (int i = 0; i < iN; ++i)
    {
        hipComplex res = make_hipComplex(0.0f, 0.0f);
        for (int j = 0; j < iN; ++j)
        {
            float fAngle = 2.0f * _pi_ * i * j / iN;
            res = hipCaddf(res, hipCmulf(
                diviceSource[iStartS + iStrideS * j],
                make_hipComplex(cosf(fAngle), bInverse ? -sinf(fAngle) : sinf(fAngle))));
        }
        diviceArrayRes[iStartT + iStrideT * i] = res;
    }
}

#pragma endregion

#pragma region kernels

/**
 * test Small DFT
 */
__global__ void _kernelTestSmallDFT(
    hipComplex* diviceArrayRes, 
    const hipComplex* __restrict__ diviceSource,
    int iN)
{
    _deviceSmallDFT(false, diviceArrayRes, diviceSource, iN, 0, 1, 0, 1);
}

/**
 * 
 */
__global__ void CTMappingForward(
    const hipComplex* __restrict__ diviceSource, 
    const int* __restrict__ subDim,
    hipComplex* diviceArrayRes)
{
    
}

//__global__ void CopyData(
//    const hipComplex* __restrict__ diviceSource, 
//    hipComplex* res)
//{
//    
//}

#pragma endregion

/**
 * If iSize = 230
 * res = [4,  2, 3, 5, 7], with 4 factors, 2x3x5x7 = 230
 */
int FindDecomp(int* res, int iSize)
{
    bool bFound = true;
    int iFactorCount = 0;
    int iMaxFactor = 0;
    while (bFound)
    {
        bFound = false;
        for (int i = 2; i < iSize; ++i)
        {
            if ((iSize / i) * i == iSize)
            {
                ++iFactorCount;
                res[iFactorCount] = i;
                iSize = iSize / i;
                if (i > iMaxFactor)
                {
                    iMaxFactor = i;
                }
                bFound = true;
                break;
            }
        }

        if (1 == iSize)
        {
            break;
        }

        if (!bFound || iFactorCount >= _FFT_MAXDIMDIVFACTORCOUNT - 1)
        {
            ++iFactorCount;
            res[iFactorCount] = iSize;
            if (iSize > iMaxFactor)
            {
                iMaxFactor = iSize;
            }
            break;
        }
    }
    res[0] = iFactorCount;

    return iMaxFactor;
}

void GenerateTestArray(hipComplex* hostArray, int iSize)
{
    for (int i = 0; i < iSize; ++i)
    {
        hostArray[i] = make_hipComplex((rand() % 101 - 50) / 50.0f, (rand() % 101 - 50) / 50.0f);
    }
}

void GenerateTestArray4D(hipComplex* hostArray)
{
    for (int i = 0; i < _FFT_TESTMATRIXV; ++i)
    {
        hostArray[i] = make_hipComplex((rand() % 101 - 50) / 50.0f, (rand() % 101 - 50) / 50.0f);
    }
}

void PrintTestArray1D(hipComplex* hostArray)
{
    SaveLog("\n{");
    for (int i = 0; i < _FFT_TESTMATRIXX; ++i)
    {
        const int iIndex = i;
        SaveLog("%1.10f %s %1.10fi",
            hostArray[i].x,
            hostArray[i].y < 0.0f ? "-" : "+",
            abs(hostArray[i].y));

        if (i == _FFT_TESTMATRIXX - 1)
        {
            SaveLog("}\n");
        }
        else
        {
            SaveLog(",");
        }
    }
}

void PrintTestArray2D(hipComplex* hostArray)
{
    SaveLog("\n{");
    for (int i = 0; i < _FFT_TESTMATRIXX; ++i)
    {
        SaveLog("{");

        for (int j = 0; j < _FFT_TESTMATRIXY; ++j)
        {
            const int iIndex = i * _FFT_TESTMATRIXY + j;
            SaveLog("%1.10f %s %1.10f I",
                hostArray[iIndex].x,
                hostArray[iIndex].y < 0.0f ? "" : "+",
                hostArray[iIndex].y);

            if (j == _FFT_TESTMATRIXY - 1)
            {
                SaveLog("}");
            }
            else
            {
                SaveLog(",");
            }
        }

        if (i == _FFT_TESTMATRIXX - 1)
        {
            SaveLog("}\n");
        }
        else
        {
            SaveLog(",\n");
        }
    }
}

void PrintTestArray3D(hipComplex* hostArray)
{
    SaveLog("\n{");
    for (int i = 0; i < _FFT_TESTMATRIXX; ++i)
    {
        SaveLog("{");

        for (int j = 0; j < _FFT_TESTMATRIXY; ++j)
        {
            SaveLog("{");
            for (int k = 0; k < _FFT_TESTMATRIXZ; ++k)
            {
                const int iIndex = i * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ
                    + j * _FFT_TESTMATRIXZ
                    + k;
                SaveLog("%1.10f %s %1.10f I",
                    hostArray[iIndex].x,
                    hostArray[iIndex].y < 0.0f ? "" : "+",
                    hostArray[iIndex].y);

                if (k == _FFT_TESTMATRIXZ - 1)
                {
                    SaveLog("}");
                }
                else
                {
                    SaveLog(", ");
                }
            }
            if (j == _FFT_TESTMATRIXY - 1)
            {
                SaveLog("}\n");
            }
            else
            {
                SaveLog(",\n");
            }
        }

        if (i == _FFT_TESTMATRIXX - 1)
        {
            SaveLog("}\n");
        }
        else
        {
            SaveLog(",\n");
        }
    }
}

void PrintTestArray4D(hipComplex* hostArray)
{
    SaveLog("\n{\n");
    for (int i = 0; i < _FFT_TESTMATRIXX; ++i)
    {
        SaveLog("{");
        for (int j = 0; j < _FFT_TESTMATRIXY; ++j)
        {
            if (0 == j)
            {
                SaveLog("{");
            }
            else
            {
                SaveLog(" {");
            }
            for (int k = 0; k < _FFT_TESTMATRIXZ; ++k)
            {
                SaveLog("{");
                for (int l = 0; l < _FFT_TESTMATRIXW; ++l)
                {
                    const int iIndex = i * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * _FFT_TESTMATRIXW
                                     + j * _FFT_TESTMATRIXZ * _FFT_TESTMATRIXW
                                     + k * _FFT_TESTMATRIXW
                                     + l;
                    SaveLog("%1.10f %s %1.10f I",
                        hostArray[iIndex].x,
                        hostArray[iIndex].y < 0.0f ? "-" : "+",
                        abs(hostArray[iIndex].y));
                    if (l == _FFT_TESTMATRIXW - 1)
                    {
                        SaveLog("}");
                    }
                    else
                    {
                        SaveLog(", ");
                    }
                }

                if (k == _FFT_TESTMATRIXZ - 1)
                {
                    SaveLog("}");
                }
                else
                {
                    SaveLog(", ");
                }
            }
            if (j == _FFT_TESTMATRIXY - 1)
            {
                SaveLog("\n}");
            }
            else
            {
                SaveLog(",\n");
            }
        }

        if (i == _FFT_TESTMATRIXX - 1)
        {
            SaveLog("}\n");
        }
        else
        {
            SaveLog(",\n");
        }
    }
}

hipComplex* GenerateTestMatrix(void)
{
    return nullptr;
}

/**
* 1D
* input [b * idist + x * istride]
* output[b * odist + x * ostride]
* 2D
* input [b * idist + (x * inembed[1] + y) * istride]
* output[b * odist + (x * onembed[1] + y) * ostride]
* 3D
* input [b * idist + ((x * inembed[1] + y) * inembed[2] + z) * istride]
* output[b * odist + ((x * onembed[1] + y) * onembed[2] + z) * ostride]
*/

int main()
{
    ClearLog();

    printf("decomp:");
    int decomp[128];
    FindDecomp(decomp, 210);
    for (int i = 0; i < decomp[0]; ++i)
    {
        printf("%d,", decomp[i + 1]);
    }
    printf("\n");


    hipComplex* dD1Res;
    hipComplex* dD1Source;
    hipComplex* dD2Res;
    hipComplex* dD2Source;
    hipComplex* dD3Res;
    hipComplex* dD3Source;
    hipComplex* dD4Res;
    hipComplex* dD4Source;
    hipComplex* hD1Res = (hipComplex*)malloc(_FFT_TESTMATRIXX * sizeof(hipComplex));
    hipComplex* hD1Source = (hipComplex*)malloc(_FFT_TESTMATRIXX * sizeof(hipComplex));
    hipComplex* hD2Res = (hipComplex*)malloc(_FFT_TESTMATRIXX * _FFT_TESTMATRIXY * sizeof(hipComplex));
    hipComplex* hD2Source = (hipComplex*)malloc(_FFT_TESTMATRIXX * _FFT_TESTMATRIXY * sizeof(hipComplex));
    hipComplex* hD3Res = (hipComplex*)malloc(_FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * sizeof(hipComplex));
    hipComplex* hD3Source = (hipComplex*)malloc(_FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * sizeof(hipComplex));
    hipComplex* hD4Res = (hipComplex*)malloc(_FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * _FFT_TESTMATRIXW * sizeof(hipComplex));
    hipComplex* hD4Source = (hipComplex*)malloc(_FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * _FFT_TESTMATRIXW * sizeof(hipComplex));

    checkCudaErrors(hipMalloc((void**)&dD1Res, _FFT_TESTMATRIXX * sizeof(hipComplex)));
    checkCudaErrors(hipMalloc((void**)&dD1Source, _FFT_TESTMATRIXX * sizeof(hipComplex)));
    checkCudaErrors(hipMalloc((void**)&dD2Res, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * sizeof(hipComplex)));
    checkCudaErrors(hipMalloc((void**)&dD2Source, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * sizeof(hipComplex)));
    checkCudaErrors(hipMalloc((void**)&dD3Res, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * sizeof(hipComplex)));
    checkCudaErrors(hipMalloc((void**)&dD3Source, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * sizeof(hipComplex)));
    checkCudaErrors(hipMalloc((void**)&dD4Res, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * _FFT_TESTMATRIXW * sizeof(hipComplex)));
    checkCudaErrors(hipMalloc((void**)&dD4Source, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * _FFT_TESTMATRIXW * sizeof(hipComplex)));

    GenerateTestArray(hD1Source, _FFT_TESTMATRIXX);
    GenerateTestArray(hD2Source, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY);
    GenerateTestArray(hD3Source, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ);
    GenerateTestArray(hD4Source, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * _FFT_TESTMATRIXW);

#pragma region 1D

    PrintTestArray1D(hD1Source);
    checkCudaErrors(hipMemcpy(dD1Source, hD1Source, _FFT_TESTMATRIXX * sizeof(hipComplex), hipMemcpyHostToDevice));
    hipfftHandle plan1d;
    hipfftPlan1d(&plan1d, _FFT_TESTMATRIXX, HIPFFT_C2C, 1);
    hipfftResult res1D = hipfftExecC2C(plan1d, dD1Source, dD1Res, HIPFFT_FORWARD);
    printf("1D res = %d\n", res1D);
    checkCudaErrors(hipMemcpy(hD1Res, dD1Res, _FFT_TESTMATRIXX * sizeof(hipComplex), hipMemcpyDeviceToHost));
    PrintTestArray1D(hD1Res);

#pragma endregion

#pragma region 2D

    PrintTestArray2D(hD2Source);
    checkCudaErrors(hipMemcpy(dD2Source, hD2Source, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * sizeof(hipComplex), hipMemcpyHostToDevice));
    hipfftHandle plan2d;
    hipfftPlan2d(&plan2d, _FFT_TESTMATRIXX, _FFT_TESTMATRIXY, HIPFFT_C2C);
    hipfftResult res2D = hipfftExecC2C(plan2d, dD2Source, dD2Res, HIPFFT_FORWARD);
    printf("2D res = %d\n", res2D);
    checkCudaErrors(hipMemcpy(hD2Res, dD2Res, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * sizeof(hipComplex), hipMemcpyDeviceToHost));
    PrintTestArray2D(hD2Res);

#pragma endregion

#pragma region 3D

    PrintTestArray3D(hD3Source);
    checkCudaErrors(hipMemcpy(dD3Source, hD3Source, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * sizeof(hipComplex), hipMemcpyHostToDevice));
    hipfftHandle plan3d;
    hipfftPlan3d(&plan3d, _FFT_TESTMATRIXX, _FFT_TESTMATRIXY, _FFT_TESTMATRIXZ, HIPFFT_C2C);
    hipfftResult res3D = hipfftExecC2C(plan3d, dD3Source, dD3Res, HIPFFT_FORWARD);
    printf("3D res = %d\n", res3D);
    checkCudaErrors(hipMemcpy(hD3Res, dD3Res, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * sizeof(hipComplex), hipMemcpyDeviceToHost));
    PrintTestArray3D(hD3Res);

#pragma endregion

#pragma region 4D

    PrintTestArray4D(hD4Source);
    checkCudaErrors(hipMemcpy(dD4Source, hD4Source, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * _FFT_TESTMATRIXW * sizeof(hipComplex), hipMemcpyHostToDevice));
    hipfftHandle plan4d1;
    int n[3] = { _FFT_TESTMATRIXY, _FFT_TESTMATRIXZ, _FFT_TESTMATRIXW };
    int inembed[3] = { _FFT_TESTMATRIXY, _FFT_TESTMATRIXZ, _FFT_TESTMATRIXW };
    int dist = _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * _FFT_TESTMATRIXW;
    hipfftPlanMany(&plan4d1, 3, n,
        inembed, 1, dist,
        inembed, 1, dist,
        HIPFFT_C2C, _FFT_TESTMATRIXX);
    
    hipfftResult res4D1 = hipfftExecC2C(plan4d1, dD4Source, dD4Res, HIPFFT_FORWARD);
    printf("4D res 1 = %d\n", res4D1);

    checkCudaErrors(hipMemcpy(hD4Res, dD4Res, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * _FFT_TESTMATRIXW * sizeof(hipComplex), hipMemcpyDeviceToHost));
    PrintTestArray4D(hD4Res);

    hipfftHandle plan4d2;
    int n2[1] = { _FFT_TESTMATRIXX };
    //note that if it was null, it will ignore the stride
    hipfftPlanMany(&plan4d2, 1, n2,
        n2, dist, 1,
        n2, dist, 1,
        HIPFFT_C2C, dist);

    //in out can be the same
    hipfftResult res4D2 = hipfftExecC2C(plan4d2, dD4Res, dD4Res, HIPFFT_FORWARD);
    printf("4D res 2 = %d\n", res4D2);

    checkCudaErrors(hipMemcpy(hD4Res, dD4Res, _FFT_TESTMATRIXX * _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * _FFT_TESTMATRIXW * sizeof(hipComplex), hipMemcpyDeviceToHost));
    PrintTestArray4D(hD4Res);

#pragma endregion

    //we have to do yzw 3D first
    //hipfftHandle plan;
    //hipfftPlan3d(&plan, _FFT_TESTMATRIXX, _FFT_TESTMATRIXY, _FFT_TESTMATRIXZ, HIPFFT_C2C);
    //int n[3] = { _FFT_TESTMATRIXW, _FFT_TESTMATRIXZ, _FFT_TESTMATRIXY };
    //int inembed[3] = { _FFT_TESTMATRIXY, _FFT_TESTMATRIXZ, _FFT_TESTMATRIXW };
    //int dist = _FFT_TESTMATRIXY * _FFT_TESTMATRIXZ * _FFT_TESTMATRIXW;
    //hipfftPlanMany(&plan, 3, n, 
    //    inembed, 1, dist,
    //    inembed, 1, dist,
    //    HIPFFT_C2C, _FFT_TESTMATRIXX);

    


    

    //=======================
    checkCudaErrors(hipFree(dD1Res));
    checkCudaErrors(hipFree(dD1Source));
    checkCudaErrors(hipFree(dD2Res));
    checkCudaErrors(hipFree(dD2Source));
    checkCudaErrors(hipFree(dD3Res));
    checkCudaErrors(hipFree(dD3Source));
    checkCudaErrors(hipFree(dD4Res));
    checkCudaErrors(hipFree(dD4Source));
    free(hD1Res);
    free(hD1Source);
    free(hD2Res);
    free(hD2Source);
    free(hD3Res);
    free(hD3Source);
    free(hD4Res);
    free(hD4Source);

    return 0;
}
