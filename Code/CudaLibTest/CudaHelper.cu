#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <stack>
#include <cstdarg>

#include <limits.h>
#include <windows.h>
#include <tchar.h>
#include <stdio.h>
#include <stdarg.h>
#include <math.h>
#include <malloc.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"

#include "hip/hip_complex.h"

#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

#ifdef __DRIVER_TYPES_H__
#ifndef DEVICE_RESET
#define DEVICE_RESET hipDeviceReset();
#endif
#else
#ifndef DEVICE_RESET
#define DEVICE_RESET
#endif
#endif

#ifdef __DRIVER_TYPES_H__
static const char *_cudaGetErrorEnum(hipError_t error) {
    return hipGetErrorName(error);
}
#endif

template <typename T> void check(T result, char const *const func, const char *const file,
    int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
        DEVICE_RESET
            // Make sure we call CUDA Device Reset before exiting
            exit(EXIT_FAILURE);
    }
}

class ClassABC
{
public:
    __host__ __device__ ClassABC() { ;  }
    int m_iValue;
};

class ClassDEF
{
public:
    __host__ __device__ ClassDEF() { ; }

    //Witout warning
    //union 
    //{
    //    ClassABC m_abc[1];
    //    int m_values[1];
    //};

    //With warning
    ClassABC m_abc[1];
};

__global__ void TestFunc()
{
    ClassDEF def[1];
}

int main()
{
    TestFunc << <1, 1 >> > ();
    return 0;
}