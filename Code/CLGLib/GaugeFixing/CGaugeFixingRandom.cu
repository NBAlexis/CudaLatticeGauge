#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CGaugeFixingRandom.cpp
// 
// DESCRIPTION:
//
//
// REVISION:
//  [09/25/2019 nbale]
//=============================================================================
#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

#pragma region kernels

__global__ void _CLG_LAUNCH_BOUND
_kernelRandomGauge(deviceSU3* pGx)
{
    intokernalInt4;

    const UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);
    const SIndex site = __idx->m_pDeviceIndexPositionToSIndex[1][uiBigIdx];

    if (site.IsDirichlet())
    {
        pGx[uiSiteIndex] = deviceSU3::makeSU3Id();
    }
    else
    {
        pGx[uiSiteIndex] = deviceSU3::makeSU3Random(_deviceGetFatIndex(uiSiteIndex, 0));
    }
}

/**
 * g(n) U_mu(n) g(n+mu)^dagger
 */
__global__ void _CLG_LAUNCH_BOUND
_kernelGaugeTransformRandom(
    const deviceSU3* __restrict__ pGx,
    deviceSU3* pGauge)
{
    intokernalInt4;

    const UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);
    const deviceSU3 left(pGx[uiSiteIndex]);

    for (BYTE dir = 0; dir < _DC_Dir; ++dir)
    {
        if (!__idx->_deviceIsBondOnSurface(uiBigIdx, dir))
        {
            UINT uiLinkDir = _deviceGetLinkIndex(uiSiteIndex, dir);
            deviceSU3 res(pGauge[uiLinkDir]);
            SSmallInt4 sWalking = _deviceSmallInt4OffsetC(sSite4, dir + 1);
            const SIndex site_p_mu = __idx->m_pDeviceIndexPositionToSIndex[1][__idx->_deviceGetBigIndex(sWalking)];
            if (!site_p_mu.IsDirichlet())
            {
                res.MulDagger(pGx[site_p_mu.m_uiSiteIndex]);
            }

            pGauge[uiLinkDir] = left.MulC(res);
        }
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelGaugeTransformFermion(
    const deviceSU3* __restrict__ pGx,
    deviceWilsonVectorSU3* pFermion)
{
    intokernalInt4;

    const UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);
    const SIndex site = __idx->m_pDeviceIndexPositionToSIndex[1][uiBigIdx];

    if (!site.IsDirichlet())
    {
        pFermion[uiSiteIndex] = pGx[uiSiteIndex].MulWilsonVector(pFermion[uiSiteIndex]);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelGaugeTransformAPhys(
    const deviceSU3* __restrict__ pGx,
    deviceSU3* pAphys)
{
    intokernalInt4;

    const BYTE uiDir = static_cast<BYTE>(_DC_Dir);
    const UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);
    const SIndex site = __idx->m_pDeviceIndexPositionToSIndex[1][uiBigIdx];

    if (!site.IsDirichlet())
    {
        for (BYTE dir = 0; dir < uiDir; ++dir)
        {
            const UINT uiLinkIndex = _deviceGetLinkIndex(uiSiteIndex, dir);
            pAphys[uiLinkIndex] = pGx[uiSiteIndex].MulC(pAphys[uiLinkIndex]);
            pAphys[uiLinkIndex].MulDagger(pGx[uiSiteIndex]);
        }
    }
}

#pragma endregion


#pragma endregion

__CLGIMPLEMENT_CLASS(CGaugeFixingRandom)

void CGaugeFixingRandom::Initial(class CLatticeData* pOwner, const CParameters& params)
{
    m_pOwner = pOwner;
    checkCudaErrors(hipMalloc((void**)& m_pG, _HC_Volume * sizeof(deviceSU3)));
}

void CGaugeFixingRandom::GaugeFixing(CFieldGauge* pResGauge)
{
    if (NULL == pResGauge || EFT_GaugeSU3 != pResGauge->GetFieldType())
    {
        appCrucial(_T("CGaugeFixingLandauCornell only implemented with gauge SU3!\n"));
        return;
    }
    CFieldGaugeSU3* pGaugeSU3 = dynamic_cast<CFieldGaugeSU3*>(pResGauge);

    preparethread;
    _kernelRandomGauge << <block, threads >> > (m_pG);
    _kernelGaugeTransformRandom << <block, threads >> > (m_pG, pGaugeSU3->m_pDeviceData);
}

void CGaugeFixingRandom::AlsoFixingFermion(CFieldFermionWilsonSquareSU3* pFermion) const
{
    preparethread;
    _kernelGaugeTransformFermion << <block, threads >> > (m_pG, pFermion->m_pDeviceData);
}

void CGaugeFixingRandom::AlsoFixingAphys(CFieldGauge* pGauge) const
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CGaugeFixingLandauCornell only implemented with gauge SU3!\n"));
        return;
    }
    CFieldGaugeSU3* pGaugeSU3 = dynamic_cast<CFieldGaugeSU3*>(pGauge);

    preparethread;
    _kernelGaugeTransformAPhys << <block, threads >> > (m_pG, pGaugeSU3->m_pDeviceData);
}

CCString CGaugeFixingRandom::GetInfos(const CCString& tab) const
{
    CCString sRet;
    sRet = sRet + tab + _T("Name : CGaugeFixingRandom\n");
    return sRet;
}

__END_NAMESPACE


//=============================================================================
// END OF FILE
//=============================================================================