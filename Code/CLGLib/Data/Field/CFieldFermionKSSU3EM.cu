#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CFieldFermionKSSU3EM.cu
// 
// DESCRIPTION:
// 
//
// REVISION:
//  [10/06/2020 nbale]
//=============================================================================

#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

__CLGIMPLEMENT_CLASS(CFieldFermionKSSU3EM)

#pragma region DOperator


#pragma region kernel

__global__ void _CLG_LAUNCH_BOUND
_kernelDFermionKSEM_Simple(
    const deviceSU3Vector* __restrict__ pDeviceData,
    const deviceSU3* __restrict__ pGauge,
    const SIndex* __restrict__ pGaugeMove,
    const SIndex* __restrict__ pFermionMove,
    const BYTE* __restrict__ pEtaTable,
    deviceSU3Vector* pResultData,
    Real fam,
    Real fqEz,
    Real fqBz,
    UBOOL bShiftCenter,
    SSmallInt4 sCenter,
    BYTE byFieldId,
    UBOOL bDDagger,
    EOperatorCoefficientType eCoeff,
    Real fCoeff,
    CLGComplex cCoeff)
{
    intokernalInt4;

    deviceSU3Vector result = deviceSU3Vector::makeZeroSU3Vector();
    pResultData[uiSiteIndex] = pDeviceData[uiSiteIndex];

    Real fX = static_cast<Real>(sSite4.x - sCenter.x);
    Real fY = static_cast<Real>(sSite4.y - sCenter.y);
    Real fZ = static_cast<Real>(sSite4.z - sCenter.z);
    //SSmallInt4 site_shift = sSite4;
    //site_shift.x = site_shift.x - 1;
    //site_shift = __deviceSiteIndexToInt4(__idx->m_pDeviceIndexPositionToSIndex[byFieldId][__bi(site_shift)].m_uiSiteIndex);
    //Real fXm = static_cast<Real>(sSite4.x - sCenter.x);
    //site_shift = sSite4;
    //site_shift.y = site_shift.y - 1;
    //site_shift = __deviceSiteIndexToInt4(__idx->m_pDeviceIndexPositionToSIndex[byFieldId][__bi(site_shift)].m_uiSiteIndex);
    //Real fYm = static_cast<Real>(sSite4.y - sCenter.y);
    //site_shift = sSite4;
    //site_shift.z = site_shift.z - 1;
    //site_shift = __deviceSiteIndexToInt4(__idx->m_pDeviceIndexPositionToSIndex[byFieldId][__bi(site_shift)].m_uiSiteIndex);
    //Real fZm = static_cast<Real>(sSite4.z - sCenter.z);

    if (bShiftCenter)
    {
        fX += F(0.5);
        fY += F(0.5);
        //fXm += F(0.5);
        //fYm += F(0.5);
    }
    fX = -fX * fqBz;
    fY = fY * fqBz;
    fZ = fZ * fqEz;
    //fXm = -fXm * fqBz;
    //fYm = fYm * fqBz;
    //fZm = fZm * fqEz;

    //idir = mu
    for (UINT idir = 0; idir < _DC_Dir; ++idir)
    {
        //Get Gamma mu
        const Real eta_mu = (1 == ((pEtaTable[uiSiteIndex] >> idir) & 1)) ? F(-1.0) : F(1.0);

        //x, mu
        const UINT linkIndex = _deviceGetLinkIndex(uiSiteIndex, idir);

        const SIndex& x_m_mu_Gauge = pGaugeMove[linkIndex];

        const SIndex& x_p_mu_Fermion = pFermionMove[2 * linkIndex];
        const SIndex& x_m_mu_Fermion = pFermionMove[2 * linkIndex + 1];

        //Assuming periodic
        //get U(x,mu), U^{dagger}(x-mu), 
        deviceSU3 x_Gauge_element = pGauge[linkIndex];
        if (0 == idir)
        {
            x_Gauge_element.MulComp(_make_cuComplex(_cos(fY), _sin(fY)));
        }
        else if (1 == idir)
        {
            x_Gauge_element.MulComp(_make_cuComplex(_cos(fX), _sin(fX)));
        }
        else if (3 == idir)
        {
            x_Gauge_element.MulComp(_make_cuComplex(_cos(fZ), _sin(fZ)));
        }
        deviceSU3 x_m_mu_Gauge_element = pGauge[_deviceGetLinkIndex(x_m_mu_Gauge.m_uiSiteIndex, idir)];
        if (0 == idir)
        {
            x_m_mu_Gauge_element.MulComp(_make_cuComplex(_cos(fY), _sin(fY)));
        }
        else if (1 == idir)
        {
            x_m_mu_Gauge_element.MulComp(_make_cuComplex(_cos(fX), _sin(fX)));
        }
        else if (3 == idir)
        {
            x_m_mu_Gauge_element.MulComp(_make_cuComplex(_cos(fZ), _sin(fZ)));
        }
        if (x_m_mu_Gauge.NeedToDagger())
        {
            x_m_mu_Gauge_element.Dagger();
        }

        //U(x,mu) phi(x+ mu)
        deviceSU3Vector u_phi_x_p_m = x_Gauge_element.MulVector(pDeviceData[x_p_mu_Fermion.m_uiSiteIndex]);
        if (x_p_mu_Fermion.NeedToOpposite())
        {
            u_phi_x_p_m.MulReal(F(-1.0));
        }

        //U^{dagger}(x-mu) phi(x-mu)
        deviceSU3Vector u_dagger_phi_x_m_m = x_m_mu_Gauge_element.MulVector(pDeviceData[x_m_mu_Fermion.m_uiSiteIndex]);
        if (x_m_mu_Fermion.NeedToOpposite())
        {
            u_phi_x_p_m.Add(u_dagger_phi_x_m_m);
        }
        else
        {
            u_phi_x_p_m.Sub(u_dagger_phi_x_m_m);
        }
        u_phi_x_p_m.MulReal(eta_mu);
        result.Add(u_phi_x_p_m);
    }

    pResultData[uiSiteIndex].MulReal(fam);
    if (bDDagger)
    {
        pResultData[uiSiteIndex].Sub(result);
    }
    else
    {
        pResultData[uiSiteIndex].Add(result);
    }

    switch (eCoeff)
    {
    case EOCT_Real:
        pResultData[uiSiteIndex].MulReal(fCoeff);
        break;
    case EOCT_Complex:
        pResultData[uiSiteIndex].MulComp(cCoeff);
        break;
    }
}

#pragma endregion

#pragma region Derivate

__global__ void _CLG_LAUNCH_BOUND
_kernelDFermionKSForceEM_Simple(
    const deviceSU3* __restrict__ pGauge,
    deviceSU3* pForce,
    const SIndex* __restrict__ pFermionMove,
    const BYTE* __restrict__ pEtaTable,
    const deviceSU3Vector* const* __restrict__ pFermionPointers,
    const Real* __restrict__ pNumerators,
    UINT uiRational,
    Real fqEz,
    Real fqBz,
    UBOOL bShiftCenter,
    SSmallInt4 sCenter,
    BYTE byFieldId)
{
    intokernalInt4;

    Real fX = static_cast<Real>(sSite4.x - sCenter.x);
    Real fY = static_cast<Real>(sSite4.y - sCenter.y);
    Real fZ = static_cast<Real>(sSite4.z - sCenter.z);
    if (bShiftCenter)
    {
        fX += F(0.5);
        fY += F(0.5);
    }
    fX = -fX * fqBz;
    fY = fY * fqBz;
    fZ = fZ * fqEz;

    //idir = mu
    for (UINT idir = 0; idir < _DC_Dir; ++idir)
    {
        //Get Gamma mu
        const Real eta_mu = (1 == ((pEtaTable[uiSiteIndex] >> idir) & 1)) ? F(-1.0) : F(1.0);
        //x, mu
        const UINT linkIndex = _deviceGetLinkIndex(uiSiteIndex, idir);

        const SIndex& x_p_mu_Fermion = pFermionMove[2 * linkIndex];

        for (UINT uiR = 0; uiR < uiRational; ++uiR)
        {
            const deviceSU3Vector* phi_i = pFermionPointers[uiR];
            const deviceSU3Vector* phi_id = pFermionPointers[uiR + uiRational];

            deviceSU3Vector toContract = pGauge[linkIndex].MulVector(phi_i[x_p_mu_Fermion.m_uiSiteIndex]);
            if (0 == idir)
            {
                toContract.MulComp(_make_cuComplex(_cos(fY), _sin(fY)));
            }
            else if (1 == idir)
            {
                toContract.MulComp(_make_cuComplex(_cos(fX), _sin(fX)));
            }
            else if (3 == idir)
            {
                toContract.MulComp(_make_cuComplex(_cos(fZ), _sin(fZ)));
            }
            deviceSU3 thisTerm = deviceSU3::makeSU3ContractV(phi_id[uiSiteIndex], toContract);

            toContract = pGauge[linkIndex].MulVector(phi_id[x_p_mu_Fermion.m_uiSiteIndex]);
            if (0 == idir)
            {
                toContract.MulComp(_make_cuComplex(_cos(fY), _sin(fY)));
            }
            else if (1 == idir)
            {
                toContract.MulComp(_make_cuComplex(_cos(fX), _sin(fX)));
            }
            else if (3 == idir)
            {
                toContract.MulComp(_make_cuComplex(_cos(fZ), _sin(fZ)));
            }
            thisTerm.Add(deviceSU3::makeSU3ContractV(toContract, phi_i[uiSiteIndex]));

            if (x_p_mu_Fermion.NeedToOpposite())
            {
                thisTerm.MulReal(eta_mu * pNumerators[uiR] * F(-1.0));
            }
            else
            {
                thisTerm.MulReal(eta_mu * pNumerators[uiR]);
            }

            thisTerm.Ta();

            pForce[linkIndex].Sub(thisTerm);
        }
    }

}

#pragma endregion


#pragma endregion

#pragma region D and derivate

void CFieldFermionKSSU3EM::DOperatorKS(void* pTargetBuffer, const void* pBuffer,
    const void* pGaugeBuffer, Real f2am,
    UBOOL bDagger, EOperatorCoefficientType eOCT,
    Real fRealCoeff, const CLGComplex& cCmpCoeff) const
{
    deviceSU3Vector* pTarget = (deviceSU3Vector*)pTargetBuffer;
    const deviceSU3Vector* pSource = (const deviceSU3Vector*)pBuffer;
    const deviceSU3* pGauge = (const deviceSU3*)pGaugeBuffer;

    preparethread;
    _kernelDFermionKSEM_Simple << <block, threads >> > (
        pSource,
        pGauge,
        appGetLattice()->m_pIndexCache->m_pGaugeMoveCache[m_byFieldId],
        appGetLattice()->m_pIndexCache->m_pFermionMoveCache[m_byFieldId],
        appGetLattice()->m_pIndexCache->m_pEtaMu,
        pTarget,
        f2am,
        m_fa2Ez,
        m_fa2Bz,
        m_bEachSiteEta,
        CCommonData::m_sCenter,
        m_byFieldId,
        bDagger,
        eOCT,
        fRealCoeff,
        cCmpCoeff);
}

void CFieldFermionKSSU3EM::DerivateD0(
    void* pForce,
    const void* pGaugeBuffer) const
{
    preparethread;
    _kernelDFermionKSForceEM_Simple << <block, threads >> > (
        (const deviceSU3*)pGaugeBuffer,
        (deviceSU3*)pForce,
        appGetLattice()->m_pIndexCache->m_pFermionMoveCache[m_byFieldId],
        appGetLattice()->m_pIndexCache->m_pEtaMu,
        m_pRationalFieldPointers,
        m_pMDNumerator,
        m_rMD.m_uiDegree,
        m_fa2Ez,
        m_fa2Bz,
        m_bEachSiteEta,
        CCommonData::m_sCenter,
        m_byFieldId);
}

#pragma endregion

void CFieldFermionKSSU3EM::InitialOtherParameters(CParameters& params)
{
    CFieldFermionKSSU3::InitialOtherParameters(params);
    params.FetchValueReal(_T("Qa2Ez"), m_fa2Ez);
    params.FetchValueReal(_T("Qa2Bz"), m_fa2Bz);
    m_bEachSiteEta = TRUE;
}

void CFieldFermionKSSU3EM::CopyTo(CField* U) const
{
    CFieldFermionKSSU3::CopyTo(U);
    CFieldFermionKSSU3EM* target = dynamic_cast<CFieldFermionKSSU3EM*>(U);
    if (NULL != target)
    {
        target->m_fa2Ez = m_fa2Ez;
        target->m_fa2Bz = m_fa2Bz;
    }
}

CCString CFieldFermionKSSU3EM::GetInfos(const CCString& tab) const
{
    CCString sRet = tab + _T("Name : CFieldFermionKSSU3R\n");
    sRet = sRet + tab + _T("Mass (2am) : ") + appFloatToString(m_f2am) + _T("\n");
    sRet = sRet + tab + _T("MD Rational (c) : ") + appFloatToString(m_rMD.m_fC) + _T("\n");
    sRet = sRet + tab + _T("MC Rational (c) : ") + appFloatToString(m_rMC.m_fC) + _T("\n");
    sRet = sRet + tab + _T("Q x a^2Ez : ") + appFloatToString(m_fa2Ez) + _T("\n");
    sRet = sRet + tab + _T("Q x a^2Bz : ") + appFloatToString(m_fa2Bz) + _T("\n");
    return sRet;
}

__END_NAMESPACE


//=============================================================================
// END OF FILE
//=============================================================================