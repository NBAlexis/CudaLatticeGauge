#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CFieldFermionKSTR<deviceVector, deviceGauge, vectorN>.cu
// 
// DESCRIPTION:
// 
//
// REVISION:
//  [09/23/2020 nbale]
//=============================================================================
#include "CLGLib_Private.h"
#include "Tools/Math/DeviceInlineTemplate.h"
#include "CFieldFermionKSTR.h"

__BEGIN_NAMESPACE


#pragma region DOperator

#pragma region kernel

/**
* When link n and n+mu, the coordinate is stick with n
* When link n and n-mu, the coordinate is stick with n-mu
* Irrelavent with tau
* Optimization: bXorY removed, block.x *= 2 
*/
template<typename deviceVector, typename deviceGauge>
__global__ void _CLG_LAUNCH_BOUND
_kernelDFermionKS_PR_XYTermT(
    const deviceVector * __restrict__ pDeviceData,
    const deviceGauge * __restrict__ pGauge,
    const BYTE * __restrict__ pEtaTable,
    deviceVector* pResultData,
    BYTE byFieldId,
    BYTE byGaugeFieldId,
    DOUBLE fOmega,
    SSmallInt4 sCenter,
    UBOOL bDDagger,
    EOperatorCoefficientType eCoeff,
    Real fCoeff,
    CLGComplex cCoeff)
{
    intokernalInt4;

    deviceVector result = _makeZero<deviceVector>();
    //const INT eta_tau = ((pEtaTable[uiSiteIndex] >> 3) & 1);
    const INT eta_tau = pEtaTable[uiSiteIndex] >> 3;

    #pragma unroll
    for (UINT idx = 0; idx < 8; ++idx)
    {
        const UBOOL bPlusMu  = idx & 2;
        const UBOOL bPlusTau = idx & 4;
        //x or y, and y or x is the derivate, not coefficient
        const UINT bXorY = idx & 1;
        const UINT bYorX = 1 - bXorY;
        SSmallInt4 sTargetSite = sSite4;
        SSmallInt4 sMidSite = sSite4;
        sTargetSite.m_byData4[bYorX] = sTargetSite.m_byData4[bYorX] + (bPlusMu ? 2 : -2);
        sMidSite.m_byData4[bYorX] = sMidSite.m_byData4[bYorX] + (bPlusMu ? 1 : -1);
        sTargetSite.w = sTargetSite.w + (bPlusTau ? 1 : -1);
        //We have anti-periodic boundary, so we need to use index out of lattice to get the correct sign
        const SIndex& sTargetBigIndex = __idx->m_pDeviceIndexPositionToSIndex[byFieldId][__bi(sTargetSite)];
        const SIndex& sMiddleBigIndex = __idx->m_pDeviceIndexPositionToSIndex[byFieldId][__bi(sMidSite)];
        sMidSite = __deviceSiteIndexToInt4(sMiddleBigIndex.m_uiSiteIndex);

        //note that bYorX = 1, it is x partial_y term, therefore is '-'
        //INT this_eta_tau = (bPlusTau ? eta_tau : ((pEtaTable[sTargetBigIndex.m_uiSiteIndex] >> 3) & 1))
        INT this_eta_tau = (bPlusTau ? eta_tau : (pEtaTable[sTargetBigIndex.m_uiSiteIndex] >> 3))
                         + bYorX;

        if (sTargetBigIndex.NeedToOpposite())
        {            
            this_eta_tau = this_eta_tau + 1;
        }

        deviceVector right = _mulVec(_deviceVXXTauOptimizedT(pGauge, sSite4, byGaugeFieldId, bXorY, bPlusMu, bPlusTau),
            pDeviceData[sTargetBigIndex.m_uiSiteIndex]);

        //when bXorY = 1, it is y partial _x, so is [1]
        //when bXorY = 0, it is x partial _y, so is [0]
        _mul(right, sMidSite.m_byData4[bXorY] - sCenter.m_byData4[bXorY] + F(0.5));

        if (!bPlusMu)
        {
            //for -2x, -2y terms, there is another minus sign
            this_eta_tau = this_eta_tau + 1;
        }

        if (this_eta_tau & 1)
        {
            _sub(result, right);
        }
        else
        {
            _add(result, right);
        }
    }

    if (bDDagger)
    {
        _mul(result, F(-0.25) * fOmega);
    }
    else
    {
        _mul(result, F(0.25) * fOmega);
    }

    switch (eCoeff)
    {
    case EOCT_Real:
        _mul(result, fCoeff);
        break;
    case EOCT_Complex:
        _mul(result, cCoeff);
        break;
    }

    _add(pResultData[uiSiteIndex], result);
}

template<typename deviceVector, typename deviceGauge>
__global__ void _CLG_LAUNCH_BOUND
_kernelDFermionKS_PR_XYTermRealT(
    const deviceVector* __restrict__ pDeviceData,
    const deviceGauge* __restrict__ pGauge,
    const BYTE* __restrict__ pEtaTable,
    deviceVector* pResultData,
    BYTE byFieldId,
    BYTE byGaugeFieldId,
    DOUBLE fOmega,
    SSmallInt4 sCenter,
    EOperatorCoefficientType eCoeff,
    Real fCoeff,
    CLGComplex cCoeff)
{
    intokernalInt4;

    deviceVector result = _makeZero<deviceVector>();
    //const INT eta_tau = ((pEtaTable[uiSiteIndex] >> 3) & 1);
    const INT eta_tau = pEtaTable[uiSiteIndex] >> 3;

    #pragma unroll
    for (UINT idx = 0; idx < 8; ++idx)
    {
        const UBOOL bPlusMu = idx & 2;
        const UBOOL bPlusTau = idx & 4;
        //x or y, and y or x is the derivate, not coefficient
        const UINT bXorY = idx & 1;
        const UINT bYorX = 1 - bXorY;
        SSmallInt4 sTargetSite = sSite4;
        SSmallInt4 sMidSite = sSite4;
        sTargetSite.m_byData4[bYorX] = sTargetSite.m_byData4[bYorX] + (bPlusMu ? 2 : -2);
        sMidSite.m_byData4[bYorX] = sMidSite.m_byData4[bYorX] + (bPlusMu ? 1 : -1);
        sTargetSite.w = sTargetSite.w + (bPlusTau ? 1 : -1);
        //We have anti-periodic boundary, so we need to use index out of lattice to get the correct sign
        const SIndex& sTargetBigIndex = __idx->m_pDeviceIndexPositionToSIndex[byFieldId][__bi(sTargetSite)];
        const SIndex& sMiddleBigIndex = __idx->m_pDeviceIndexPositionToSIndex[byFieldId][__bi(sMidSite)];
        sMidSite = __deviceSiteIndexToInt4(sMiddleBigIndex.m_uiSiteIndex);

        //note that bYorX = 1, it is x partial_y term, therefore is '-'
        //INT this_eta_tau = (bPlusTau ? eta_tau : ((pEtaTable[sTargetBigIndex.m_uiSiteIndex] >> 3) & 1))
        INT this_eta_tau = (bPlusTau ? eta_tau : (pEtaTable[sTargetBigIndex.m_uiSiteIndex] >> 3))
            + bYorX;

        if (sTargetBigIndex.NeedToOpposite())
        {
            this_eta_tau = this_eta_tau + 1;
        }

        deviceVector right = _mulVec(_deviceVXXTauOptimizedT(pGauge, sSite4, byGaugeFieldId, bXorY, bPlusMu, bPlusTau),
            pDeviceData[sTargetBigIndex.m_uiSiteIndex]);

        //when bXorY = 1, it is y partial _x, so is [1]
        //when bXorY = 0, it is x partial _y, so is [0]
        _mul(right, sMidSite.m_byData4[bXorY] - sCenter.m_byData4[bXorY] + F(0.5));

        if (!bPlusMu)
        {
            //for -2x, -2y terms, there is another minus sign
            this_eta_tau = this_eta_tau + 1;
        }

        if (this_eta_tau & 1)
        {
            _sub(result, right);
        }
        else
        {
            _add(result, right);
        }
    }

    //if (bDDagger)
    //{
    //    _mul(result, F(-0.25) * fOmega);
    //}
    //else
    //{
        //_mul(result, F(0.25) * fOmega);
    //}
    _mul(result, _make_cuComplex(F(0.0), F(-0.25) * fOmega));

    switch (eCoeff)
    {
    case EOCT_Real:
        _mul(result, fCoeff);
        break;
    case EOCT_Complex:
        _mul(result, cCoeff);
        break;
    }

    _add(pResultData[uiSiteIndex], result);
}

template<typename deviceVector, typename deviceGauge>
__global__ void _CLG_LAUNCH_BOUND
_kernelDFermionKS_PR_XYTau_TermT(
    const deviceVector* __restrict__ pDeviceData,
    const deviceGauge* __restrict__ pGauge,
    deviceVector* pResultData,
    BYTE byFieldId,
    BYTE byGaugeFieldId,
    DOUBLE fOmega,
    UBOOL bDDagger,
    EOperatorCoefficientType eCoeff,
    Real fCoeff,
    CLGComplex cCoeff)
{
    intokernalInt4;

    deviceVector result = _makeZero<deviceVector>();

    #pragma unroll
    for (UINT idx = 0; idx < 8; ++idx)
    {
        const UBOOL bPlusX = (0 != (idx & 1));
        const UBOOL bPlusY = (0 != (idx & 2));
        const UBOOL bPlusT = (0 != (idx & 4));

        SSmallInt4 sOffset = sSite4;
        sOffset.x = sOffset.x + (bPlusX ? 1 : -1);
        sOffset.y = sOffset.y + (bPlusY ? 1 : -1);
        sOffset.w = sOffset.w + (bPlusT ? 1 : -1);

        //We have anti-periodic boundary, so we need to use index out of lattice to get the correct sign
        const SIndex& sTargetBigIndex = __idx->m_pDeviceIndexPositionToSIndex[byFieldId][__bi(sOffset)];
        
        const deviceVector right = _mulVec(_deviceVXYTOptimizedT(pGauge, sSite4, byGaugeFieldId, bPlusX, bPlusY, bPlusT), pDeviceData[sTargetBigIndex.m_uiSiteIndex]);
        const SSmallInt4 site_target = __deviceSiteIndexToInt4(sTargetBigIndex.m_uiSiteIndex);

        //eta124 of site is almost always -target, so use left or right is same
        //The only exception is on the boundary
        INT eta124 = bPlusT ? (sSite4.y + sSite4.z) : (site_target.y + site_target.z + 1);

        if (sTargetBigIndex.NeedToOpposite())
        {
            eta124 = eta124 + 1;
        }

        if (eta124 & 1)
        {
            _sub(result, right);
        }
        else
        {
            _add(result, right);
        }
    }

    if (bDDagger)
    {
        _mul(result, -F(0.125) * fOmega);
    }
    else
    {
        _mul(result, F(0.125) * fOmega);
    }

    switch (eCoeff)
    {
    case EOCT_Real:
        _mul(result, fCoeff);
        break;
    case EOCT_Complex:
        _mul(result, cCoeff);
        break;
    }

    _add(pResultData[uiSiteIndex], result);
}

template<typename deviceVector, typename deviceGauge>
__global__ void _CLG_LAUNCH_BOUND
_kernelDFermionKS_PR_XYTau_TermRealT(
    const deviceVector* __restrict__ pDeviceData,
    const deviceGauge* __restrict__ pGauge,
    deviceVector* pResultData,
    BYTE byFieldId,
    BYTE byGaugeFieldId,
    DOUBLE fOmega,
    EOperatorCoefficientType eCoeff,
    Real fCoeff,
    CLGComplex cCoeff)
{
    intokernalInt4;

    deviceVector result = _makeZero<deviceVector>();

    #pragma unroll
    for (UINT idx = 0; idx < 8; ++idx)
    {
        const UBOOL bPlusX = (0 != (idx & 1));
        const UBOOL bPlusY = (0 != (idx & 2));
        const UBOOL bPlusT = (0 != (idx & 4));

        SSmallInt4 sOffset = sSite4;
        sOffset.x = sOffset.x + (bPlusX ? 1 : -1);
        sOffset.y = sOffset.y + (bPlusY ? 1 : -1);
        sOffset.w = sOffset.w + (bPlusT ? 1 : -1);

        //We have anti-periodic boundary, so we need to use index out of lattice to get the correct sign
        const SIndex& sTargetBigIndex = __idx->m_pDeviceIndexPositionToSIndex[byFieldId][__bi(sOffset)];

        const deviceVector right = _mulVec(_deviceVXYTOptimizedT(pGauge, sSite4, byGaugeFieldId, bPlusX, bPlusY, bPlusT), pDeviceData[sTargetBigIndex.m_uiSiteIndex]);
        const SSmallInt4 site_target = __deviceSiteIndexToInt4(sTargetBigIndex.m_uiSiteIndex);

        //eta124 of site is almost always -target, so use left or right is same
        //The only exception is on the boundary
        INT eta124 = bPlusT ? (sSite4.y + sSite4.z) : (site_target.y + site_target.z + 1);

        if (sTargetBigIndex.NeedToOpposite())
        {
            eta124 = eta124 + 1;
        }

        if (eta124 & 1)
        {
            _sub(result, right);
        }
        else
        {
            _add(result, right);
        }
    }

    //if (bDDagger)
    //{
    //    _mul(result, -F(0.125) * fOmega);
    //}
    //else
    //{
    //    _mul(result, F(0.125) * fOmega);
    //}
    _mul(result, _make_cuComplex(F(0.0), F(-0.125) * fOmega));

    switch (eCoeff)
    {
    case EOCT_Real:
        _mul(result, fCoeff);
        break;
    case EOCT_Complex:
        _mul(result, cCoeff);
        break;
    }

    _add(pResultData[uiSiteIndex], result);
}

#pragma endregion

#pragma region Derivate

/**
 * Have n, n->n1, n->n2,
 * 1. we need to obtain V_(n, n1) , V_(n, n2)
 * 2. we need phi(n1), phi(n2), phid(n1), phid(n2)
 *
 * byContribution: 0 for mu, 1 for tau, 2 for both mu and tau
 *
 * iTau = 1 for +t, -1 for -t
 */
template<typename deviceVector, typename deviceGauge>
__global__ void _CLG_LAUNCH_BOUND
_kernelDFermionKSForce_PR_XYTermT( 
    const deviceGauge* __restrict__ pGauge,
    deviceGauge* pForce,
    const BYTE* __restrict__ pEtaTable,
    const deviceVector* const* __restrict__ pFermionPointers,
    const Real* __restrict__ pNumerators,
    UINT uiRational,
    BYTE byFieldId,
    DOUBLE fOmega,
    BYTE byMu, INT iTau,
    INT pathLdir1, INT pathLdir2, INT pathLdir3, BYTE Llength,
    INT pathRdir1, INT pathRdir2, INT pathRdir3, BYTE Rlength,
    BYTE byContribution)
{
    intokernalInt4;
    //const UINT uiBigIdx = __bi(sSite4);

    //=================================
    // 1. Find n1, n2
    INT Ldirs[3] = { pathLdir1, pathLdir2, pathLdir3 };
    INT Rdirs[3] = { pathRdir1, pathRdir2, pathRdir3 };
    SSmallInt4 site_n1 = _deviceSmallInt4OffsetC(sSite4, Ldirs, Llength);
    const SIndex& sn1 = __idx->m_pDeviceIndexPositionToSIndex[byFieldId][__bi(site_n1)];
    const SIndex& sn2 = __idx->m_pDeviceIndexPositionToSIndex[byFieldId][__bi(_deviceSmallInt4OffsetC(sSite4, Rdirs, Rlength))];
    //const SSmallInt4 middleSite = _deviceSmallInt4OffsetC(site_n1, byMu + 1);
    //From now on, site_n1 is smiddle
    site_n1 = _deviceSmallInt4OffsetC(site_n1, byMu + 1);
    const SIndex& smiddle = __idx->m_pDeviceIndexPositionToSIndex[byFieldId][__bi(site_n1)];
    
    site_n1 = __deviceSiteIndexToInt4(smiddle.m_uiSiteIndex);
    //y Dx and -x Dy
    const Real fNv = (0 == byMu)
        ? static_cast<Real>(site_n1.y - _DC_Centery + F(0.5))
        : static_cast<Real>(_DC_Centerx - site_n1.x - F(0.5));

    //=================================
    // 2. Find V(n,n1), V(n,n2)
    const deviceGauge vnn1 = _deviceLinkT(pGauge, sSite4, Llength, 1, Ldirs);
    const deviceGauge vnn2 = _deviceLinkT(pGauge, sSite4, Rlength, 1, Rdirs);

    for (BYTE rfieldId = 0; rfieldId < uiRational; ++rfieldId)
    {
        const deviceVector* phi_i = pFermionPointers[rfieldId];
        const deviceVector* phi_id = pFermionPointers[rfieldId + uiRational];
        //=================================
        // 3. Find phi_{1,2,3,4}(n1), phi_i(n2)
        deviceVector phi1 = _mulVec(vnn1, phi_id[sn1.m_uiSiteIndex]);
        deviceVector phi2 = _mulVec(vnn2, phi_i[sn2.m_uiSiteIndex]);
        deviceVector phi3 = _mulVec(vnn1, phi_i[sn1.m_uiSiteIndex]);
        deviceVector phi4 = _mulVec(vnn2, phi_id[sn2.m_uiSiteIndex]);
        if (sn1.NeedToOpposite())
        {
            _mul(phi1, F(-1.0));
            _mul(phi3, F(-1.0));
        }
        if (sn2.NeedToOpposite())
        {
            _mul(phi2, F(-1.0));
            _mul(phi4, F(-1.0));
        }
        deviceGauge res = _makeContract<deviceGauge, deviceVector>(phi1, phi2);
        _add(res, _makeContract<deviceGauge, deviceVector>(phi4, phi3));
        _ta(res);
        const Real eta_tau = (iTau > 0 ? 
            ((pEtaTable[sn1.m_uiSiteIndex] >> 3) & 1) 
            : ((pEtaTable[sn2.m_uiSiteIndex] >> 3) & 1) )
            ? F(-1.0) : F(1.0);
        _mul(res, OneOver12 * fOmega * fNv * pNumerators[rfieldId] * eta_tau);

        //For mu
        if (0 == byContribution || 2 == byContribution)
        {
            const UINT linkIndex = _deviceGetLinkIndex(uiSiteIndex, byMu);
            _sub(pForce[linkIndex], res);
        }

        //For tau
        if (1 == byContribution || 2 == byContribution)
        {
            const UINT linkIndex = _deviceGetLinkIndex(uiSiteIndex, 3);
            if (iTau > 0)
            {
                _sub(pForce[linkIndex], res);
            }
            else
            {
                _add(pForce[linkIndex], res);
            }
        }
    }
}

/**
 *
 */
template<typename deviceVector, typename deviceGauge>
__global__ void _CLG_LAUNCH_BOUND
_kernelDFermionKSForce_PR_XYTau_TermT(
    const deviceGauge* __restrict__ pGauge,
    deviceGauge* pForce,
    const deviceVector* const* __restrict__ pFermionPointers,
    const Real* __restrict__ pNumerators,
    UINT uiRational,
    BYTE byFieldId,
    DOUBLE fOmega,
    INT pathLdir1, INT pathLdir2, INT pathLdir3, BYTE Llength,
    INT pathRdir1, INT pathRdir2, INT pathRdir3, BYTE Rlength)
{
    intokernalInt4;
    //const UINT uiBigIdx = __bi(sSite4);

    //=================================
    // 1. Find n1, n2
    INT Ldirs[3] = { pathLdir1, pathLdir2, pathLdir3 };
    INT Rdirs[3] = { pathRdir1, pathRdir2, pathRdir3 };
    const SSmallInt4 siten1 = _deviceSmallInt4OffsetC(sSite4, Ldirs, Llength);
    const SSmallInt4 siten2 = _deviceSmallInt4OffsetC(sSite4, Rdirs, Rlength);
    const SIndex& sn1 = __idx->m_pDeviceIndexPositionToSIndex[byFieldId][__bi(siten1)];
    const SIndex& sn2 = __idx->m_pDeviceIndexPositionToSIndex[byFieldId][__bi(siten2)];

    //Why use sn2? shouldn't it be sn1?
    const Real eta124 = _deviceEta124(__deviceSiteIndexToInt4(sn1.m_uiSiteIndex));
    //=================================
    // 2. Find V(n,n1), V(n,n2)
    const deviceGauge vnn1 = _deviceLinkT(pGauge, sSite4, Llength, 1, Ldirs);
    const deviceGauge vnn2 = _deviceLinkT(pGauge, sSite4, Rlength, 1, Rdirs);

    for (BYTE rfieldId = 0; rfieldId < uiRational; ++rfieldId)
    {
        const deviceVector* phi_i = pFermionPointers[rfieldId];
        const deviceVector* phi_id = pFermionPointers[rfieldId + uiRational];

        //=================================
        // 3. Find phi_{1,2,3,4}(n1), phi_i(n2)
        deviceVector phi1 = _mulVec(vnn1, phi_id[sn1.m_uiSiteIndex]);
        deviceVector phi2 = _mulVec(vnn2, phi_i[sn2.m_uiSiteIndex]);
        deviceVector phi3 = _mulVec(vnn1, phi_i[sn1.m_uiSiteIndex]);
        deviceVector phi4 = _mulVec(vnn2, phi_id[sn2.m_uiSiteIndex]);
        if (sn1.NeedToOpposite())
        {
            _mul(phi1, F(-1.0));
            _mul(phi3, F(-1.0));
        }
        if (sn2.NeedToOpposite())
        {
            _mul(phi2, F(-1.0));
            _mul(phi4, F(-1.0));
        }
        deviceGauge res = _makeContract<deviceGauge, deviceVector>(phi1, phi2);
        //This was phi2 phi1+ * eta124(n1) - phi3 phi4+ * eta124(n2)
        //The sign of the second term is because of 'dagger'
        //However, eta124(n1) = -eta124(n2), so use Add directly.
        _add(res, _makeContract<deviceGauge, deviceVector>(phi4, phi3));
        _ta(res);
        _mul(res, OneOver48 * static_cast<Real>(fOmega) * pNumerators[rfieldId] * eta124);

        //Use eta124 of n2 so Add left Sub right
        //Change to use eta124 of n1, Sub left and Add right
        if (pathLdir1 > 0)
        {
            const UINT linkIndex = _deviceGetLinkIndex(uiSiteIndex, pathLdir1 - 1);
            _add(pForce[linkIndex], res);
        }

        if (pathRdir1 > 0)
        {
            const UINT linkIndex = _deviceGetLinkIndex(uiSiteIndex, pathRdir1 - 1);
            _sub(pForce[linkIndex], res);
        }
    }

}

#pragma endregion


#pragma endregion

#pragma region D and derivate

template<typename deviceVector, typename deviceGauge, INT vectorN>
void CFieldFermionKSTR<deviceVector, deviceGauge, vectorN>::DOperatorKS(void* pTargetBuffer, const void* pBuffer,
    const void* pGaugeBuffer, BYTE byGaugeFieldId, Real f2am,
    UBOOL bDagger, EOperatorCoefficientType eOCT,
    Real fRealCoeff, const CLGComplex& cCmpCoeff) const
{
    CFieldFermionKST<deviceVector, deviceGauge, vectorN>::DOperatorKS(pTargetBuffer, pBuffer, pGaugeBuffer, byGaugeFieldId, f2am, bDagger, eOCT, fRealCoeff, cCmpCoeff);

    deviceVector* pTarget = (deviceVector*)pTargetBuffer;
    const deviceVector* pSource = (const deviceVector*)pBuffer;
    const deviceGauge* pGauge = (const deviceGauge*)pGaugeBuffer;


    preparethread;
    if (m_bRealRotation)
    {
        if (bDagger)
        {
            appCrucial(_T("D dagger is not supported for real rotation!\n"));
        }

        _kernelDFermionKS_PR_XYTermRealT << <block, threads >> > (
            pSource,
            pGauge,
            appGetLattice()->m_pIndexCache->m_pEtaMu,
            pTarget,
            this->m_byFieldId,
            byGaugeFieldId,
            CCommonData::m_fOmega,
            _HC_Center,
            eOCT,
            fRealCoeff,
            cCmpCoeff);

        _kernelDFermionKS_PR_XYTau_TermRealT << <block, threads >> > (
            pSource,
            pGauge,
            pTarget,
            this->m_byFieldId,
            byGaugeFieldId,
            CCommonData::m_fOmega,
            eOCT,
            fRealCoeff,
            cCmpCoeff);

        return;
    }

    _kernelDFermionKS_PR_XYTermT << <block, threads >> > (
        pSource,
        pGauge,
        appGetLattice()->m_pIndexCache->m_pEtaMu,
        pTarget,
        this->m_byFieldId,
        byGaugeFieldId,
        CCommonData::m_fOmega,
        _HC_Center,
        bDagger,
        eOCT,
        fRealCoeff,
        cCmpCoeff);

#if 1

    _kernelDFermionKS_PR_XYTau_TermT << <block, threads >> > (
        pSource,
        pGauge,
        pTarget,
        this->m_byFieldId,
        byGaugeFieldId,
        CCommonData::m_fOmega,
        bDagger,
        eOCT,
        fRealCoeff,
        cCmpCoeff);


#endif

}

template<typename deviceVector, typename deviceGauge, INT vectorN>
void CFieldFermionKSTR<deviceVector, deviceGauge, vectorN>::DerivateD0(
    void* pForce,
    const void* pGaugeBuffer, BYTE byGaugeFieldId) const
{
    CFieldFermionKST<deviceVector, deviceGauge, vectorN>::DerivateD0(pForce, pGaugeBuffer, byGaugeFieldId);

    if (m_bRealRotation)
    {
        appCrucial(_T("DerivateD0 is not supported for real rotation!\n"));
    }

    preparethread;
    #pragma region X Y Term

    INT mu[2] = { 0, 1 };
    for (INT imu = 0; imu < 2; ++imu)
    {
        INT dirs[6][3] =
        {
            {4, mu[imu] + 1, mu[imu] + 1},
            {mu[imu] + 1, 4, mu[imu] + 1},
            {mu[imu] + 1, mu[imu] + 1, 4},
            //{4, -mu[imu] - 1, -mu[imu] - 1},
            //{-mu[imu] - 1, 4, -mu[imu] - 1},
            //{-mu[imu] - 1, -mu[imu] - 1, 4},
            {mu[imu] + 1, mu[imu] + 1, -4},
            {mu[imu] + 1, -4, mu[imu] + 1},
            {-4, mu[imu] + 1, mu[imu] + 1},
        };

        INT iTau[6] = { 1, 1, 1, -1, -1, -1 };
        BYTE contributionOf[6][4] =
        {
            {1, 0, 0, 3},
            {0, 1, 0, 3},
            {0, 0, 1, 3},
            //{1, 3, 0, 0},
            //{3, 2, 3, 0},
            //{3, 0, 2, 3},
            {0, 0, 3, 1},
            {0, 3, 2, 3},
            {3, 2, 0, 3},
        };

        for (INT pathidx = 0; pathidx < 6; ++pathidx)
        {
            for (INT iSeperation = 0; iSeperation < 4; ++iSeperation)
            {
                if (3 == contributionOf[pathidx][iSeperation])
                {
                    continue;
                }

                INT L[3] = { 0, 0, 0 };
                INT R[3] = { 0, 0, 0 };
                BYTE LLength = 0;
                BYTE RLength = 0;

                Seperate(dirs[pathidx], iSeperation, L, R, LLength, RLength);

                _kernelDFermionKSForce_PR_XYTermT << <block, threads >> > (
                    (const deviceGauge*)pGaugeBuffer,
                    (deviceGauge*)pForce,
                    appGetLattice()->m_pIndexCache->m_pEtaMu,
                    this->m_pRationalFieldPointers,
                    this->m_pMDNumerator,
                    this->m_rMD.m_uiDegree,
                    this->m_byFieldId,
                    CCommonData::m_fOmega, 
                    static_cast<BYTE>(imu), iTau[pathidx],
                    L[0], L[1], L[2], LLength,
                    R[0], R[1], R[2], RLength,
                    contributionOf[pathidx][iSeperation]
                    );
            }
        }
    }

    #pragma endregion

#if 1

    #pragma region Polarization term

    //===========================
    //polarization terms
    //ilinkType is +-x +-y +t,
    //INT linkTypes[4][3] =
    //{
    //    {1, 2, 4},
    //    {1, 2, -4},
    //    {-1, 2, 4},
    //    {-1, 2, -4}
    //};
    INT linkTypes[4][3] =
    {
        {1, 2, 4},
        {1, -2, 4},
        {-1, 2, 4},
        {-1, -2, 4}
    };

    for (INT ilinkType = 0; ilinkType < 4; ++ilinkType)
    {
        INT sixlinks[6][3] =
        {
            {linkTypes[ilinkType][0], linkTypes[ilinkType][1], linkTypes[ilinkType][2]},
            {linkTypes[ilinkType][0], linkTypes[ilinkType][2], linkTypes[ilinkType][1]},
            {linkTypes[ilinkType][1], linkTypes[ilinkType][0], linkTypes[ilinkType][2]},
            {linkTypes[ilinkType][1], linkTypes[ilinkType][2], linkTypes[ilinkType][0]},
            {linkTypes[ilinkType][2], linkTypes[ilinkType][0], linkTypes[ilinkType][1]},
            {linkTypes[ilinkType][2], linkTypes[ilinkType][1], linkTypes[ilinkType][0]}
        };

        for (INT isixtype = 0; isixtype < 6; ++isixtype)
        {
            //bearly no change of time, because force calculation is not frequent
            /*
            _giveupkernelDFermionKSForce_PR_XYTau_Term2 << <block, threads >> > (
                (const deviceGauge*)pGaugeBuffer,
                (deviceGauge*)pForce,
                m_pRationalFieldPointers,
                m_pMDNumerator,
                m_rMD.m_uiDegree,
                m_byFieldId,
                CCommonData::m_fOmega,
                sixlinks[isixtype][0], sixlinks[isixtype][1], sixlinks[isixtype][2]
                );
            */

            for (INT iSeperation = 0; iSeperation < 4; ++iSeperation)
            {
                INT L[3] = { 0, 0, 0 };
                INT R[3] = { 0, 0, 0 };
                BYTE LLength = 0;
                BYTE RLength = 0;

                Seperate(sixlinks[isixtype], iSeperation, L, R, LLength, RLength);

                const UBOOL bHasLeft = (LLength > 0) && (L[0] > 0);
                const UBOOL bHasRight = (RLength > 0) && (R[0] > 0);

                if (bHasLeft || bHasRight)
                {
                    _kernelDFermionKSForce_PR_XYTau_TermT << <block, threads >> > (
                        (const deviceGauge*)pGaugeBuffer,
                        (deviceGauge*)pForce,
                        this->m_pRationalFieldPointers,
                        this->m_pMDNumerator,
                        this->m_rMD.m_uiDegree,
                        this->m_byFieldId,
                        CCommonData::m_fOmega,
                        L[0], L[1], L[2], LLength,
                        R[0], R[1], R[2], RLength
                        );
                }
            }
        }
    }
    
    #pragma endregion
#endif
}

#pragma endregion

template<typename deviceVector, typename deviceGauge, INT vectorN>
void CFieldFermionKSTR<deviceVector, deviceGauge, vectorN>::InitialOtherParameters(CParameters& params)
{
    CFieldFermionKST<deviceVector, deviceGauge, vectorN>::InitialOtherParameters(params);
    this->m_bEachSiteEta = TRUE;

    INT iReal = 0;
    if (params.FetchValueINT(_T("RealRotation"), iReal))
    {
        m_bRealRotation = (0 != iReal);
    }

    Real fValue = F(0.1);
    if (params.FetchValueReal(_T("Omega"), fValue))
    {
        m_fOmega = fValue;
    }
}

template<typename deviceVector, typename deviceGauge, INT vectorN>
void CFieldFermionKSTR<deviceVector, deviceGauge, vectorN>::CopyTo(CField* U) const
{
    CFieldFermionKST<deviceVector, deviceGauge, vectorN>::CopyTo(U);
    CFieldFermionKSTR<deviceVector, deviceGauge, vectorN>* pOther = dynamic_cast<CFieldFermionKSTR<deviceVector, deviceGauge, vectorN>*>(U);
    if (NULL != pOther)
    {
        pOther->m_bRealRotation = m_bRealRotation;
        pOther->m_fOmega = m_fOmega;
    }
}

template<typename deviceVector, typename deviceGauge, INT vectorN>
CCString CFieldFermionKSTR<deviceVector, deviceGauge, vectorN>::GetInfos(const CCString& tab) const
{
    CCString sRet = CFieldFermionKST<deviceVector, deviceGauge, vectorN>::GetInfos(tab);
    sRet = sRet + tab + _T("Omega : ") + appToString(m_fOmega) + _T("\n");
    sRet = sRet + tab + _T("RealRotation : ") + appToString(m_bRealRotation) + _T("\n");
    return sRet;
}

__CLG_FORCETEMPLATE_CONSTRUCTOR(CFieldFermionKSTR, U1, CLGComplex, CLGComplex, 1)

__CLGIMPLEMENT_CLASS(CFieldFermionKSU1R)

__END_NAMESPACE


//=============================================================================
// END OF FILE
//=============================================================================