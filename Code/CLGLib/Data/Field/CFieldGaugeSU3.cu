#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CFieldGaugeSU3.cu
// 
// DESCRIPTION:
// This is the device implementations of gauge SU3
//
// The SU3 Matrix is
// 0 1 2
// 3 4 5
// 6 7 8
//
// Number of threads: < 1024
// Number of blocks: V / 1024
//
// threadIdx.xyz = xyz, and we loop for t and dir
//
// REVISION:
//  [12/4/2018 nbale]
//=============================================================================

#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

__CLGIMPLEMENT_CLASS(CFieldGaugeSU3)

#pragma region Kernels

/**
* Initial SU3 Field with a value
*/
__global__
void _kernelInitialSU3Feield(deviceSU3 *pDevicePtr, EFieldInitialType eInitialType)
{
    deviceSU3 id = deviceSU3::makeSU3Id();
    deviceSU3 zero = deviceSU3::makeSU3Zero();

    gaugeSU3KernelFuncionStart

        switch (eInitialType)
        {
        case EFIT_Zero:
        {
            pDevicePtr[_deviceGetLinkIndex(coord, idir)] = zero;
        }
        break;
        case EFIT_Identity:
        {
            pDevicePtr[_deviceGetLinkIndex(coord, idir)] = id;
        }
        break;
        case EFIT_Random:
        {
            pDevicePtr[_deviceGetLinkIndex(coord, idir)] = deviceSU3::makeSU3Random(_deviceGetFatIndex(coord, idir + 1));
        }
        break;
        case EFIT_RandomGenerator:
        {
            pDevicePtr[_deviceGetLinkIndex(coord, idir)] = deviceSU3::makeSU3RandomGenerator(_deviceGetFatIndex(coord, idir + 1));
        }
        break;
        default:
        {
            printf("SU3 Field cannot be initialized with this type!");
        }
        break;
        }

    gaugeSU3KernelFuncionEnd
}

__global__
void _kernelAxpySU3A(deviceSU3 *pDevicePtr, const deviceSU3* __restrict__ x, _Complex a)
{
    gaugeSU3KernelFuncionStart

    UINT uiLinkIndex = _deviceGetLinkIndex(coord, idir);
    pDevicePtr[uiLinkIndex].Add(x[uiLinkIndex].Mulc(a));

    gaugeSU3KernelFuncionEnd
}

__global__
void _kernelAxpySU3Real(deviceSU3 *pDevicePtr, const deviceSU3* __restrict__ x, Real a)
{
    gaugeSU3KernelFuncionStart

        UINT uiLinkIndex = _deviceGetLinkIndex(coord, idir);
    pDevicePtr[uiLinkIndex].Add(x[uiLinkIndex].Mulc(a));

    gaugeSU3KernelFuncionEnd
}

__global__
void _kernelAxpyPlusSU3(deviceSU3 *pDevicePtr, const deviceSU3* __restrict__ x)
{
    gaugeSU3KernelFuncionStart

    UINT uiLinkIndex = _deviceGetLinkIndex(coord, idir);
    pDevicePtr[uiLinkIndex].Add(x[uiLinkIndex]);

    gaugeSU3KernelFuncionEnd
}

__global__
void _kernelAxpyMinusSU3(deviceSU3 *pDevicePtr, const deviceSU3* __restrict__ x)
{
    gaugeSU3KernelFuncionStart

    UINT uiLinkIndex = _deviceGetLinkIndex(coord, idir);
    pDevicePtr[uiLinkIndex].Sub(x[uiLinkIndex]);

    gaugeSU3KernelFuncionEnd
}

__global__
void _kernelScalarMultiplySU3Complex(deviceSU3 *pDevicePtr, _Complex a)
{
    gaugeSU3KernelFuncionStart

    UINT uiLinkIndex = _deviceGetLinkIndex(coord, idir);
    pDevicePtr[uiLinkIndex].Mul(a);

    gaugeSU3KernelFuncionEnd
}

__global__
void _kernelScalarMultiplySU3Real(deviceSU3 *pDevicePtr, Real a)
{
    gaugeSU3KernelFuncionStart

    UINT uiLinkIndex = _deviceGetLinkIndex(coord, idir);
    pDevicePtr[uiLinkIndex].Mul(a);

    gaugeSU3KernelFuncionEnd
}

/**
* debug kernel
*/
__global__ void _kernelPrintSU3(const deviceSU3 * __restrict__ pDeviceData)
{
    intokernal;

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        for (UINT idir = 0; idir < uiDir; ++idir)
        {
            UINT linkIndex = _deviceGetLinkIndex(coord, idir);
            printf("link at %d: %f+%f i, %f+%f i, %f+%f i, %f+%f i, %f+%f i, %f+%f i, %f+%f i, %f+%f i, %f+%f i\n",
                linkIndex,
                pDeviceData[linkIndex].m_me[0].x, pDeviceData[linkIndex].m_me[0].y,
                pDeviceData[linkIndex].m_me[1].x, pDeviceData[linkIndex].m_me[1].y,
                pDeviceData[linkIndex].m_me[2].x, pDeviceData[linkIndex].m_me[2].y,
                pDeviceData[linkIndex].m_me[3].x, pDeviceData[linkIndex].m_me[3].y,
                pDeviceData[linkIndex].m_me[4].x, pDeviceData[linkIndex].m_me[4].y,
                pDeviceData[linkIndex].m_me[5].x, pDeviceData[linkIndex].m_me[5].y,
                pDeviceData[linkIndex].m_me[6].x, pDeviceData[linkIndex].m_me[6].y,
                pDeviceData[linkIndex].m_me[7].x, pDeviceData[linkIndex].m_me[7].y,
                pDeviceData[linkIndex].m_me[8].x, pDeviceData[linkIndex].m_me[8].y
            );
        }
    }
}

/**
* calculate Staple and Force At Site
*/
__global__
void _kernelStapleAtSiteSU3(
    const deviceSU3 * __restrict__ pDeviceData,
    deviceSU3 *pStapleData, //can be NULL
    deviceSU3 *pForceData,
    _Complex minusBetaOverN)
{
    intokernal;

    minusBetaOverN.x = minusBetaOverN.x * (Real)0.5;
    SIndex plaquttes[kMaxPlaqutteCache];
    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        for (UINT idir = 0; idir < uiDir; ++idir)
        {
            UINT linkIndex = _deviceGetLinkIndex(coord, idir);
            UINT uiPlaqutteCount = 0;
            UINT uiPlaqutteLength = 0;

            //int2.x is linkIndex
            //int2.y is fieldIndex (may on bounday)
            //sign of int2.y is whether inverse
            __idx->_deviceGetPlaquttesAtLink(plaquttes, uiPlaqutteCount, uiPlaqutteLength, linkIndex);

            deviceSU3 res = deviceSU3::makeSU3Zero();

            for (int i = 0; i < uiPlaqutteCount; ++i)
            {
                SIndex first = plaquttes[uiPlaqutteCount * (uiPlaqutteLength - 1)];
                deviceSU3 toAdd(pDeviceData[_deviceGetLinkIndex(first.m_uiSiteIndex, first.m_byDir)]);
                if (first.NeedToDagger())
                {
                    toAdd.Dagger();
                }

                for (int j = 1; j < uiPlaqutteLength - 1; ++j)
                {
                    SIndex nextlink = plaquttes[uiPlaqutteCount * (uiPlaqutteLength - 1) + j];
                    deviceSU3 toMul(pDeviceData[_deviceGetLinkIndex(nextlink.m_uiSiteIndex, nextlink.m_byDir)]);
                    if (nextlink.NeedToDagger())
                    {
                        toMul.Dagger();
                    }
                    toAdd.Mul(toMul);
                }
                res.Add(toAdd);
            }
            if (NULL != pStapleData)
            {
                pStapleData[linkIndex] = res;
            }

            res.Dagger();
            //staple calculated
            deviceSU3 force(pDeviceData[linkIndex]);
            force.Mul(res);
            force.TrTa();
            force.Mul(minusBetaOverN);

            //force is additive
            pForceData[linkIndex].Add(force);
        }
    }
}

/**
* calculate Staple and eneregy At Site
*/
__global__
void _kernelPlaqutteEnergySU3(
    const deviceSU3 * __restrict__ pDeviceData,
    Real minusBetaOverN,
    Real* results)
{
    intokernal;

    Real resThisThread = 0;
    SIndex plaquttes[kMaxPlaqutteCache];
    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        for (UINT idir = 0; idir < uiDir; ++idir)
        {
            UINT linkIndex = _deviceGetLinkIndex(coord, idir);
            UINT uiPlaqutteCount = 0;
            UINT uiPlaqutteLength = 0;

            //int2.x is linkIndex
            //int2.y is fieldIndex (may on bounday)
            //sign of int2.y is whether inverse
            __idx->_deviceGetPlaquttesAtLink(plaquttes, uiPlaqutteCount, uiPlaqutteLength, linkIndex);

            deviceSU3 res = deviceSU3::makeSU3Zero();

            for (int i = 0; i < uiPlaqutteCount; ++i)
            {
                SIndex first = plaquttes[uiPlaqutteCount * (uiPlaqutteLength - 1)];

                deviceSU3 toAdd(pDeviceData[_deviceGetLinkIndex(first.m_uiSiteIndex, first.m_byDir)]);
                if (first.NeedToDagger())
                {
                    toAdd.Dagger();
                }

                for (int j = 1; j < uiPlaqutteLength - 1; ++j)
                {
                    SIndex nextlink = plaquttes[uiPlaqutteCount * (uiPlaqutteLength - 1) + j];
                    deviceSU3 toMul(pDeviceData[_deviceGetLinkIndex(nextlink.m_uiSiteIndex, nextlink.m_byDir)]);
                    if (nextlink.NeedToDagger())
                    {
                        toMul.Dagger();
                    }
                    toAdd.Mul(toMul);
                }
                res.Add(toAdd);
            }

            res.Dagger();
            deviceSU3 gaugeOnLink(pDeviceData[linkIndex]);
            gaugeOnLink.Mul(res);
            resThisThread += (3 - gaugeOnLink.ReTr()); //Re[Tr(1-U)] = 3 - Re[Tr(U)]
        }
    }

    results[threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z] = resThisThread * minusBetaOverN;
}

/**
*
*/
__global__
void _kernelExpMultSU3(
    const deviceSU3 * __restrict__ pMyDeviceData,
    _Complex a,
    deviceSU3 *pU)
{
    intokernal;

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        for (UINT idir = 0; idir < uiDir; ++idir)
        {
            UINT linkIndex = _deviceGetLinkIndex(coord, idir);

            deviceSU3 expP = pMyDeviceData[linkIndex].Exp(a, _DC_ExpPrecision);
            expP.Mul(pU[linkIndex]);
            expP.Norm();
            pU[linkIndex] = expP;
        }
    }
}


/**
* Trace (P^2)
*/
__global__ 
void _kernelCalculateKinematicEnergySU3(const deviceSU3 * __restrict__ pMyDeviceData, Real* results)
{
    intokernal;

    Real resThisThread = 0;
    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        for (UINT idir = 0; idir < uiDir; ++idir)
        {
            UINT linkIndex = _deviceGetLinkIndex(coord, idir);
            resThisThread += pMyDeviceData[linkIndex].Daggerc().Mulc(pMyDeviceData[linkIndex]).ReTr();
        }
    }

    results[threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z] = resThisThread;
}

#pragma endregion

#pragma region CFunctions

extern "C" {

    void _callKernelAxpySU3(deviceSU3 *pDevicePtr, const deviceSU3* __restrict__ x, UBOOL bAdd)
    {
        preparethread;
        if (bAdd)
        {
            _kernelAxpyPlusSU3 << <block, threads >> > (pDevicePtr, x);
        }
        else 
        {
            _kernelAxpyMinusSU3 << <block, threads >> > (pDevicePtr, x);
        }
    }

    void _callKernelAxpySU3A(deviceSU3 *pDevicePtr, const deviceSU3* __restrict__ x, const _Complex& a)
    {
        preparethread;
        _kernelAxpySU3A << <block, threads >> > (pDevicePtr, x, a);
    }

    void _callKernelAxpySU3Real(deviceSU3 *pDevicePtr, const deviceSU3* __restrict__ x, Real a)
    {
        preparethread;
        _kernelAxpySU3Real << <block, threads >> > (pDevicePtr, x, a);
    }

    void _callKernelScalarMultiplyComplex(deviceSU3 *pDevicePtr, const _Complex& a)
    {
        preparethread;
        _kernelScalarMultiplySU3Complex << <block, threads >> > (pDevicePtr, a);
    }

    void _callKernelScalarMultiplyReal(deviceSU3 *pDevicePtr, Real a)
    {
        preparethread;
        _kernelScalarMultiplySU3Real << <block, threads >> > (pDevicePtr, a);
    }

    void _callKernelPrint(const deviceSU3 * __restrict__ pDeviceData)
    {
        preparethread;
        _kernelPrintSU3 << < block, threads >> > (pDeviceData);
    }

    void _callKernelExpMultSU3(
        const deviceSU3 * __restrict__ pMyDeviceData,
        const _Complex& a,
        deviceSU3 *pU)
    {
        preparethread;
        _kernelExpMultSU3 << < block, threads >> > (pMyDeviceData, a, pU);
    }

    void _callKernelInitialSU3Feield(deviceSU3 *pDevicePtr, EFieldInitialType eInitialType)
    {
        preparethread;
        _kernelInitialSU3Feield << <block, threads >> > (pDevicePtr, eInitialType);
    }

    void _callKernelStapleAtSiteSU3(
        const deviceSU3 * __restrict__ pDeviceData,
        deviceSU3 *pStapleData, //can be NULL
        deviceSU3 *pForceData,
        const _Complex& minusBetaOverN)
    {
        preparethread;
        _kernelStapleAtSiteSU3 << <block, threads >> > (pDeviceData, pStapleData, pForceData, minusBetaOverN);
    }

    Real _callKernelPlaqutteEneregySU3(
        const deviceSU3 * __restrict__ pDeviceData,
        const _Complex& minusBetaOverN,
        UINT uiDataSize,
        Real* output
    )
    {
        preparethread;
        _kernelPlaqutteEnergySU3 << <block, threads >> > (pDeviceData, -minusBetaOverN.x, output);

        //Now the results are in output.
        thrust::device_ptr<Real> dp(output);
        thrust::device_vector<Real> d_x(dp, dp + uiDataSize);
        return thrust::reduce(d_x.begin(), d_x.end(), (Real)0, thrust::plus<Real>());
    }

    Real _callKernelKinematicEnergySU3(const deviceSU3 * __restrict__ pDeviceData, Real* output, UINT uiDataSize)
    {
        preparethread;
        _kernelCalculateKinematicEnergySU3 << <block, threads >> > (pDeviceData, output);
        thrust::device_ptr<Real> dp(output);
        thrust::device_vector<Real> d_x(dp, dp + uiDataSize);
        return thrust::reduce(d_x.begin(), d_x.end(), (Real)0, thrust::plus<Real>());
    }
}

#pragma endregion

void CFieldGaugeSU3::AxpyPlus(const CField* x)
{
    if (NULL == x || EFT_GaugeSU3 != x->GetFieldType())
    {
        appCrucial("CFieldGaugeSU3: axpy failed because the otherfield is not SU3");
        return;
    }

    const CFieldGaugeSU3* pSU3x = dynamic_cast<const CFieldGaugeSU3*>(x);
    _callKernelAxpySU3(m_pDeviceData, pSU3x->m_pDeviceData, TRUE);

}

void CFieldGaugeSU3::AxpyMinus(const CField* x)
{
    if (NULL == x || EFT_GaugeSU3 != x->GetFieldType())
    {
        appCrucial("CFieldGaugeSU3: axpy failed because the otherfield is not SU3");
        return;
    }

    const CFieldGaugeSU3* pSU3x = dynamic_cast<const CFieldGaugeSU3*>(x);
    _callKernelAxpySU3(m_pDeviceData, pSU3x->m_pDeviceData, FALSE);

}

void CFieldGaugeSU3::ScalarMultply(const _Complex& a)
{
    _callKernelScalarMultiplyComplex(m_pDeviceData, a);
}

void CFieldGaugeSU3::ScalarMultply(Real a)
{
    _callKernelScalarMultiplyReal(m_pDeviceData, a);
}

void CFieldGaugeSU3::Axpy(Real a, const CField* x)
{
    if (NULL == x || EFT_GaugeSU3 != x->GetFieldType())
    {
        appCrucial("CFieldGaugeSU3: axpy failed because the otherfield is not SU3");
        return;
    }

    const CFieldGaugeSU3* pSU3x = dynamic_cast<const CFieldGaugeSU3*>(x);
    _callKernelAxpySU3Real(m_pDeviceData, pSU3x->m_pDeviceData, a);
}

void CFieldGaugeSU3::Axpy(const _Complex& a, const CField* x)
{
    if (NULL == x || EFT_GaugeSU3 != x->GetFieldType())
    {
        appCrucial("CFieldGaugeSU3: axpy failed because the otherfield is not SU3");
        return;
    }

    const CFieldGaugeSU3* pSU3x = dynamic_cast<const CFieldGaugeSU3*>(x);
    _callKernelAxpySU3A(m_pDeviceData, pSU3x->m_pDeviceData, a);
}


void CFieldGaugeSU3::Zero()
{
    _callKernelInitialSU3Feield(m_pDeviceData, EFIT_Zero);
}

void CFieldGaugeSU3::Indentity()
{
    _callKernelInitialSU3Feield(m_pDeviceData, EFIT_Identity);
}

void CFieldGaugeSU3::MakeRandomGenerator()
{
    _callKernelInitialSU3Feield(m_pDeviceData, EFIT_RandomGenerator);
}

/**
*
*/
void CFieldGaugeSU3::InitialField(EFieldInitialType eInitialType)
{
    _callKernelInitialSU3Feield(m_pDeviceData, eInitialType);
}

/**
* (1) calculate staples
* (2) calculate force(additive)
* (3) calculate energy
*/
void CFieldGaugeSU3::CalculateForceAndStaple(CFieldGauge* pForce, CFieldGauge* pStable, const _Complex& minusBetaOverN) const
{
    if (NULL == pForce || EFT_GaugeSU3 != pForce->GetFieldType())
    {
        appCrucial("CFieldGaugeSU3: force field is not SU3");
        return;
    }
    if (NULL != pStable && EFT_GaugeSU3 != pStable->GetFieldType())
    {
        appCrucial("CFieldGaugeSU3: stape field is not SU3");
        return;
    }

    CFieldGaugeSU3* pForceSU3 = dynamic_cast<CFieldGaugeSU3*>(pForce);
    CFieldGaugeSU3* pStableSU3 = NULL == pStable ? NULL : dynamic_cast<CFieldGaugeSU3*>(pStable);

    _callKernelStapleAtSiteSU3(m_pDeviceData, NULL == pStableSU3 ? NULL : pStableSU3->m_pDeviceData, pForceSU3->m_pDeviceData, minusBetaOverN);
}

Real CFieldGaugeSU3::CalculatePlaqutteEnergy(const _Complex& minusBetaOverN) const
{
    return _callKernelPlaqutteEneregySU3(m_pDeviceData, minusBetaOverN, _HC_ThreadCount, _D_RealThreadBuffer) / m_uiLinkeCount;
}

Real CFieldGaugeSU3::CalculateKinematicEnergy() const
{
    return _callKernelKinematicEnergySU3(m_pDeviceData, _D_RealThreadBuffer, _HC_ThreadCount);
}

CFieldGaugeSU3::CFieldGaugeSU3()
{
    checkCudaErrors(hipMalloc((void **)&m_pDeviceData, sizeof(deviceSU3) * m_uiLinkeCount));
}

CFieldGaugeSU3::~CFieldGaugeSU3()
{
    checkCudaErrors(hipFree(m_pDeviceData));
}

void CFieldGaugeSU3::ExpMult(const _Complex& a, CField* U) const
{
    if (NULL == U || EFT_GaugeSU3 != U->GetFieldType())
    {
        appCrucial("CFieldGaugeSU3: U field is not SU3");
        return;
    }

    CFieldGaugeSU3* pUField = dynamic_cast<CFieldGaugeSU3*>(U);

    preparethread;
    _callKernelExpMultSU3(m_pDeviceData, a, pUField->m_pDeviceData);
}


void CFieldGaugeSU3::CopyTo(CField* pTarget) const
{
    if (NULL == pTarget || EFT_GaugeSU3 != pTarget->GetFieldType())
    {
        appCrucial("CFieldGaugeSU3: target field is not SU3");
        return;
    }
    CFieldGaugeSU3* pTargetField = dynamic_cast<CFieldGaugeSU3*>(pTarget);
    checkCudaErrors(hipMemcpy(pTargetField->m_pDeviceData, m_pDeviceData, sizeof(deviceSU3) * m_uiLinkeCount, hipMemcpyDeviceToDevice));
}

void CFieldGaugeSU3::DebugPrintMe() const
{
    _callKernelPrint(m_pDeviceData);
}

__END_NAMESPACE


//=============================================================================
// END OF FILE
//=============================================================================