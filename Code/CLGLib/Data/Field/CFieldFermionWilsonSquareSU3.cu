#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CFieldFermionWilsonSquareSU3.cu
// 
// DESCRIPTION:
// This is the device implementations of Wilson fermion
//
// This implementation assumes SU3 and square lattice
//
// REVISION:
//  [12/27/2018 nbale]
//=============================================================================

#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

__CLGIMPLEMENT_CLASS(CFieldFermionWilsonSquareSU3)

#pragma region Kernel

__global__ void _kernelPrintFermionWilsonSquareSU3(const deviceWilsonVectorSU3 * __restrict__ pData)
{
    intokernal;

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        UINT siteIndexX = _deviceGetSiteIndex(coord);

        printf("%d,%d,%d,%d=((%1.2f %1.2fi, %1.2f %1.2fi, %1.2f %1.2fi),(%1.2f %1.2fi, %1.2f %1.2fi, %1.2f %1.2fi),(%1.2f %1.2fi, %1.2f %1.2fi, %1.2f %1.2fi),(%1.2f %1.2fi, %1.2f %1.2fi, %1.2f %1.2fi))\n", 
            coord[0], coord[1], coord[2], coord[3],
            pData[siteIndexX].m_d[0].m_ve[0].x, pData[siteIndexX].m_d[0].m_ve[0].y,
            pData[siteIndexX].m_d[0].m_ve[1].x, pData[siteIndexX].m_d[0].m_ve[1].y,
            pData[siteIndexX].m_d[0].m_ve[2].x, pData[siteIndexX].m_d[0].m_ve[2].y,

            pData[siteIndexX].m_d[1].m_ve[0].x, pData[siteIndexX].m_d[1].m_ve[0].y,
            pData[siteIndexX].m_d[1].m_ve[1].x, pData[siteIndexX].m_d[1].m_ve[1].y,
            pData[siteIndexX].m_d[1].m_ve[2].x, pData[siteIndexX].m_d[1].m_ve[2].y,

            pData[siteIndexX].m_d[2].m_ve[0].x, pData[siteIndexX].m_d[2].m_ve[0].y,
            pData[siteIndexX].m_d[2].m_ve[1].x, pData[siteIndexX].m_d[2].m_ve[1].y,
            pData[siteIndexX].m_d[2].m_ve[2].x, pData[siteIndexX].m_d[2].m_ve[2].y,

            pData[siteIndexX].m_d[3].m_ve[0].x, pData[siteIndexX].m_d[3].m_ve[0].y,
            pData[siteIndexX].m_d[3].m_ve[1].x, pData[siteIndexX].m_d[3].m_ve[1].y,
            pData[siteIndexX].m_d[3].m_ve[2].x, pData[siteIndexX].m_d[3].m_ve[2].y
            );
    }
}

__global__ void _kernelAxpyPlusFermionWilsonSquareSU3(
    deviceWilsonVectorSU3 * pMe, 
    const deviceWilsonVectorSU3 * __restrict__ pOther)
{
    intokernal;

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        UINT siteIndex = _deviceGetSiteIndex(coord);
        pMe[siteIndex].Add(pOther[siteIndex]);
    }
}

__global__ void _kernelAxpyMinusFermionWilsonSquareSU3(
    deviceWilsonVectorSU3 * pMe, 
    const deviceWilsonVectorSU3 * __restrict__ pOther)
{
    intokernal;

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        UINT siteIndex = _deviceGetSiteIndex(coord);
        pMe[siteIndex].Sub(pOther[siteIndex]);
    }
}

__global__ void _kernelAxpyComplexFermionWilsonSquareSU3(
    deviceWilsonVectorSU3 * pMe, 
    const deviceWilsonVectorSU3 * __restrict__ pOther, _Complex a)
{
    intokernal;

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        UINT siteIndex = _deviceGetSiteIndex(coord);
        pMe[siteIndex].Add(pOther[siteIndex].MulCompC(a));
    }
}

__global__ void _kernelAxpyRealFermionWilsonSquareSU3(
    deviceWilsonVectorSU3 * pMe, 
    const deviceWilsonVectorSU3 * __restrict__ pOther, Real a)
{
    intokernal;

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        UINT siteIndex = _deviceGetSiteIndex(coord);
        pMe[siteIndex].Add(pOther[siteIndex].MulRealC(a));
    }
}

__global__ void _kernelDotFermionWilsonSquareSU3(
    const deviceWilsonVectorSU3 * __restrict__ pMe, 
    const deviceWilsonVectorSU3 * __restrict__ pOther, 
    _Complex * result)
{
    intokernal;
    _Complex res = _make_cuComplex(F(0.0), F(0.0));
    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        UINT siteIndex = _deviceGetSiteIndex(coord);
        res = _cuCaddf(res, pMe[siteIndex].ConjugateDotC(pOther[siteIndex]));
    }
    result[__thread_id] = res;
}

__global__ void _kernelScalarMultiplyComplex(
    deviceWilsonVectorSU3 * pMe, 
    _Complex a)
{
    intokernal;
    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        UINT siteIndex = _deviceGetSiteIndex(coord);
        pMe[siteIndex].MulComp(a);
    }
}

__global__ void _kernelScalarMultiplyReal(
    deviceWilsonVectorSU3 * pMe, 
    Real a)
{
    intokernal;
    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        UINT siteIndex = _deviceGetSiteIndex(coord);
        pMe[siteIndex].MulReal(a);
    }
}

/**
*
*/
__global__ void _kernelInitialFermionWilsonSquareSU3(
    deviceWilsonVectorSU3 *pDevicePtr, 
    EFieldInitialType eInitialType)
{
    intokernal;

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        UINT siteIndexX = _deviceGetSiteIndex(coord);
        UINT fatIndex = _deviceGetFatIndex(siteIndexX, 0);

        switch (eInitialType)
        {
            case EFIT_Zero:
                {
                    pDevicePtr[siteIndexX] = deviceWilsonVectorSU3::makeZeroWilsonVectorSU3();
                }
                break;
            case EFIT_RandomGaussian:
                {
                    pDevicePtr[siteIndexX] = deviceWilsonVectorSU3::makeRandomGaussian(fatIndex);
                }
                break;
            default:
                {
                    printf("Wilson Fermion Field cannot be initialized with this type!");
                }
            break;
        }
    }
}

/**
* Dw phi(x) = phi(x) - kai sum _mu (1-gamma _mu) U(x,mu) phi(x+ mu) + (1+gamma _mu) U^{dagger}(x-mu) phi(x-mu)
* U act on su3
* gamma act on spinor
*
* If bDagger, it is gamma5, D, gamma5
*
*/
__global__ void _kernelDFermionWilsonSquareSU3(
    const deviceWilsonVectorSU3* __restrict__ pDeviceData,
    const deviceSU3* __restrict__ pGauge,
    deviceWilsonVectorSU3* pResultData,
    Real kai,
    BYTE byFieldId,
    UBOOL bDiracChiralGamma,
    UBOOL bDDagger)
{
    intokernaldir;

    gammaMatrix gamma5 = bDiracChiralGamma ? __diracGamma->m_gm[GAMMA5] : __chiralGamma->m_gm[GAMMA5];

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        //x
        UINT siteIndexX = _deviceGetSiteIndex(coord);
        deviceWilsonVectorSU3 result = deviceWilsonVectorSU3::makeZeroWilsonVectorSU3();
        pResultData[siteIndexX] = pDeviceData[siteIndexX];
        if (bDDagger)
        {
            pResultData[siteIndexX] = gamma5.MulWilsonC(pResultData[siteIndexX]);
        }

        //idir = mu
        for (UINT idir = 0; idir < uiDir; ++idir)
        {
            //Get Gamma mu
            gammaMatrix gammaMu = bDiracChiralGamma ? 
                  __diracGamma->m_gm[GAMMA1 + idir]
                : __chiralGamma->m_gm[GAMMA1 + idir];

            //x, mu
            UINT linkIndex = _deviceGetLinkIndex(siteIndexX, idir);

            SIndex x_m_mu_Gauge = __idx->_deviceGaugeIndexWalk(siteIndexX, -(idir + 1));
            SIndex x_p_mu_Fermion = __idx->_deviceFermionIndexWalk(byFieldId, siteIndexX, (idir + 1));
            SIndex x_m_mu_Fermion = __idx->_deviceFermionIndexWalk(byFieldId, siteIndexX, -(idir + 1));          

            //Assuming periodic
            //get U(x,mu), U^{dagger}(x-mu), 
            deviceSU3 x_Gauge_element = pGauge[linkIndex];
            deviceSU3 x_m_mu_Gauge_element = pGauge[_deviceGetLinkIndex(x_m_mu_Gauge.m_uiSiteIndex, idir)];
            x_m_mu_Gauge_element.Dagger();

            deviceWilsonVectorSU3 x_p_mu_Fermion_element = pDeviceData[x_p_mu_Fermion.m_uiSiteIndex];
            deviceWilsonVectorSU3 x_m_mu_Fermion_element = pDeviceData[x_m_mu_Fermion.m_uiSiteIndex];

            if (bDDagger)
            {
                x_p_mu_Fermion_element = gamma5.MulWilsonC(x_p_mu_Fermion_element);
                x_m_mu_Fermion_element = gamma5.MulWilsonC(x_m_mu_Fermion_element);
            }

            //hopping terms
            
            //U(x,mu) phi(x+ mu)
            deviceWilsonVectorSU3 u_phi_x_p_m = x_Gauge_element.MulWilsonVector(x_p_mu_Fermion_element);
            result.Add(u_phi_x_p_m);

            //- gammamu U(x,mu) phi(x+ mu)
            result.Sub(gammaMu.MulWilsonC(u_phi_x_p_m));

            //U^{dagger}(x-mu) phi(x-mu)
            deviceWilsonVectorSU3 u_dagger_phi_x_m_m = x_m_mu_Gauge_element.MulWilsonVector(x_m_mu_Fermion_element);
            result.Add(u_dagger_phi_x_m_m);

            //gammamu U^{dagger}(x-mu) phi(x-mu)
            result.Add(gammaMu.MulWilsonC(u_dagger_phi_x_m_m));
        }

        //result = phi(x) - kai sum _mu result
        result.MulReal(kai);
        pResultData[siteIndexX].Sub(result);
        if (bDDagger)
        {
            pResultData[siteIndexX] = gamma5.MulWilsonC(pResultData[siteIndexX]);
        }
    }
}

/**
* The output is on a gauge field
* Therefor cannot make together with _kernelDWilson
*
*/
__global__ void _kernelDWilsonForceSU3(
    const deviceWilsonVectorSU3* __restrict__ pInverseD,
    const deviceWilsonVectorSU3* __restrict__ pInverseDDdagger,
    const deviceSU3* __restrict__ pGauge,
    deviceSU3* pForce,
    Real fKai,
    BYTE byFieldId,
    UBOOL bDiracChiralGamma)
{
    intokernaldir;

    //gammaMatrix gamma5 = bDiracChiralGamma ? __diracGamma->m_gm[GAMMA5] : __chiralGamma->m_gm[GAMMA5];
    _Complex cKai = _make_cuComplex(F(0.0), -fKai);

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        //x
        UINT siteIndexX = _deviceGetSiteIndex(coord);
        deviceWilsonVectorSU3 x_Left(pInverseDDdagger[siteIndexX]);
        deviceWilsonVectorSU3 x_Right(pInverseD[siteIndexX]);

        //idir = mu
        for (UINT idir = 0; idir < uiDir; ++idir)
        {
            //Get Gamma mu
            gammaMatrix gammaMu = bDiracChiralGamma ?
                  __diracGamma->m_gm[GAMMA1 + idir]
                : __chiralGamma->m_gm[GAMMA1 + idir];

            //x, mu
            UINT linkIndex = _deviceGetLinkIndex(coord, idir);

            SIndex x_m_mu_Gauge = __idx->_deviceGaugeIndexWalk(siteIndexX, -(idir + 1));
            SIndex x_p_mu_Fermion = __idx->_deviceFermionIndexWalk(byFieldId, siteIndexX, (idir + 1));
            SIndex x_m_mu_Fermion = __idx->_deviceFermionIndexWalk(byFieldId, siteIndexX, -(idir + 1));

            deviceWilsonVectorSU3 x_p_mu_Right(pInverseD[x_p_mu_Fermion.m_uiSiteIndex]);
            deviceWilsonVectorSU3 x_m_mu_Left(pInverseDDdagger[x_m_mu_Fermion.m_uiSiteIndex]);

            deviceSU3 x_Gauge_element = pGauge[linkIndex];

            for (UINT i = 0; i < 8; ++i)
            {
                //get Ti U(x,mu) phi(x+mu)
                deviceWilsonVectorSU3 Ti_U_x_mu_phi = __SU3Generators[i].MulC(x_Gauge_element).MulWilsonVector(x_p_mu_Right);
                //get U^{dagger}(x) Ti phi(x), 
                deviceWilsonVectorSU3 Udagger_x_m_mu_Ti_phi = x_Gauge_element.DaggerMulC(__SU3Generators[i]).MulWilsonVector(x_Right);

                //hopping terms
                //(1-gamma _mu) Ti U(x,mu) phi(x+ mu) - (1+gamma _mu) U^{dagger}(x) Ti phi(x)

                //(1 - gamma_mu) Ti U(x,mu) phi(x+ mu)
                deviceSU3 res = deviceSU3::makeSU3Contract(x_Left, Ti_U_x_mu_phi.SubC(gammaMu.MulWilsonC(Ti_U_x_mu_phi)));

                //- (1 + gamma _mu)U^{dagger}(x) Ti phi(x)
                res.Sub(deviceSU3::makeSU3Contract(x_m_mu_Left, Udagger_x_m_mu_Ti_phi.AddC(gammaMu.MulWilsonC(Udagger_x_m_mu_Ti_phi))));

                pForce[linkIndex].Add(res.Im2C().MulC(__SU3Generators[i]).MulCompC(cKai));
            }
        }
    }
}

__global__ void _kernelApplyGammaSU3(deviceWilsonVectorSU3* pDeviceData, UINT uiGamma, UBOOL bDiracChiralGamma)
{
    intokernal;

    gammaMatrix theMatrix = bDiracChiralGamma ? __diracGamma->m_gm[uiGamma] : __chiralGamma->m_gm[uiGamma];

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        UINT siteIndexX = _deviceGetSiteIndex(coord);
        pDeviceData[siteIndexX] = theMatrix.MulWilsonC(pDeviceData[siteIndexX]);
    }
}

#pragma endregion

CFieldFermionWilsonSquareSU3::CFieldFermionWilsonSquareSU3() : CFieldFermion(), m_fKai(F(0.125))
{
    checkCudaErrors(hipMalloc((void**)&m_pDeviceData, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount));
    checkCudaErrors(hipMalloc((void**)&m_pDeviceDataCopy, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount));
}

CFieldFermionWilsonSquareSU3::~CFieldFermionWilsonSquareSU3()
{
    checkCudaErrors(hipFree(m_pDeviceData));
    checkCudaErrors(hipFree(m_pDeviceDataCopy));
}

/**
*
*/
void CFieldFermionWilsonSquareSU3::InitialField(EFieldInitialType eInitialType)
{
    preparethread;
    _kernelInitialFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData, eInitialType);
}

void CFieldFermionWilsonSquareSU3::InitialOtherParameters(CParameters& params)
{
    params.FetchValueReal(_T("Hopping"), m_fKai);
    if (m_fKai < F(0.00000001))
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3: Kai is nearly 0, such that Dphi \approx phi! This will cause problem!\n"));
    }

    if (m_fKai > F(0.12500001))
    {
        appGeneral(_T("CFieldFermionWilsonSquareSU3: Kai = 1/sqrt{2am+8}, note: this kai>1/8\n"));
    }
}

void CFieldFermionWilsonSquareSU3::DebugPrintMe() const
{
    preparethread;
    _kernelPrintFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData);
}

void CFieldFermionWilsonSquareSU3::CopyTo(CField* U) const
{
    if (NULL == U || EFT_FermionWilsonSquareSU3 != U->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only copy to CFieldFermionWilsonSquareSU3!"));
        return;
    }

    CField::CopyTo(U);

    CFieldFermionWilsonSquareSU3 * pField = dynamic_cast<CFieldFermionWilsonSquareSU3*>(U);
    checkCudaErrors(hipMemcpy(pField->m_pDeviceData, m_pDeviceData, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount, hipMemcpyDeviceToDevice));
    pField->m_byFieldId = m_byFieldId;
    pField->m_fKai = m_fKai;
}

void CFieldFermionWilsonSquareSU3::AxpyPlus(const CField* x)
{
    if (NULL == x || EFT_FermionWilsonSquareSU3 != x->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only copy to CFieldFermionWilsonSquareSU3!"));
        return;
    }
    const CFieldFermionWilsonSquareSU3 * pField = dynamic_cast<const CFieldFermionWilsonSquareSU3*>(x);

    preparethread;
    _kernelAxpyPlusFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData, pField->m_pDeviceData);
}

void CFieldFermionWilsonSquareSU3::AxpyMinus(const CField* x)
{
    if (NULL == x || EFT_FermionWilsonSquareSU3 != x->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only copy to CFieldFermionWilsonSquareSU3!"));
        return;
    }
    const CFieldFermionWilsonSquareSU3 * pField = dynamic_cast<const CFieldFermionWilsonSquareSU3*>(x);

    preparethread;
    _kernelAxpyMinusFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData, pField->m_pDeviceData);
}

void CFieldFermionWilsonSquareSU3::Axpy(Real a, const CField* x)
{
    if (NULL == x || EFT_FermionWilsonSquareSU3 != x->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only copy to CFieldFermionWilsonSquareSU3!"));
        return;
    }
    const CFieldFermionWilsonSquareSU3 * pField = dynamic_cast<const CFieldFermionWilsonSquareSU3*>(x);

    preparethread;
    _kernelAxpyRealFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData, pField->m_pDeviceData, a);
}

void CFieldFermionWilsonSquareSU3::Axpy(const _Complex& a, const CField* x)
{
    if (NULL == x || EFT_FermionWilsonSquareSU3 != x->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only copy to CFieldFermionWilsonSquareSU3!"));
        return;
    }
    const CFieldFermionWilsonSquareSU3 * pField = dynamic_cast<const CFieldFermionWilsonSquareSU3*>(x);

    preparethread;
    _kernelAxpyComplexFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData, pField->m_pDeviceData, a);
}

_Complex CFieldFermionWilsonSquareSU3::Dot(const CField* x) const
{
    if (NULL == x || EFT_FermionWilsonSquareSU3 != x->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only copy to CFieldFermionWilsonSquareSU3!"));
        return _make_cuComplex(0,0);
    }
    const CFieldFermionWilsonSquareSU3 * pField = dynamic_cast<const CFieldFermionWilsonSquareSU3*>(x);

    preparethread;
    _kernelDotFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData, pField->m_pDeviceData, _D_ComplexThreadBuffer);

    return appGetCudaHelper()->ThreadBufferSum(_D_ComplexThreadBuffer);
}

void CFieldFermionWilsonSquareSU3::ScalarMultply(const _Complex& a)
{
    preparethread;
    _kernelScalarMultiplyComplex << <block, threads >> >(m_pDeviceData, a);
}

void CFieldFermionWilsonSquareSU3::ScalarMultply(Real a)
{
    preparethread;
    _kernelScalarMultiplyReal << <block, threads >> >(m_pDeviceData, a);
}

void CFieldFermionWilsonSquareSU3::ApplyGamma(EGammaMatrix eGamma)
{
    preparethread;
    _kernelApplyGammaSU3 << <block, threads >> >(m_pDeviceData, static_cast<UINT>(eGamma), TRUE);
}
/**
* generate phi by gaussian random.
* phi = D phi
*/
void CFieldFermionWilsonSquareSU3::PrepareForHMC(const CFieldGauge* pGauge)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    preparethread;
    _kernelInitialFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceDataCopy, EFIT_RandomGaussian);
    _kernelDFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceDataCopy, pFieldSU3->m_pDeviceData, m_pDeviceData, m_fKai, m_byFieldId, TRUE, FALSE);

}

//Kai should be part of D operator
void CFieldFermionWilsonSquareSU3::D(const CField* pGauge)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    checkCudaErrors(hipMemcpy(m_pDeviceDataCopy, m_pDeviceData, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount, hipMemcpyDeviceToDevice));

    preparethread;
    _kernelDFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceDataCopy, pFieldSU3->m_pDeviceData, m_pDeviceData, m_fKai, m_byFieldId, TRUE, FALSE);
}

//Kai should be part of D operator
void CFieldFermionWilsonSquareSU3::Ddagger(const CField* pGauge)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    checkCudaErrors(hipMemcpy(m_pDeviceDataCopy, m_pDeviceData, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount, hipMemcpyDeviceToDevice));

    preparethread;
    _kernelDFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceDataCopy, pFieldSU3->m_pDeviceData, m_pDeviceData, m_fKai, m_byFieldId, TRUE, TRUE);
}

void CFieldFermionWilsonSquareSU3::DDdagger(const CField* pGauge)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    preparethread;
    _kernelDFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData, pFieldSU3->m_pDeviceData, m_pDeviceDataCopy, m_fKai, m_byFieldId, TRUE, TRUE);
    _kernelDFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceDataCopy, pFieldSU3->m_pDeviceData, m_pDeviceData, m_fKai, m_byFieldId, TRUE, FALSE);
}

UBOOL CFieldFermionWilsonSquareSU3::InverseD(const CField* pGauge)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return FALSE;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    //Find a solver to solve me.
    return appGetFermionSolver()->Solve(this, /*this is const*/this, pFieldSU3, EFO_F_D);
}

UBOOL CFieldFermionWilsonSquareSU3::InverseDdagger(const CField* pGauge)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return FALSE;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    //Find a solver to solve me.
    return appGetFermionSolver()->Solve(this, /*this is const*/this, pFieldSU3, EFO_F_Ddagger);
}

UBOOL CFieldFermionWilsonSquareSU3::InverseDDdagger(const CField* pGauge)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return FALSE;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    //Find a solver to solve me.
    return appGetFermionSolver()->Solve(this, /*this is const*/this, pFieldSU3, EFO_F_DDdagger);
}

UBOOL CFieldFermionWilsonSquareSU3::CalculateForce(const CFieldGauge* pGauge, CFieldGauge* pForce) const
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return FALSE;
    }
    if (NULL == pForce || EFT_GaugeSU3 != pForce->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return FALSE;
    }

    const CFieldGaugeSU3 * pGaugeSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);
    CFieldGaugeSU3 * pForceSU3 = dynamic_cast<CFieldGaugeSU3*>(pForce);

    CField * pDDaggerPhi = appGetLattice()->GetPooledFieldById(m_byFieldId);
    CField * pDPhi = appGetLattice()->GetPooledFieldById(m_byFieldId);
    if (NULL == pDDaggerPhi || EFT_FermionWilsonSquareSU3 != pDDaggerPhi->GetFieldType()
     || NULL == pDPhi || EFT_FermionWilsonSquareSU3 != pDPhi->GetFieldType())
    {
        appCrucial(_T("Pooled field not found!\n"));
        if (NULL != pDDaggerPhi)
        {
            pDDaggerPhi->Return();
        }
        if (NULL != pDPhi)
        {
            pDPhi->Return();
        }
        return FALSE;
    }
    CFieldFermionWilsonSquareSU3* pDDaggerPhiWilson = dynamic_cast<CFieldFermionWilsonSquareSU3*>(pDDaggerPhi);
    CFieldFermionWilsonSquareSU3* pDPhiWilson = dynamic_cast<CFieldFermionWilsonSquareSU3*>(pDPhi);
    CopyTo(pDDaggerPhiWilson);
    if (!pDDaggerPhiWilson->InverseDDdagger(pGaugeSU3))
    {
        appCrucial(_T("Sparse Linear Solver failed...\n"));
        pDDaggerPhi->Return();
        pDPhi->Return();
        return FALSE;
    }
    //phi 2 = D^{-1}phi = D+ (DD+)^{-1} phi
    //It is faster to calcuate D+ phi2 then D^{-1} phi
    pDDaggerPhiWilson->CopyTo(pDPhiWilson);
    pDPhiWilson->Ddagger(pGaugeSU3);

    preparethread;
    _kernelDWilsonForceSU3 << <block, threads >> > (pDPhiWilson->m_pDeviceData, pDDaggerPhiWilson->m_pDeviceData, pGaugeSU3->m_pDeviceData, pForceSU3->m_pDeviceData, m_fKai, m_byFieldId, TRUE);

    pDDaggerPhi->Return();
    pDPhi->Return();

    return TRUE;
}

__END_NAMESPACE


//=============================================================================
// END OF FILE
//=============================================================================