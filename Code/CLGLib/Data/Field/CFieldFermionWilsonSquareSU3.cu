#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CFieldFermionWilsonSquareSU3.cu
// 
// DESCRIPTION:
// This is the device implementations of Wilson fermion
//
// This implementation assumes SU3 and square lattice
//
// REVISION:
//  [12/27/2018 nbale]
//=============================================================================

#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

__CLGIMPLEMENT_CLASS(CFieldFermionWilsonSquareSU3)

#pragma region Kernel

__global__ void _kernelPrintFermionWilsonSquareSU3(const deviceWilsonVectorSU3 * __restrict__ pData)
{
    intokernal;

    printf("%d=((%1.2f %1.2fi, %1.2f %1.2fi, %1.2f %1.2fi),(%1.2f %1.2fi, %1.2f %1.2fi, %1.2f %1.2fi),(%1.2f %1.2fi, %1.2f %1.2fi, %1.2f %1.2fi),(%1.2f %1.2fi, %1.2f %1.2fi, %1.2f %1.2fi))\n",
        uiSiteIndex,
        pData[uiSiteIndex].m_d[0].m_ve[0].x, pData[uiSiteIndex].m_d[0].m_ve[0].y,
        pData[uiSiteIndex].m_d[0].m_ve[1].x, pData[uiSiteIndex].m_d[0].m_ve[1].y,
        pData[uiSiteIndex].m_d[0].m_ve[2].x, pData[uiSiteIndex].m_d[0].m_ve[2].y,

        pData[uiSiteIndex].m_d[1].m_ve[0].x, pData[uiSiteIndex].m_d[1].m_ve[0].y,
        pData[uiSiteIndex].m_d[1].m_ve[1].x, pData[uiSiteIndex].m_d[1].m_ve[1].y,
        pData[uiSiteIndex].m_d[1].m_ve[2].x, pData[uiSiteIndex].m_d[1].m_ve[2].y,

        pData[uiSiteIndex].m_d[2].m_ve[0].x, pData[uiSiteIndex].m_d[2].m_ve[0].y,
        pData[uiSiteIndex].m_d[2].m_ve[1].x, pData[uiSiteIndex].m_d[2].m_ve[1].y,
        pData[uiSiteIndex].m_d[2].m_ve[2].x, pData[uiSiteIndex].m_d[2].m_ve[2].y,

        pData[uiSiteIndex].m_d[3].m_ve[0].x, pData[uiSiteIndex].m_d[3].m_ve[0].y,
        pData[uiSiteIndex].m_d[3].m_ve[1].x, pData[uiSiteIndex].m_d[3].m_ve[1].y,
        pData[uiSiteIndex].m_d[3].m_ve[2].x, pData[uiSiteIndex].m_d[3].m_ve[2].y
    );
}

__global__ void _kernelAxpyPlusFermionWilsonSquareSU3(
    deviceWilsonVectorSU3 * pMe, 
    const deviceWilsonVectorSU3 * __restrict__ pOther)
{
    intokernal;
    pMe[uiSiteIndex].Add(pOther[uiSiteIndex]);
}

__global__ void _kernelAxpyMinusFermionWilsonSquareSU3(
    deviceWilsonVectorSU3 * pMe, 
    const deviceWilsonVectorSU3 * __restrict__ pOther)
{
    intokernal;
    pMe[uiSiteIndex].Sub(pOther[uiSiteIndex]);
}

__global__ void _kernelAxpyComplexFermionWilsonSquareSU3(
    deviceWilsonVectorSU3 * pMe, 
    const deviceWilsonVectorSU3 * __restrict__ pOther, _Complex a)
{
    intokernal;
    pMe[uiSiteIndex].Add(pOther[uiSiteIndex].MulCompC(a));
}

__global__ void _kernelAxpyRealFermionWilsonSquareSU3(
    deviceWilsonVectorSU3 * pMe, 
    const deviceWilsonVectorSU3 * __restrict__ pOther, Real a)
{
    intokernal;
    pMe[uiSiteIndex].Add(pOther[uiSiteIndex].MulRealC(a));
}

__global__ void _kernelDotFermionWilsonSquareSU3(
    const deviceWilsonVectorSU3 * __restrict__ pMe, 
    const deviceWilsonVectorSU3 * __restrict__ pOther, 
    _Complex * result)
{
    intokernal;
    result[uiSiteIndex] = pMe[uiSiteIndex].ConjugateDotC(pOther[uiSiteIndex]);
}

__global__ void _kernelScalarMultiplyComplex(
    deviceWilsonVectorSU3 * pMe, 
    _Complex a)
{
    intokernal;
    pMe[uiSiteIndex].MulComp(a);
}

__global__ void _kernelScalarMultiplyReal(
    deviceWilsonVectorSU3 * pMe, 
    Real a)
{
    intokernal;
    pMe[uiSiteIndex].MulReal(a);
}

/**
*
*/
__global__ void _kernelInitialFermionWilsonSquareSU3(
    deviceWilsonVectorSU3 *pDevicePtr, 
    EFieldInitialType eInitialType)
{
    intokernal;

    switch (eInitialType)
    {
    case EFIT_Zero:
    {
        pDevicePtr[uiSiteIndex] = deviceWilsonVectorSU3::makeZeroWilsonVectorSU3();
    }
    break;
    case EFIT_RandomGaussian:
    {
        pDevicePtr[uiSiteIndex] = deviceWilsonVectorSU3::makeRandomGaussian(_deviceGetFatIndex(uiSiteIndex, 0));
    }
    break;
    default:
    {
        printf("Wilson Fermion Field cannot be initialized with this type!");
    }
    break;
    }
}

/**
* Dw phi(x) = phi(x) - kai sum _mu (1-gamma _mu) U(x,mu) phi(x+ mu) + (1+gamma _mu) U^{dagger}(x-mu) phi(x-mu)
* U act on su3
* gamma act on spinor
*
* If bDagger, it is gamma5, D, gamma5
*
*/
__global__ void _kernelDFermionWilsonSquareSU3(
    const deviceWilsonVectorSU3* __restrict__ pDeviceData,
    const deviceSU3* __restrict__ pGauge,
    deviceWilsonVectorSU3* pResultData,
    Real kai,
    BYTE byFieldId,
    UBOOL bDiracChiralGamma,
    UBOOL bDDagger,
    EOperatorCoefficientType eCoeff,
    Real fCoeff,
    _Complex cCoeff)
{
    intokernaldir;

    gammaMatrix gamma5 = bDiracChiralGamma ? __diracGamma[GAMMA5] : __chiralGamma[GAMMA5];

    deviceWilsonVectorSU3 result = deviceWilsonVectorSU3::makeZeroWilsonVectorSU3();
    pResultData[uiSiteIndex] = pDeviceData[uiSiteIndex];
    if (bDDagger)
    {
        pResultData[uiSiteIndex] = gamma5.MulWilsonC(pResultData[uiSiteIndex]);
    }

    //idir = mu
    for (UINT idir = 0; idir < uiDir; ++idir)
    {
        //Get Gamma mu
        gammaMatrix gammaMu = bDiracChiralGamma ? __diracGamma[GAMMA1 + idir] : __chiralGamma[GAMMA1 + idir];

        //x, mu
        UINT linkIndex = _deviceGetLinkIndex(uiSiteIndex, idir);

        SIndex x_m_mu_Gauge = __idx->_deviceGaugeIndexWalk(uiSiteIndex, -(idir + 1));
        SIndex x_p_mu_Fermion = __idx->_deviceFermionIndexWalk(byFieldId, uiSiteIndex, (idir + 1));
        SIndex x_m_mu_Fermion = __idx->_deviceFermionIndexWalk(byFieldId, uiSiteIndex, -(idir + 1));

        //Assuming periodic
        //get U(x,mu), U^{dagger}(x-mu), 
        deviceSU3 x_Gauge_element = pGauge[linkIndex];
        deviceSU3 x_m_mu_Gauge_element = pGauge[_deviceGetLinkIndex(x_m_mu_Gauge.m_uiSiteIndex, idir)];
        x_m_mu_Gauge_element.Dagger();

        deviceWilsonVectorSU3 x_p_mu_Fermion_element = pDeviceData[x_p_mu_Fermion.m_uiSiteIndex];
        deviceWilsonVectorSU3 x_m_mu_Fermion_element = pDeviceData[x_m_mu_Fermion.m_uiSiteIndex];

        if (bDDagger)
        {
            x_p_mu_Fermion_element = gamma5.MulWilsonC(x_p_mu_Fermion_element);
            x_m_mu_Fermion_element = gamma5.MulWilsonC(x_m_mu_Fermion_element);
        }

        //hopping terms

        //U(x,mu) phi(x+ mu)
        deviceWilsonVectorSU3 u_phi_x_p_m = x_Gauge_element.MulWilsonVector(x_p_mu_Fermion_element);
        result.Add(u_phi_x_p_m);

        //- gammamu U(x,mu) phi(x+ mu)
        result.Sub(gammaMu.MulWilsonC(u_phi_x_p_m));

        //U^{dagger}(x-mu) phi(x-mu)
        deviceWilsonVectorSU3 u_dagger_phi_x_m_m = x_m_mu_Gauge_element.MulWilsonVector(x_m_mu_Fermion_element);
        result.Add(u_dagger_phi_x_m_m);

        //gammamu U^{dagger}(x-mu) phi(x-mu)
        result.Add(gammaMu.MulWilsonC(u_dagger_phi_x_m_m));
    }

    //result = phi(x) - kai sum _mu result
    result.MulReal(kai);
    pResultData[uiSiteIndex].Sub(result);

    if (bDDagger)
    {
        pResultData[uiSiteIndex] = gamma5.MulWilsonC(pResultData[uiSiteIndex]);
    }

    switch (eCoeff)
    {
    case EOCT_Real:
        pResultData[uiSiteIndex].MulReal(fCoeff);
        break;
    case EOCT_Complex:
        pResultData[uiSiteIndex].MulComp(cCoeff);
        break;
    }
}

/**
* The output is on a gauge field
* Therefor cannot make together with _kernelDWilson
*
*/
__global__ void _kernelDWilsonForceSU3(
    const deviceWilsonVectorSU3* __restrict__ pInverseD,
    const deviceWilsonVectorSU3* __restrict__ pInverseDDdagger,
    const deviceSU3* __restrict__ pGauge,
    deviceSU3* pForce,
    deviceSU3* pCachedForce,
    Real fKai,
    BYTE byFieldId,
    UBOOL bDiracChiralGamma)
{
    intokernaldir;

    deviceWilsonVectorSU3 x_Left(pInverseDDdagger[uiSiteIndex]);
    deviceWilsonVectorSU3 x_Right(pInverseD[uiSiteIndex]);

    //idir = mu
    for (UINT idir = 0; idir < uiDir; ++idir)
    {
        //Get Gamma mu
        gammaMatrix gammaMu = bDiracChiralGamma ? __diracGamma[GAMMA1 + idir] : __chiralGamma[GAMMA1 + idir];

        //x, mu
        UINT linkIndex = _deviceGetLinkIndex(uiSiteIndex, idir);

        SIndex x_m_mu_Gauge = __idx->_deviceGaugeIndexWalk(uiSiteIndex, -(idir + 1));
        SIndex x_p_mu_Fermion = __idx->_deviceFermionIndexWalk(byFieldId, uiSiteIndex, (idir + 1));

        deviceWilsonVectorSU3 x_p_mu_Right(pInverseD[x_p_mu_Fermion.m_uiSiteIndex]);
        deviceWilsonVectorSU3 x_p_mu_Left(pInverseDDdagger[x_p_mu_Fermion.m_uiSiteIndex]);
        deviceSU3 x_Gauge_element = pGauge[linkIndex];

        deviceWilsonVectorSU3 right1(x_p_mu_Right);
        right1.Sub(gammaMu.MulWilsonC(right1));
        deviceSU3 mid = deviceSU3::makeSU3Contract(x_Left, right1);

        deviceWilsonVectorSU3 right2(x_Right);
        right2.Add(gammaMu.MulWilsonC(right2));
        mid.Add(deviceSU3::makeSU3Contract(right2, x_p_mu_Left));

        deviceSU3 forceOfThisLink = x_Gauge_element.MulC(mid);
        forceOfThisLink.Ta();
        forceOfThisLink.MulReal(fKai);

        pForce[linkIndex].Add(forceOfThisLink);
        if (NULL != pCachedForce)
        {
            pCachedForce[linkIndex] = forceOfThisLink;
        }
    }
}

__global__ void _kernelApplyGammaSU3(deviceWilsonVectorSU3* pDeviceData, UINT uiGamma, UBOOL bDiracChiralGamma)
{
    intokernal;
    pDeviceData[uiSiteIndex] = (bDiracChiralGamma ? __diracGamma[uiGamma] : __chiralGamma[uiGamma]).MulWilsonC(pDeviceData[uiSiteIndex]);
}

#pragma endregion

CFieldFermionWilsonSquareSU3::CFieldFermionWilsonSquareSU3() : CFieldFermion(), m_fKai(F(0.125))
{
    checkCudaErrors(hipMalloc((void**)&m_pDeviceData, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount));
    checkCudaErrors(hipMalloc((void**)&m_pDeviceDataCopy, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount));
}

CFieldFermionWilsonSquareSU3::~CFieldFermionWilsonSquareSU3()
{
    checkCudaErrors(hipFree(m_pDeviceData));
    checkCudaErrors(hipFree(m_pDeviceDataCopy));
}

/**
*
*/
void CFieldFermionWilsonSquareSU3::InitialField(EFieldInitialType eInitialType)
{
    preparethread;
    _kernelInitialFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData, eInitialType);
}

void CFieldFermionWilsonSquareSU3::InitialFieldWithFile(const CCString& sFileName, EFieldFileType eFieldType)
{
    if (eFieldType != EFFT_CLGBin)
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3::InitialFieldWithFile: Only support CLG Bin File\n"));
        return;
    }

    UINT uiSize = static_cast<UINT>(sizeof(FLOAT) * 24 * m_uiSiteCount);
    deviceWilsonVectorSU3* readData = (deviceWilsonVectorSU3*)malloc(sizeof(deviceWilsonVectorSU3) * m_uiSiteCount);
    BYTE* data = appGetFileSystem()->ReadAllBytes(sFileName.c_str(), uiSize);
    for (UINT i = 0; i < m_uiSiteCount; ++i)
    {
        FLOAT thisSite[24];
        memcpy(thisSite, data + i * sizeof(FLOAT) * 24, sizeof(FLOAT) * 24);
        for (UINT j = 0; j < 4; ++j)
        {
            for (UINT k = 0; k < 3; ++k)
            {
                readData[i].m_d[j].m_ve[k] = _make_cuComplex(
                    static_cast<Real>(thisSite[2 * (j * 3 + k)]), 
                    static_cast<Real>(thisSite[2 * (j * 3 + k) + 1]));
            }
        }
    }
    checkCudaErrors(hipMemcpy(m_pDeviceData, readData, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount, hipMemcpyHostToDevice));
    free(data);
    free(readData);
}

void CFieldFermionWilsonSquareSU3::InitialOtherParameters(CParameters& params)
{
    params.FetchValueReal(_T("Hopping"), m_fKai);
    if (m_fKai < F(0.00000001))
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3: Kai is nearly 0, such that Dphi \approx phi! This will cause problem!\n"));
    }
    CCommonData::m_fKai = m_fKai;
}

void CFieldFermionWilsonSquareSU3::DebugPrintMe() const
{
    preparethread;
    _kernelPrintFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData);
}

void CFieldFermionWilsonSquareSU3::CopyTo(CField* U) const
{
    if (NULL == U || EFT_FermionWilsonSquareSU3 != U->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only copy to CFieldFermionWilsonSquareSU3!"));
        return;
    }

    CField::CopyTo(U);

    CFieldFermionWilsonSquareSU3 * pField = dynamic_cast<CFieldFermionWilsonSquareSU3*>(U);
    checkCudaErrors(hipMemcpy(pField->m_pDeviceData, m_pDeviceData, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount, hipMemcpyDeviceToDevice));
    pField->m_byFieldId = m_byFieldId;
    pField->m_fKai = m_fKai;
}

void CFieldFermionWilsonSquareSU3::AxpyPlus(const CField* x)
{
    if (NULL == x || EFT_FermionWilsonSquareSU3 != x->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only copy to CFieldFermionWilsonSquareSU3!"));
        return;
    }
    const CFieldFermionWilsonSquareSU3 * pField = dynamic_cast<const CFieldFermionWilsonSquareSU3*>(x);

    preparethread;
    _kernelAxpyPlusFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData, pField->m_pDeviceData);
}

void CFieldFermionWilsonSquareSU3::AxpyMinus(const CField* x)
{
    if (NULL == x || EFT_FermionWilsonSquareSU3 != x->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only copy to CFieldFermionWilsonSquareSU3!"));
        return;
    }
    const CFieldFermionWilsonSquareSU3 * pField = dynamic_cast<const CFieldFermionWilsonSquareSU3*>(x);

    preparethread;
    _kernelAxpyMinusFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData, pField->m_pDeviceData);
}

void CFieldFermionWilsonSquareSU3::Axpy(Real a, const CField* x)
{
    if (NULL == x || EFT_FermionWilsonSquareSU3 != x->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only copy to CFieldFermionWilsonSquareSU3!"));
        return;
    }
    const CFieldFermionWilsonSquareSU3 * pField = dynamic_cast<const CFieldFermionWilsonSquareSU3*>(x);

    preparethread;
    _kernelAxpyRealFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData, pField->m_pDeviceData, a);
}

void CFieldFermionWilsonSquareSU3::Axpy(const _Complex& a, const CField* x)
{
    if (NULL == x || EFT_FermionWilsonSquareSU3 != x->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only copy to CFieldFermionWilsonSquareSU3!"));
        return;
    }
    const CFieldFermionWilsonSquareSU3 * pField = dynamic_cast<const CFieldFermionWilsonSquareSU3*>(x);

    preparethread;
    _kernelAxpyComplexFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData, pField->m_pDeviceData, a);
}

_Complex CFieldFermionWilsonSquareSU3::Dot(const CField* x) const
{
    if (NULL == x || EFT_FermionWilsonSquareSU3 != x->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only copy to CFieldFermionWilsonSquareSU3!"));
        return _make_cuComplex(0,0);
    }
    const CFieldFermionWilsonSquareSU3 * pField = dynamic_cast<const CFieldFermionWilsonSquareSU3*>(x);

    preparethread;
    _kernelDotFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData, pField->m_pDeviceData, _D_ComplexThreadBuffer);

    return appGetCudaHelper()->ThreadBufferSum(_D_ComplexThreadBuffer);
}

void CFieldFermionWilsonSquareSU3::ScalarMultply(const _Complex& a)
{
    preparethread;
    _kernelScalarMultiplyComplex << <block, threads >> >(m_pDeviceData, a);
}

void CFieldFermionWilsonSquareSU3::ScalarMultply(Real a)
{
    preparethread;
    _kernelScalarMultiplyReal << <block, threads >> >(m_pDeviceData, a);
}

void CFieldFermionWilsonSquareSU3::ApplyGamma(EGammaMatrix eGamma)
{
    preparethread;
    _kernelApplyGammaSU3 << <block, threads >> >(m_pDeviceData, static_cast<UINT>(eGamma), TRUE);
}
/**
* generate phi by gaussian random.
* phi = D phi
*/
void CFieldFermionWilsonSquareSU3::PrepareForHMC(const CFieldGauge* pGauge)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    preparethread;
    _kernelInitialFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceDataCopy, EFIT_RandomGaussian);
    _kernelDFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceDataCopy, pFieldSU3->m_pDeviceData, m_pDeviceData, m_fKai, m_byFieldId, TRUE, FALSE, EOCT_None, F(1.0), _make_cuComplex(F(1.0), F(0.0)));

    //cache a inverse DDdagger field
    CFieldCache* pCache = appGetLattice()->m_pFieldCache;
    CField* pField = pCache->GetCachedField(CFieldCache::CachedInverseDDdaggerField);
    if (NULL == pField)
    {
        pField = GetCopy();
        pCache->CacheField(CFieldCache::CachedInverseDDdaggerField, pField);
    }
    else
    {
        CopyTo(pField);
    }
    CFieldFermionWilsonSquareSU3* pCachedSU3 = dynamic_cast<CFieldFermionWilsonSquareSU3*>(pField);
    pCachedSU3->InverseDDdagger(pGauge);
}

//Kai should be part of D operator
void CFieldFermionWilsonSquareSU3::D(const CField* pGauge, EOperatorCoefficientType eCoeffType, Real fCoeffReal, Real fCoeffImg)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    checkCudaErrors(hipMemcpy(m_pDeviceDataCopy, m_pDeviceData, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount, hipMemcpyDeviceToDevice));

    Real fRealCoeff = fCoeffReal;
    _Complex cCompCoeff = _make_cuComplex(fCoeffReal, fCoeffImg);
    if (EOCT_Minus == eCoeffType)
    {
        eCoeffType = EOCT_Real;
        fRealCoeff = F(-1.0);
    }

    preparethread;
    _kernelDFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceDataCopy, pFieldSU3->m_pDeviceData, m_pDeviceData, m_fKai, m_byFieldId, TRUE, FALSE, eCoeffType, fRealCoeff, cCompCoeff);
}

//Kai should be part of D operator
void CFieldFermionWilsonSquareSU3::Ddagger(const CField* pGauge, EOperatorCoefficientType eCoeffType, Real fCoeffReal, Real fCoeffImg)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    checkCudaErrors(hipMemcpy(m_pDeviceDataCopy, m_pDeviceData, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount, hipMemcpyDeviceToDevice));

    Real fRealCoeff = fCoeffReal;
    _Complex cCompCoeff = _make_cuComplex(fCoeffReal, fCoeffImg);
    if (EOCT_Minus == eCoeffType)
    {
        eCoeffType = EOCT_Real;
        fRealCoeff = F(-1.0);
    }

    preparethread;
    _kernelDFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceDataCopy, pFieldSU3->m_pDeviceData, m_pDeviceData, m_fKai, m_byFieldId, TRUE, TRUE, eCoeffType, fRealCoeff, cCompCoeff);
}

void CFieldFermionWilsonSquareSU3::DDdagger(const CField* pGauge, EOperatorCoefficientType eCoeffType, Real fCoeffReal, Real fCoeffImg)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    Real fRealCoeff = fCoeffReal;
    _Complex cCompCoeff = _make_cuComplex(fCoeffReal, fCoeffImg);
    if (EOCT_Minus == eCoeffType)
    {
        eCoeffType = EOCT_Real;
        fRealCoeff = F(-1.0);
    }

    preparethread;
    //Ddagger first, m_pDeviceDataCopy = D+ m_pDeviceData
    _kernelDFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceData, pFieldSU3->m_pDeviceData, m_pDeviceDataCopy, m_fKai, m_byFieldId, TRUE, TRUE, EOCT_None, F(1.0), _make_cuComplex(F(1.0), F(0.0)));
    //Then D, m_pDeviceData = D m_pDeviceDataCopy
    _kernelDFermionWilsonSquareSU3 << <block, threads >> > (m_pDeviceDataCopy, pFieldSU3->m_pDeviceData, m_pDeviceData, m_fKai, m_byFieldId, TRUE, FALSE, eCoeffType, fRealCoeff, cCompCoeff);
}

UBOOL CFieldFermionWilsonSquareSU3::InverseD(const CField* pGauge)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return FALSE;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    //Find a solver to solve me.
    return appGetFermionSolver()->Solve(this, /*this is const*/this, pFieldSU3, EFO_F_D);
}

UBOOL CFieldFermionWilsonSquareSU3::InverseDdagger(const CField* pGauge)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return FALSE;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    //Find a solver to solve me.
    return appGetFermionSolver()->Solve(this, /*this is const*/this, pFieldSU3, EFO_F_Ddagger);
}

UBOOL CFieldFermionWilsonSquareSU3::InverseDDdagger(const CField* pGauge)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return FALSE;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    //Find a solver to solve me.
    return appGetFermionSolver()->Solve(this, /*this is const*/this, pFieldSU3, EFO_F_DDdagger);
}

UBOOL CFieldFermionWilsonSquareSU3::CalculateForce(const CFieldGauge* pGauge, CFieldGauge* pForce, CFieldGauge* pCachedForce) const
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return FALSE;
    }
    if (NULL == pForce || EFT_GaugeSU3 != pForce->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return FALSE;
    }
    if (NULL != pCachedForce && EFT_GaugeSU3 != pCachedForce->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return FALSE;
    }

    const CFieldGaugeSU3 * pGaugeSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);
    CFieldGaugeSU3 * pForceSU3 = dynamic_cast<CFieldGaugeSU3*>(pForce);
    CFieldGaugeSU3 * pCacheForceForceSU3 = NULL;
    if (NULL != pCachedForce)
    {
        pCacheForceForceSU3 = dynamic_cast<CFieldGaugeSU3*>(pCachedForce);
    }

    CField * pDDaggerPhi = appGetLattice()->GetPooledFieldById(m_byFieldId);
    CField * pDPhi = appGetLattice()->GetPooledFieldById(m_byFieldId);
    CField * pCachedField = appGetLattice()->m_pFieldCache->GetCachedField(CFieldCache::CachedInverseDDdaggerField);
    if (NULL == pDDaggerPhi || EFT_FermionWilsonSquareSU3 != pDDaggerPhi->GetFieldType()
     || NULL == pDPhi || EFT_FermionWilsonSquareSU3 != pDPhi->GetFieldType())
    {
        appCrucial(_T("Pooled field not found!\n"));
        if (NULL != pDDaggerPhi)
        {
            pDDaggerPhi->Return();
        }
        if (NULL != pDPhi)
        {
            pDPhi->Return();
        }
        return FALSE;
    }
    CFieldFermionWilsonSquareSU3* pDDaggerPhiWilson = dynamic_cast<CFieldFermionWilsonSquareSU3*>(pDDaggerPhi);
    CFieldFermionWilsonSquareSU3* pDPhiWilson = dynamic_cast<CFieldFermionWilsonSquareSU3*>(pDPhi);
    //if (!pDDaggerPhiWilson->InverseDDdagger(pGaugeSU3))
    if (!appGetFermionSolver()->Solve(pDDaggerPhiWilson, this, pGaugeSU3, EFO_F_DDdagger, pCachedField))
    {
        appCrucial(_T("Sparse Linear Solver failed...\n"));
        pDDaggerPhi->Return();
        pDPhi->Return();
        return FALSE;
    }
    //phi 2 = D^{-1}phi = D+ (DD+)^{-1} phi
    //It is faster to calcuate D+ phi2 then D^{-1} phi
    pDDaggerPhiWilson->CopyTo(pDPhiWilson);
    if (NULL != pCachedField)
    {
        //The gauge field is changing slowly, and D depends only on gauge, also change slowly
        //Use the last solution as start point will accelerate the solver, so we cache it
        pDDaggerPhiWilson->CopyTo(pCachedField);
    }
    pDPhiWilson->Ddagger(pGaugeSU3);

    preparethread;
    _kernelDWilsonForceSU3 << <block, threads >> > (
        pDPhiWilson->m_pDeviceData,
        pDDaggerPhiWilson->m_pDeviceData,
        pGaugeSU3->m_pDeviceData,
        pForceSU3->m_pDeviceData,
        NULL == pCacheForceForceSU3 ? NULL : pCacheForceForceSU3->m_pDeviceData,
        m_fKai, m_byFieldId, TRUE);

    pDDaggerPhi->Return();
    pDPhi->Return();

    return TRUE;
}

void CFieldFermionWilsonSquareSU3::SetKai(Real fKai)
{
    m_fKai = fKai;
    CCommonData::m_fKai = fKai;
}

void CFieldFermionWilsonSquareSU3::SaveToFile(const CCString &fileName) const
{
    deviceWilsonVectorSU3* toSave = (deviceWilsonVectorSU3*)malloc(sizeof(deviceWilsonVectorSU3) * m_uiSiteCount);
    UINT uiSize = static_cast<UINT>(sizeof(FLOAT) * m_uiSiteCount * 24);
    BYTE* saveData = (BYTE*)malloc(static_cast<size_t>(uiSize));
    checkCudaErrors(hipMemcpy(toSave, m_pDeviceData, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount, hipMemcpyDeviceToHost));
    for (UINT i = 0; i < m_uiSiteCount; ++i)
    {
        FLOAT oneSite[24];
        for (UINT j = 0; j < 4; ++j)
        {
            for (UINT k = 0; k < 3; ++k)
            {
                oneSite[2 * (j * 3 + k)] = static_cast<FLOAT>(toSave[i].m_d[j].m_ve[k].x);
                oneSite[2 * (j * 3 + k) + 1] = static_cast<FLOAT>(toSave[i].m_d[j].m_ve[k].y);
            }
        }
        memcpy(saveData + sizeof(FLOAT) * i * 24, oneSite, sizeof(FLOAT) * 24);
    }
    appGetFileSystem()->WriteAllBytes(fileName.c_str(), saveData, uiSize);
    free(saveData);
    free(toSave);
}

CCString CFieldFermionWilsonSquareSU3::GetInfos(const CCString &tab) const
{
    CCString sRet;
    sRet = tab + _T("Name : CFieldFermionWilsonSquareSU3\n");
    sRet = sRet + tab + _T("Hopping : ") + appFloatToString(CCommonData::m_fKai) + _T("\n");
    return sRet;
}

__END_NAMESPACE


//=============================================================================
// END OF FILE
//=============================================================================