#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CFieldFermionWilsonSquareSU3.cu
// 
// DESCRIPTION:
// This is the device implementations of Wilson fermion
//
// This implementation assumes SU3 and square lattice
//
// REVISION:
//  [12/27/2018 nbale]
//=============================================================================

#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

__CLGIMPLEMENT_CLASS(CFieldFermionWilsonSquareSU3)

#pragma region Kernel

__global__ void _kernelPrintFermionWilsonSquareSU3(const deviceWilsonVectorSU3 * __restrict__ pData)
{
    intokernal;

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        UINT siteIndexX = _deviceGetSiteIndex(coord);

        printf("xyzt:%d,%d,%d,%d = ((%f+%f i, %f+%f i, %f+%f i),(%f+%f i, %f+%f i, %f+%f i),(%f+%f i, %f+%f i, %f+%f i),(%f+%f i, %f+%f i, %f+%f i))\n", 
            coord[0], coord[1], coord[2], coord[3],
            pData[siteIndexX].m_d[0].m_ve[0].x, pData[siteIndexX].m_d[0].m_ve[0].y,
            pData[siteIndexX].m_d[0].m_ve[1].x, pData[siteIndexX].m_d[0].m_ve[1].y,
            pData[siteIndexX].m_d[0].m_ve[2].x, pData[siteIndexX].m_d[0].m_ve[2].y,

            pData[siteIndexX].m_d[1].m_ve[0].x, pData[siteIndexX].m_d[1].m_ve[0].y,
            pData[siteIndexX].m_d[1].m_ve[1].x, pData[siteIndexX].m_d[1].m_ve[1].y,
            pData[siteIndexX].m_d[1].m_ve[2].x, pData[siteIndexX].m_d[1].m_ve[2].y,

            pData[siteIndexX].m_d[2].m_ve[0].x, pData[siteIndexX].m_d[2].m_ve[0].y,
            pData[siteIndexX].m_d[2].m_ve[1].x, pData[siteIndexX].m_d[2].m_ve[1].y,
            pData[siteIndexX].m_d[2].m_ve[2].x, pData[siteIndexX].m_d[2].m_ve[2].y,

            pData[siteIndexX].m_d[3].m_ve[0].x, pData[siteIndexX].m_d[3].m_ve[0].y,
            pData[siteIndexX].m_d[3].m_ve[1].x, pData[siteIndexX].m_d[3].m_ve[1].y,
            pData[siteIndexX].m_d[3].m_ve[2].x, pData[siteIndexX].m_d[3].m_ve[2].y
            );
    }
}

/**
* phi dagger, phi
*/
__global__ void _kernelDotFermionWilsonSquareSU3(const deviceWilsonVectorSU3 * __restrict__ pLeft,
                                           const deviceWilsonVectorSU3 * __restrict__ pRight,
                                           deviceSU3* result)
{
    intokernal;

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;

        for (int idir = 0; idir < uiDir; ++idir)
        {
            UINT linkIndex = _deviceGetLinkIndex(coord, idir);

            deviceSU3 resultThisLink = deviceSU3::makeSU3Zero();
            for (int i = 0; i < 8; ++i)
            {
                _Complex omega = pLeft[linkIndex * 8 + i].ConjugateDotC(pRight[linkIndex * 8 + i]);
                resultThisLink.Add(__SU3Generators[i]->Mulc(omega));
            }
            result[linkIndex] = resultThisLink;
        }
    }
}

/**
*
*/
__global__ void _kernelInitialFermionWilsonSquareSU3(deviceWilsonVectorSU3 *pDevicePtr, EFieldInitialType eInitialType)
{
    intokernal;

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        UINT siteIndexX = _deviceGetSiteIndex(coord);
        UINT fatIndex = _deviceGetFatIndex(siteIndexX, 0);

        switch (eInitialType)
        {
            case EFIT_Zero:
                {
                    pDevicePtr[siteIndexX].MakeZero();
                }
                break;
            case EFIT_RandomGaussian:
                {
                    pDevicePtr[siteIndexX].MakeRandomGaussian(fatIndex);
                }
                break;
            default:
                {
                    printf("Wilson Fermion Field cannot be initialized with this type!");
                }
            break;
        }
    }
}

/**
* Dw phi(x) = phi(x) - kai sum _mu (1-gamma _mu) U(x,mu) phi(x+ mu) + (1+gamma _mu) U^{dagger}(x-mu) phi(x-mu)
* U act on su3
* gamma act on spinor
*
* If bDagger, it is gamma5, D, gamma5
*
*/
__global__ void _kernelDFermionWilsonSquareSU3(const deviceWilsonVectorSU3* __restrict__ pDeviceData,
                                  const deviceSU3* __restrict__ pGauge,
                                  deviceWilsonVectorSU3* pResultData,
                                  Real kai,
                                  BYTE byFieldId,
                                  UBOOL bDiracChiralGamma,
                                  UBOOL bDDagger)
{
    intokernal;

    gammaMatrix gamma5 = bDiracChiralGamma ? __diracGamma->m_gm[gammaMatrixSet::GAMMA5] : __chiralGamma->m_gm[gammaMatrixSet::GAMMA5];

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        //x
        UINT siteIndexX = _deviceGetSiteIndex(coord);
        deviceWilsonVectorSU3 result;
        deviceWilsonVectorSU3 x_Fermion_element = pDeviceData[siteIndexX];
        if (bDDagger)
        {
            x_Fermion_element = gamma5.MulC(x_Fermion_element);
        }

        //idir = mu
        for (UINT idir = 0; idir < uiDir; ++idir)
        {
            //Get Gamma mu
            gammaMatrix gammaMu = bDiracChiralGamma ? 
                  __diracGamma->m_gm[gammaMatrixSet::GAMMA1 + idir]
                : __chiralGamma->m_gm[gammaMatrixSet::GAMMA1 + idir];

            //x, mu
            UINT linkIndex = _deviceGetLinkIndex(coord, idir);

            SIndex x_m_mu_Gauge = __idx->_deviceGaugeIndexWalk(siteIndexX, -(idir + 1));
            SIndex x_p_mu_Fermion = __idx->_deviceFermionIndexWalk(byFieldId, siteIndexX, (idir + 1));
            SIndex x_m_mu_Fermion = __idx->_deviceFermionIndexWalk(byFieldId, siteIndexX, -(idir + 1));

            //Assuming periodic
            //get U(x,mu), U^{dagger}(x-mu), 
            deviceSU3 x_Gauge_element = pGauge[linkIndex];
            deviceSU3 x_m_mu_Gauge_element = pGauge[_deviceGetLinkIndex(x_m_mu_Gauge.m_uiSiteIndex, idir)];
            x_m_mu_Gauge_element.Dagger();
            deviceWilsonVectorSU3 x_p_mu_Fermion_element = pDeviceData[x_p_mu_Fermion.m_uiSiteIndex];
            deviceWilsonVectorSU3 x_m_mu_Fermion_element = pDeviceData[x_m_mu_Fermion.m_uiSiteIndex];
            if (bDDagger)
            {
                x_p_mu_Fermion_element = gamma5.MulC(x_p_mu_Fermion_element);
                x_m_mu_Fermion_element = gamma5.MulC(x_m_mu_Fermion_element);
            }

            //hopping terms
            for (UINT iSpinor = 0; iSpinor < 4; ++iSpinor) //Wilson fermion is 4-spinor
            {
                //U(x,mu) phi(x+ mu)
                result.m_d[iSpinor] = result.m_d[iSpinor].AddC(x_Gauge_element.Mul(x_p_mu_Fermion_element.m_d[iSpinor]));

                //- gammamu U(x,mu) phi(x+ mu)
                result.m_d[iSpinor] = result.m_d[iSpinor].SubC(x_Gauge_element.Mul(gammaMu.MulC(x_p_mu_Fermion_element, iSpinor)));

                //U^{dagger}(x-mu) phi(x-mu)
                result.m_d[iSpinor] = result.m_d[iSpinor].AddC(x_m_mu_Gauge_element.Mul(x_m_mu_Fermion_element.m_d[iSpinor]));

                //gammamu U^{dagger}(x-mu) phi(x-mu)
                result.m_d[iSpinor] = result.m_d[iSpinor].AddC(x_m_mu_Gauge_element.Mul(gammaMu.MulC(x_m_mu_Fermion_element, iSpinor)));
            }
        }

        //result = phi(x) - kai sum _mu result
        result.Mul(_make_cuComplex(kai, 0));
        pResultData[siteIndexX] = x_Fermion_element.SubC(result);
        if (bDDagger)
        {
            pResultData[siteIndexX] = gamma5.MulC(pResultData[siteIndexX]);
        }
    }
}

/**
* The output is on a gauge field
* Therefor cannot make together with _kernelDWilson
*
*/
__global__ void _kernelDWilsonMuSU3(const deviceWilsonVectorSU3* __restrict__ pDeviceData,
                                    const deviceSU3* __restrict__ pGauge,
                                    deviceWilsonVectorSU3* pResultDataArray,
                                    Real kai,
                                    BYTE byFieldId,
                                    UBOOL bDiracChiralGamma,
                                    UBOOL bDDagger,
                                    UBOOL bPartialOmega)
{
    intokernal;

    gammaMatrix gamma5 = bDiracChiralGamma ? __diracGamma->m_gm[gammaMatrixSet::GAMMA5] : __chiralGamma->m_gm[gammaMatrixSet::GAMMA5];

    for (UINT it = 0; it < uiTLength; ++it)
    {
        coord[3] = it;
        //x
        UINT siteIndexX = _deviceGetSiteIndex(coord);
        deviceWilsonVectorSU3 x_Fermion_element = pDeviceData[siteIndexX];
        if (bDDagger)
        {
            x_Fermion_element = gamma5.MulC(x_Fermion_element);
        }

        //idir = mu
        for (UINT idir = 0; idir < uiDir; ++idir)
        {
            deviceWilsonVectorSU3 result[8];

            //Get Gamma mu
            gammaMatrix gammaMu = bDiracChiralGamma ?
                  __diracGamma->m_gm[gammaMatrixSet::GAMMA1 + idir]
                : __chiralGamma->m_gm[gammaMatrixSet::GAMMA1 + idir];

            //x, mu
            UINT linkIndex = _deviceGetLinkIndex(coord, idir);

            SIndex x_m_mu_Gauge = __idx->_deviceGaugeIndexWalk(siteIndexX, -(idir + 1));
            SIndex x_p_mu_Fermion = __idx->_deviceFermionIndexWalk(byFieldId, siteIndexX, (idir + 1));
            SIndex x_m_mu_Fermion = __idx->_deviceFermionIndexWalk(byFieldId, siteIndexX, -(idir + 1));

            //Assuming periodic
            //get U(x,mu), U^{dagger}(x-mu), 
            deviceSU3 x_Gauge_element = pGauge[linkIndex];
            deviceSU3 x_m_mu_Gauge_element = pGauge[_deviceGetLinkIndex(x_m_mu_Gauge.m_uiSiteIndex, idir)];
            x_m_mu_Gauge_element.Dagger();
            deviceWilsonVectorSU3 x_p_mu_Fermion_element = pDeviceData[x_p_mu_Fermion.m_uiSiteIndex];
            deviceWilsonVectorSU3 x_m_mu_Fermion_element = pDeviceData[x_m_mu_Fermion.m_uiSiteIndex];
            if (bDDagger)
            {
                x_p_mu_Fermion_element = gamma5.MulC(x_p_mu_Fermion_element);
                x_m_mu_Fermion_element = gamma5.MulC(x_m_mu_Fermion_element);
            }

            //hopping terms
            for (UINT iSpinor = 0; iSpinor < 4; ++iSpinor) //Wilson fermion is 4-spinor
            {
                for (int i = 0; i < 8; ++i)
                {
                    if (!bPartialOmega)
                    {
                        //U(x,mu) phi(x+ mu)
                        result[i].m_d[iSpinor] = result[i].m_d[iSpinor].AddC(x_Gauge_element.Mul(x_p_mu_Fermion_element.m_d[iSpinor]));

                        //- gammamu U(x,mu) phi(x+ mu)
                        result[i].m_d[iSpinor] = result[i].m_d[iSpinor].SubC(x_Gauge_element.Mul(gammaMu.MulC(x_p_mu_Fermion_element, iSpinor)));

                        //U^{dagger}(x-mu) phi(x-mu)
                        result[i].m_d[iSpinor] = result[i].m_d[iSpinor].AddC(x_m_mu_Gauge_element.Mul(x_m_mu_Fermion_element.m_d[iSpinor]));

                        //gammamu U^{dagger}(x-mu) phi(x-mu)
                        result[i].m_d[iSpinor] = result[i].m_d[iSpinor].AddC(x_m_mu_Gauge_element.Mul(gammaMu.MulC(x_m_mu_Fermion_element, iSpinor)));
                    }
                    else
                    {
                        //U(x,mu) phi(x+ mu)
                        result[i].m_d[iSpinor] = result[i].m_d[iSpinor].AddC(__SU3Generators[i]->Mulc(x_Gauge_element).Mul(x_p_mu_Fermion_element.m_d[iSpinor]));

                        //- gammamu U(x,mu) phi(x+ mu)
                        result[i].m_d[iSpinor] = result[i].m_d[iSpinor].SubC(__SU3Generators[i]->Mulc(x_Gauge_element).Mul(gammaMu.MulC(x_p_mu_Fermion_element, iSpinor)));

                        //U^{dagger}(x-mu) phi(x-mu)
                        result[i].m_d[iSpinor] = result[i].m_d[iSpinor].AddC(__SU3Generators[i]->Mulc(x_m_mu_Gauge_element).Mul(x_m_mu_Fermion_element.m_d[iSpinor]));

                        //gammamu U^{dagger}(x-mu) phi(x-mu)
                        result[i].m_d[iSpinor] = result[i].m_d[iSpinor].AddC(__SU3Generators[i]->Mulc(x_m_mu_Gauge_element).Mul(gammaMu.MulC(x_m_mu_Fermion_element, iSpinor)));
                    }
                }

            }

            for (int i = 0; i < 8; ++i)
            {
                if (!bPartialOmega)
                {
                    //result = phi(x) - kai sum _mu result
                    result[i].Mul(_make_cuComplex(kai, 0));
                    pResultDataArray[linkIndex * 8 + i] = x_Fermion_element.SubC(result[i]);
                    if (bDDagger)
                    {
                        pResultDataArray[linkIndex * 8 + i] = gamma5.MulC(pResultDataArray[linkIndex * 8 + i]);
                    }
                }
                else
                {
                    result[i].Mul(_make_cuComplex(0, -kai));
                    pResultDataArray[linkIndex * 8 + i] = result[i];
                    if (bDDagger)
                    {
                        pResultDataArray[linkIndex * 8 + i] = gamma5.MulC(pResultDataArray[linkIndex * 8 + i]);
                    }
                }
            }
        }
    }
}

#pragma endregion

extern "C"
{
    void _cInitialFermionWilsonSquareSU3(deviceWilsonVectorSU3 *pDevicePtr, EFieldInitialType eInitialType)
    {
        preparethread;
        _kernelInitialFermionWilsonSquareSU3 << <block, threads >> > (pDevicePtr, eInitialType);
    }

    void _cDFermionWilsonSquareSU3(const deviceWilsonVectorSU3* __restrict__ pDeviceData,
                      const deviceSU3* __restrict__ pGauge,
                      deviceWilsonVectorSU3* pResultData,
                      Real kai,
                      BYTE byFieldId,
                      UBOOL bDiracChiralGamma,
                      UBOOL bDDagger)
    {
        preparethread;
        _kernelDFermionWilsonSquareSU3 << <block, threads >> > (pDeviceData, pGauge, pResultData, kai, byFieldId, bDiracChiralGamma, bDDagger);
    }

    void _cDWilsonMuSU3(const deviceWilsonVectorSU3* __restrict__ pDeviceData,
                     const deviceSU3* __restrict__ pGauge,
                     deviceWilsonVectorSU3* pResultDataArray,
                     Real kai,
                     BYTE byFieldId,
                     UBOOL bDiracChiralGamma,
                     UBOOL bDDagger,
                     UBOOL bPartialOmega)
    {
        preparethread;
        _kernelDWilsonMuSU3 << <block, threads >> > (pDeviceData, pGauge, pResultDataArray, kai, byFieldId, bDiracChiralGamma, bDiracChiralGamma, bDDagger, bPartialOmega);
    }


    void _cDotFermionWilsonSquareSU3(const deviceWilsonVectorSU3 * __restrict__ pLeft,
                  const deviceWilsonVectorSU3 * __restrict__ pRight,
                  _Complex* result)
    {
        preparethread;
        _kernelDotFermionWilsonSquareSU3 << <block, threads >> > (pLeft, pRight, result);
    }

    void _cPrintFermionWilsonSquareSU3(const deviceWilsonVectorSU3 * __restrict__ pData)
    {
        preparethread;
        _kernelPrintFermionWilsonSquareSU3 << <block, threads >> > (pData);
    }
}

CFieldFermionWilsonSquareSU3::CFieldFermionWilsonSquareSU3() : CFieldFermion()
{
    checkCudaErrors(hipMalloc((void**)&m_pDeviceData, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount));
    checkCudaErrors(hipMalloc((void**)&m_pDeviceDataCopy, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount));

    checkCudaErrors(hipMalloc((void**)&m_pForceRightVector, sizeof(deviceWilsonVectorSU3) * m_uiLinkeCount * 8));
    checkCudaErrors(hipMalloc((void**)&m_pForceRightVectorCopy, sizeof(deviceWilsonVectorSU3) * m_uiLinkeCount * 8));
    checkCudaErrors(hipMalloc((void**)&m_pForceLeftVector, sizeof(deviceWilsonVectorSU3) * m_uiLinkeCount * 8));
    checkCudaErrors(hipMalloc((void**)&m_pForceLeftVectorCopy, sizeof(deviceWilsonVectorSU3) * m_uiLinkeCount * 8));
}

CFieldFermionWilsonSquareSU3::~CFieldFermionWilsonSquareSU3()
{
    checkCudaErrors(hipFree(m_pDeviceData));
    checkCudaErrors(hipFree(m_pDeviceDataCopy));
    checkCudaErrors(hipFree(m_pForceRightVector));
    checkCudaErrors(hipFree(m_pForceRightVectorCopy));
    checkCudaErrors(hipFree(m_pForceLeftVector));
    checkCudaErrors(hipFree(m_pForceLeftVectorCopy));
}

/**
*
*/
void CFieldFermionWilsonSquareSU3::InitialField(EFieldInitialType eInitialType)
{
    _cInitialFermionWilsonSquareSU3(m_pDeviceData, eInitialType);
}

void CFieldFermionWilsonSquareSU3::DebugPrintMe() const
{
    _cPrintFermionWilsonSquareSU3(m_pDeviceData);
}

void CFieldFermionWilsonSquareSU3::CopyTo(CField* U) const
{
    if (NULL == U || EFT_FermionWilsonSquareSU3 != U->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only copy to CFieldFermionWilsonSquareSU3!"));
        return;
    }
    CFieldFermionWilsonSquareSU3 * pField = dynamic_cast<CFieldFermionWilsonSquareSU3*>(U);
    checkCudaErrors(hipMemcpy(pField->m_pDeviceData, m_pDeviceData, sizeof(deviceWilsonVectorSU3) * m_uiSiteCount, hipMemcpyDeviceToDevice));
}

/**
* generate phi by gaussian random.
* phi = D phi
*/
void CFieldFermionWilsonSquareSU3::PrepareForHMC(const CFieldGauge* pGauge)
{
    if (NULL == pGauge || EFT_GaugeSU3 != pGauge->GetFieldType())
    {
        appCrucial(_T("CFieldFermionWilsonSquareSU3 can only play with gauge SU3!"));
        return;
    }
    const CFieldGaugeSU3 * pFieldSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);

    _cInitialFermionWilsonSquareSU3(m_pDeviceDataCopy, EFIT_RandomGaussian);
    _cDFermionWilsonSquareSU3(m_pDeviceDataCopy, pFieldSU3->m_pDeviceData, m_pDeviceData, m_fKai, m_byFieldId, TRUE, FALSE);
}

void CFieldFermionWilsonSquareSU3::CalculateForce(const CFieldGauge* pGauge, CFieldGauge* pForce)
{

}

__END_NAMESPACE


//=============================================================================
// END OF FILE
//=============================================================================