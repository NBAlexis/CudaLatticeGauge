#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CIndexSquare.cu
// 
// DESCRIPTION:
// This is the class for index on square lattice
//
// REVISION:
//  [12/5/2018 nbale]
//=============================================================================
#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

/**
* For square lattice, we assume dimenssion = direction
* bdir, mu: bond direction, or link direction
* pdir, nu: plaqutte direction, the second bond direction
*
* Forward
*      c
*    +-<-+
*  d |   | b ^
*    *->-+
*      a
*  a = [site][b_dir]
*  b = [site+b_dir][p_dir]
*  c = [site+p_dir][b_dir]^-1
*  d = [site][p_dir]^-1
*      a
*    *-<-+
*  d |   | b ^
*    +->-+
*      c
*  a = [site][b_dir]^-1
*  d = [site-p_dir][p_dir]^-1
*  c = [site-p_dir][b_dir]
*  b = [site-p_dir+b_dir][p_dir]
*  
*  Here we use Staple
*    [site][p_dir], [site+p_dir][b_dir], [site+b_dir][p_dir]^1
*  + [site-p_dir][p_dir]^-1, [site-p_dir][b_dir], [site-p_dir+b_dir][p_dir]
*  The return, int2(x = linkIndex; |y| - 1 = fieldIndex, 0 if it is not boundary, sign of y is for inverse)  
*/
__device__ void CIndexSquare::_deviceGetPlaquttesAtLink(int2* retV, UINT& count, UINT& plaqutteLength, UINT uiLinkIndex, UINT st) const
{
    UINT uiDim = _DC_Dim;
    //UINT* length = pLattice->m_uiLatticeLength;
    //UINT* mult = pLattice->m_uiLatticeMultipy;

    //for square, dir should equal to dim
    assert(uiDim == _DC_Dir);

    count = 2 * (uiDim - 1);
    plaqutteLength = 4; //for square

    //For square lattice, we assume dimenssion = number of direction
    UINT uiSiteIndex = uiLinkIndex / uiDim;
    UINT uiLinkDir = uiLinkIndex % uiDim;
    UINT uiMaxDim = (0 == (st & CIndex::kTime)) ? 3 : 4;
    UINT uiMinDim = (0 == (st & CIndex::kSpace)) ? 3 : 0;

    //Note, 2D is z, t
    //3D is y, z, t
    //4D is x, y, z, t
    //so i = 4 - dim to 3, (for example, dim = 2, it is 2, 3; dim = 4, it is 0, 1, 2, 3).
    if (uiMinDim < 4 - uiDim)
    {
        uiMinDim = 4 - uiDim;
    }

    //uiLinkIndex is bdir
    //i is pdir
    UINT iListIndex = 0;
    for (int i = uiMinDim; i < uiMaxDim; ++i)
    {
        if (i != uiLinkDir)
        {
            int4 xyzt = __deviceSiteIndexToInt4(uiSiteIndex);

            //=============================================
            //add forward
            //[site][p_dir], [site+p_dir][b_dir], [site+b_dir][p_dir]^1
            retV[iListIndex].x = uiSiteIndex + i;
            retV[iListIndex].y = 1;
            ++iListIndex;

            int4 fsite3 = _deviceMoveSquareSite(xyzt, i + 1);
            uint2 fsiteIndex3 = m_pBoundaryCondition->_devcieGetMappedIndex(fsite3, xyzt);
            retV[iListIndex].x = fsiteIndex3.x + uiLinkDir;
            retV[iListIndex].y = (fsiteIndex3.y + 1);
            ++iListIndex;

            int4 fsite2 = _deviceMoveSquareSite(xyzt, uiLinkDir + 1);
            uint2 fsiteIndex2 = m_pBoundaryCondition->_devcieGetMappedIndex(fsite2, xyzt);
            retV[iListIndex].x = fsiteIndex2.x + i;
            retV[iListIndex].y = -(fsiteIndex2.y + 1);
            ++iListIndex;

            //=============================================
            //add backward
            //[site-p_dir][p_dir]^-1, [site-p_dir][b_dir], [site-p_dir+b_dir][p_dir]
            int4 bsite2 = _deviceMoveSquareSite(xyzt, -(i + 1));
            uint2 bsiteIndex2 = m_pBoundaryCondition->_devcieGetMappedIndex(bsite2, xyzt);
            retV[iListIndex].x = bsiteIndex2.x + i;
            retV[iListIndex].y = -(bsiteIndex2.y + 1);
            ++iListIndex;

            retV[iListIndex].x = bsiteIndex2.x + uiLinkDir;
            retV[iListIndex].y = bsiteIndex2.y + 1;
            ++iListIndex;

            int4 bsite4 = _deviceMoveSquareSite(bsite2, uiLinkDir + 1);
            uint2 bsiteIndex4 = m_pBoundaryCondition->_devcieGetMappedIndex(bsite4, bsite2);
            retV[iListIndex].x = bsiteIndex4.x + i;
            retV[iListIndex].y = bsiteIndex2.y + 1;
            ++iListIndex;
        }
    }
}


__END_NAMESPACE


//=============================================================================
// END OF FILE
//=============================================================================