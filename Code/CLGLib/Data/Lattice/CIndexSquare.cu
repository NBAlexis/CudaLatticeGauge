#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CIndexSquare.cu
// 
// DESCRIPTION:
// This is the class for index on square lattice
//
// REVISION:
//  [12/5/2018 nbale]
//=============================================================================
#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

/**
* For square lattice, we assume dimenssion = direction
* bdir, mu: bond direction, or link direction
* pdir, nu: plaqutte direction, the second bond direction
*
* Forward
*      c
*    +-<-+
*  d |   | b ^
*    *->-+
*      a
*  a = [site][b_dir]
*  b = [site+b_dir][p_dir]
*  c = [site+p_dir][b_dir]^-1
*  d = [site][p_dir]^-1
*      a
*    *-<-+
*  d |   | b ^
*    +->-+
*      c
*  a = [site][b_dir]^-1
*  d = [site-p_dir][p_dir]^-1
*  c = [site-p_dir][b_dir]
*  b = [site-p_dir+b_dir][p_dir]
*  
*  Here we use Staple
*    [site][p_dir], [site+p_dir][b_dir], [site+b_dir][p_dir]^1
*  + [site-p_dir][p_dir]^-1, [site-p_dir][b_dir], [site-p_dir+b_dir][p_dir]
*  The return, int2(x = linkIndex; |y| - 1 = fieldIndex, 0 if it is not boundary, sign of y is for inverse)  
*/
__device__ void CIndexSquare::_deviceGetPlaquttesAtLink(SIndex* retV, UINT& count, UINT& plaqutteLength, UINT uiLinkIndex, UINT st) const
{
    UINT uiDim = _DC_Dim;

    //for square, dir should equal to dim
    assert(uiDim == _DC_Dir);

    count = 2 * (uiDim - 1);
    plaqutteLength = 4; //for square

    //For square lattice, we assume dimenssion = number of direction
    UINT uiSiteIndex = uiLinkIndex / uiDim;
    UINT uiLinkDir = uiLinkIndex % uiDim;
    UINT uiMaxDim = (0 == (st & CIndex::kTime)) ? 3 : 4;
    UINT uiMinDim = (0 == (st & CIndex::kSpace)) ? 3 : 0;

    //Note, 2D is z, t
    //3D is y, z, t
    //4D is x, y, z, t
    //so i = 4 - dim to 3, (for example, dim = 2, it is 2, 3; dim = 4, it is 0, 1, 2, 3).
    if (uiMinDim < 4 - uiDim)
    {
        uiMinDim = 4 - uiDim;
    }

    //uiLinkIndex is bdir
    //i is pdir
    UINT iListIndex = 0;
    for (int i = uiMinDim; i < uiMaxDim; ++i)
    {
        if (i != uiLinkDir)
        {
            int4 xyzt = __deviceSiteIndexToInt4(uiSiteIndex);

            //=============================================
            //add forward
            //[site][p_dir], [site+p_dir][b_dir], [site+b_dir][p_dir]^1
            retV[iListIndex] = SIndex(uiSiteIndex, i);
            ++iListIndex;

            int4 fsite = _deviceMoveSquareSite(xyzt, i + 1);
            retV[iListIndex] = m_pBoundaryCondition->_devcieGetMappedIndex(fsite, xyzt);
            retV[iListIndex].m_byDir = uiLinkDir;
            ++iListIndex;

            fsite = _deviceMoveSquareSite(xyzt, uiLinkDir + 1);
            retV[iListIndex] = m_pBoundaryCondition->_devcieGetMappedIndex(fsite, xyzt);
            retV[iListIndex].m_byDir = i;
            retV[iListIndex].m_byTag = _kDagger;
            ++iListIndex;

            //=============================================
            //add backward
            //[site-p_dir][p_dir]^-1, [site-p_dir][b_dir], [site-p_dir+b_dir][p_dir]
            int4 bsite2 = _deviceMoveSquareSite(xyzt, -(i + 1));
            retV[iListIndex] = m_pBoundaryCondition->_devcieGetMappedIndex(bsite2, xyzt);
            retV[iListIndex].m_byDir = i;
            retV[iListIndex].m_byTag = _kDagger;
            ++iListIndex;

            retV[iListIndex] = SIndex(retV[iListIndex - 1].m_uiSiteIndex, uiLinkDir);
            ++iListIndex;

            fsite = _deviceMoveSquareSite(bsite2, uiLinkDir + 1);
            retV[iListIndex] = m_pBoundaryCondition->_devcieGetMappedIndex(fsite, bsite2);
            retV[iListIndex].m_byDir = i;
            ++iListIndex;
        }
    }

    assert(count * 3 == iListIndex);
}

__device__ void CIndexSquare::_deviceGetPlaquttesAtSite(SIndex* retV, UINT& count, UINT& plaqutteLength, UINT uiSiteIndex, UINT st) const
{
    UINT uiDim = _DC_Dim;

    //for square, dir should equal to dim
    assert(uiDim == _DC_Dir);

    count = uiDim * (uiDim - 1) / 2;
    plaqutteLength = 4; //for square

    UINT uiMaxDim = (0 == (st & CIndex::kTime)) ? 3 : 4;
    
    UINT iListIndex = 0;
    for (UINT uiLink = 0; uiLink < uiMaxDim; ++uiLink)
    {
        UINT uiMinDim = (0 == (st & CIndex::kSpace)) ? 3 : uiLink + 1;
        for (UINT uiPlaq = uiMinDim; uiPlaq < uiMaxDim; ++uiPlaq)
        {
            int4 xyzt = __deviceSiteIndexToInt4(uiSiteIndex);

            retV[iListIndex] = SIndex(uiSiteIndex, uiLink);
            ++iListIndex;

            int4 fsite1 = _deviceMoveSquareSite(xyzt, uiLink + 1);
            retV[iListIndex] = m_pBoundaryCondition->_devcieGetMappedIndex(fsite1, xyzt);
            retV[iListIndex].m_byDir = uiPlaq;
            ++iListIndex;

            int4 fsite2 = _deviceMoveSquareSite(xyzt, uiPlaq + 1);
            retV[iListIndex] = m_pBoundaryCondition->_devcieGetMappedIndex(fsite2, xyzt);
            retV[iListIndex].m_byDir = uiLink;
            retV[iListIndex].m_byTag = _kDagger;
            ++iListIndex;

            retV[iListIndex] = SIndex(uiSiteIndex, uiPlaq);
            retV[iListIndex].m_byTag = _kDagger;
            ++iListIndex;
        }
    }

    assert(count * 4 == iListIndex);
}

/**
* virtual, cannot inline
*/
__device__ SIndex CIndexSquare::_deviceFermionIndexWalk(BYTE uiFieldId, UINT uiSiteIndex, INT iWalkDir) const
{
    int4 fromSite = __deviceSiteIndexToInt4(uiSiteIndex);
    int4 siteInt4 = _deviceMoveSquareSite(fromSite, iWalkDir);
    return m_pBoundaryCondition->_devcieGetFermionMappedIndex(uiFieldId, siteInt4, fromSite);
}

/**
* virtual, cannot inline
*/
__device__ SIndex CIndexSquare::_deviceGaugeIndexWalk(UINT uiSiteIndex, INT iWalkDir) const
{
    int4 fromSite = __deviceSiteIndexToInt4(uiSiteIndex);
    int4 siteInt4 = _deviceMoveSquareSite(fromSite, iWalkDir);
    return m_pBoundaryCondition->_devcieGetMappedIndex(siteInt4, fromSite);
}

__END_NAMESPACE


//=============================================================================
// END OF FILE
//=============================================================================