#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CIndexSquare.cu
// 
// DESCRIPTION:
// This is the class for index on square lattice
//
// REVISION:
//  [12/5/2018 nbale]
//=============================================================================
#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

/**
* For square lattice, we assume dimenssion = direction
* bdir, mu: bond direction, or link direction
* pdir, nu: plaqutte direction, the second bond direction
*
* Forward
*      c
*    +-<-+
*  d |   | b ^
*    *->-+
*      a
*  a = [site][b_dir]
*  b = [site+b_dir][p_dir]
*  c = [site+p_dir][b_dir]^-1
*  d = [site][p_dir]^-1
*      a
*    *-<-+
*  d |   | b ^
*    +->-+
*      c
*  a = [site][b_dir]^-1
*  d = [site-p_dir][p_dir]^-1
*  c = [site-p_dir][b_dir]
*  b = [site-p_dir+b_dir][p_dir]
*  
*  Here we use Staple
*    [site][p_dir], [site+p_dir][b_dir], [site+b_dir][p_dir]^1
*  + [site-p_dir][p_dir]^-1, [site-p_dir][b_dir], [site-p_dir+b_dir][p_dir]
*  The return, int2(x = linkIndex; |y| - 1 = fieldIndex, 0 if it is not boundary, sign of y is for inverse)  
*/
__device__ int2* CIndexSquare::GetPlaquttesAtLink(UINT& count, UINT& plaqutteLength, UINT uiDim, UINT uiLinkIndex, const UINT* length, const UINT* mult, UINT st)
{
    int2* retV;
    count = 2 * (uiDim - 1);
    plaqutteLength = 4; //for square
    hipMalloc((void **)&retV, sizeof(int2) * count * (plaqutteLength - 1));

    //For square lattice, we assume dimenssion = number of direction
    UINT uiSiteIndex = uiLinkIndex / uiDim;
    UINT uiLinkDir = uiLinkIndex % uiDim;
    UINT uiMaxDim = (0 == (st & CIndex::kTime)) ? 3 : 4;
    UINT uiMinDim = (0 == (st & CIndex::kSpace)) ? 3 : 0;

    //Note, 2D is z, t
    //3D is y, z, t
    //4D is x, y, z, t
    //so i = 4 - dim to 3, (for example, dim = 2, it is 2, 3; dim = 4, it is 0, 1, 2, 3).
    if (uiMinDim < 4 - uiDim)
    {
        uiMinDim = 4 - uiDim;
    }

    //uiLinkIndex is bdir
    //i is pdir
    UINT elementCount = mult[4]; //for example, for SU3 it is 9
    UINT iListIndex = 0;
    for (int i = uiMinDim; i < uiMaxDim; ++i)
    {
        if (i != uiLinkDir)
        {
            int4 xyzt = GetXYZTSquare(uiSiteIndex, mult);

            //=============================================
            //add forward
            //[site][p_dir], [site+p_dir][b_dir], [site+b_dir][p_dir]^1
            retV[iListIndex].x = uiSiteIndex + i * elementCount;
            retV[iListIndex].y = 1;
            ++iListIndex;

            int4 fsite3 = MoveSquareSite(xyzt, i + 1);
            uint2 fsiteIndex3 = m_pBoundaryCondition->GetMappedIndex(fsite3, xyzt, length, mult);
            retV[iListIndex].x = fsiteIndex3.x + uiLinkDir * elementCount;
            retV[iListIndex].y = (fsiteIndex3.y + 1);
            ++iListIndex;

            int4 fsite2 = MoveSquareSite(xyzt, uiLinkDir + 1);
            uint2 fsiteIndex2 = m_pBoundaryCondition->GetMappedIndex(fsite2, xyzt, length, mult);
            retV[iListIndex].x = fsiteIndex2.x + i * elementCount;
            retV[iListIndex].y = -(fsiteIndex2.y + 1);
            ++iListIndex;

            //=============================================
            //add backward
            //[site-p_dir][p_dir]^-1, [site-p_dir][b_dir], [site-p_dir+b_dir][p_dir]
            int4 bsite2 = MoveSquareSite(xyzt, -(i + 1));
            uint2 bsiteIndex2 = m_pBoundaryCondition->GetMappedIndex(bsite2, xyzt, length, mult);
            retV[iListIndex].x = bsiteIndex2.x + i * elementCount;
            retV[iListIndex].y = -(bsiteIndex2.y + 1);
            ++iListIndex;

            retV[iListIndex].x = bsiteIndex2.x + uiLinkDir * elementCount;
            retV[iListIndex].y = bsiteIndex2.y + 1;
            ++iListIndex;

            int4 bsite4 = MoveSquareSite(bsite2, uiLinkDir + 1);
            uint2 bsiteIndex4 = m_pBoundaryCondition->GetMappedIndex(bsite4, bsite2, length, mult);
            retV[iListIndex].x = bsiteIndex4.x + i * elementCount;
            retV[iListIndex].y = bsiteIndex2.y + 1;
            ++iListIndex;
        }
    }
    return retV;
}

__END_NAMESPACE


//=============================================================================
// END OF FILE
//=============================================================================