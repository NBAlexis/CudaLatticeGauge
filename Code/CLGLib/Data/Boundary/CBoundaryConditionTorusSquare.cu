#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CBoundaryConditionTorusSquare.cpp
// 
// DESCRIPTION:
// This is the periodic boundary condition
//
// REVISION:
//  [12/5/2018 nbale]
//=============================================================================

#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

__CLGIMPLEMENT_CLASS(CBoundaryConditionTorusSquare)

#pragma region kernels

__global__ void _CLG_LAUNCH_BOUND
_kernalBakeEdgeTorusBoundary(
    SSmallInt4 bc, 
    SIndex* pDeviceData, 
    uint3 mods)
{
    UINT idxAll = threadIdx.x + blockDim.x * blockIdx.x;
    SSmallInt4 coord;
    coord.x = static_cast<SBYTE>(idxAll / mods.x);
    coord.y = static_cast<SBYTE>((idxAll % mods.x) / mods.y);
    coord.z = static_cast<SBYTE>((idxAll % mods.y) / mods.z);
    coord.w = static_cast<SBYTE>(idxAll % mods.z);

    SSmallInt4 realCoord = coord;
    realCoord.x -= CIndexData::kCacheIndexEdge;
    realCoord.y -= CIndexData::kCacheIndexEdge;
    realCoord.z -= CIndexData::kCacheIndexEdge;
    realCoord.w -= CIndexData::kCacheIndexEdge;

    SBYTE signchange = 1;
    for (UINT uiDir = 4 - _DC_Dir; uiDir < _DC_Dir; ++uiDir)
    {
        if (realCoord.m_byData4[uiDir] < 0)
        {
            realCoord.m_byData4[uiDir] = realCoord.m_byData4[uiDir] + _constIntegers[ECI_Lx + uiDir];
            signchange = signchange * bc.m_byData4[uiDir];
        }
        else if (realCoord.m_byData4[uiDir] >= _constIntegers[ECI_Lx + uiDir])
        {
            realCoord.m_byData4[uiDir] = realCoord.m_byData4[uiDir] - _constIntegers[ECI_Lx + uiDir];
            signchange = signchange * bc.m_byData4[uiDir];
        }
    }

    UINT uiSiteIndex = _deviceGetSiteIndex(realCoord);
    pDeviceData[idxAll] = SIndex(uiSiteIndex);
    pDeviceData[idxAll].m_byTag = signchange < 0 ? _kDaggerOrOpposite : 0;
}

#pragma endregion

CBoundaryConditionTorusSquare::CBoundaryConditionTorusSquare() : CBoundaryCondition()
{
    for (UINT i = 0; i < _kMaxFieldCount; ++i)
    {
        m_FieldBC[i].x = 1;
        m_FieldBC[i].y = 1;
        m_FieldBC[i].z = 1;
        m_FieldBC[i].w = -1;
    }
    m_FieldBC[0].w = 1;
    m_FieldBC[1].w = 1;
}

void CBoundaryConditionTorusSquare::SetFieldSpecificBc(BYTE byFieldId, const SBoundCondition& bc)
{
    assert(byFieldId < _kMaxFieldCount);
    m_FieldBC[byFieldId] = bc.m_sPeriodic;
}

void CBoundaryConditionTorusSquare::BakeEdgePoints(BYTE byFieldId, SIndex* deviceBuffer) const
{
    uint4 biggerLattice;
    biggerLattice.x = _HC_Lx + 2 * CIndexData::kCacheIndexEdge;
    biggerLattice.y = _HC_Ly + 2 * CIndexData::kCacheIndexEdge;
    biggerLattice.z = _HC_Lz + 2 * CIndexData::kCacheIndexEdge;
    biggerLattice.w = _HC_Lt + 2 * CIndexData::kCacheIndexEdge;
    uint3 biggerLatticeMod;

    UINT uiVolumn = biggerLattice.x * biggerLattice.y * biggerLattice.z * biggerLattice.w;
    UINT threadPerSite = CIndexSquare::GetDecompose(uiVolumn);
    dim3 threads(threadPerSite, 1, 1);
    dim3 blocks(uiVolumn / threadPerSite, 1, 1);
    biggerLatticeMod.x = biggerLattice.y * biggerLattice.z * biggerLattice.w;
    biggerLatticeMod.y = biggerLattice.z * biggerLattice.w;
    biggerLatticeMod.z = biggerLattice.w;

    _kernalBakeEdgeTorusBoundary << <blocks, threads >> > (m_FieldBC[byFieldId], deviceBuffer, biggerLatticeMod);
}

__END_NAMESPACE


//=============================================================================
// END OF FILE
//=============================================================================
