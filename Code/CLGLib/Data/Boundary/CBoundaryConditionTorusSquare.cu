#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CBoundaryConditionTorusSquare.cpp
// 
// DESCRIPTION:
// This is the periodic boundary condition
//
// REVISION:
//  [12/5/2018 nbale]
//=============================================================================

#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

__CLGIMPLEMENT_CLASS(CBoundaryConditionTorusSquare)

#pragma region kernels

__global__ void _CLG_LAUNCH_BOUND
_kernalBakeEdgeTorusBoundary(
    SSmallInt4 bc, 
    const SSmallInt4* __restrict__ pMapping,
    SIndex* pDeviceData, 
    uint3 mods)
{
    UINT idxAll = threadIdx.x + blockDim.x * blockIdx.x;
    SSmallInt4 realCoord(pMapping[idxAll]);
    //realCoord.x = static_cast<SBYTE>(idxAll / mods.x) - CIndexData::kCacheIndexEdge;
    //realCoord.y = static_cast<SBYTE>((idxAll % mods.x) / mods.y) - CIndexData::kCacheIndexEdge;
    //realCoord.z = static_cast<SBYTE>((idxAll % mods.y) / mods.z) - CIndexData::kCacheIndexEdge;
    //realCoord.w = static_cast<SBYTE>(idxAll % mods.z) - CIndexData::kCacheIndexEdge;

    //UBOOL bDebug = FALSE;
    //if (realCoord.m_byData4[3] < 0 || realCoord.m_byData4[3] > 1)
    //{
    //    bDebug = TRUE;
    //}
    //SSmallInt4 old(realCoord);

    SBYTE signchange = 1;
    for (UINT uiDir = 0; uiDir < 4; ++uiDir)
    {
        while (realCoord.m_byData4[uiDir] < 0)
        {
            realCoord.m_byData4[uiDir] = realCoord.m_byData4[uiDir] + _constIntegers[ECI_Lx + uiDir];
            signchange = signchange * bc.m_byData4[uiDir];
        }

        while (realCoord.m_byData4[uiDir] >= _constIntegers[ECI_Lx + uiDir])
        {
            realCoord.m_byData4[uiDir] = realCoord.m_byData4[uiDir] - _constIntegers[ECI_Lx + uiDir];
            signchange = signchange * bc.m_byData4[uiDir];
        }
    }

    //if (bDebug)
    //{
    //    printf("%d %d %d %d to %d %d %d %d\n", old.x, old.y, old.z, old.w, realCoord.x, realCoord.y, realCoord.z, realCoord.w);
    //}

    const UINT uiSiteIndex = _deviceGetSiteIndex(realCoord);
    pDeviceData[idxAll] = SIndex(uiSiteIndex);
    pDeviceData[idxAll].m_byTag = signchange < 0 ? _kDaggerOrOpposite : 0;
}

/**
* Nothing to write, just initial as 0
*/
//__global__ void _CLG_LAUNCH_BOUND
//_kernalBakeBondInfo_Torus(BYTE* pDeviceData)
//{
//    UINT idxAll = threadIdx.x + blockDim.x * blockIdx.x;
//    for (UINT i = 0; i < _DC_Dir; ++i)
//    {
//        pDeviceData[idxAll * _DC_Dir + i] = 0;
//    }
//}

__global__ void _CLG_LAUNCH_BOUND
_kernalBakeBoundGlueTorusBoundary(
    SSmallInt4 bc,
    const SSmallInt4* __restrict__ pMapping,
    SIndex* pDeviceData,
    uint3 mods)
{
    UINT idxAll = threadIdx.x + blockDim.x * blockIdx.x;
    SSmallInt4 realCoord(pMapping[idxAll]);
    //realCoord.x = static_cast<SBYTE>(idxAll / mods.x) - CIndexData::kCacheIndexEdge;
    //realCoord.y = static_cast<SBYTE>((idxAll % mods.x) / mods.y) - CIndexData::kCacheIndexEdge;
    //realCoord.z = static_cast<SBYTE>((idxAll % mods.y) / mods.z) - CIndexData::kCacheIndexEdge;
    //realCoord.w = static_cast<SBYTE>(idxAll % mods.z) - CIndexData::kCacheIndexEdge;

    SBYTE signchange = 1;
    for (UINT uiDir = 0; uiDir < 4; ++uiDir)
    {
        while (realCoord.m_byData4[uiDir] < 0)
        {
            realCoord.m_byData4[uiDir] = realCoord.m_byData4[uiDir] + _constIntegers[ECI_Lx + uiDir];
            signchange = signchange * bc.m_byData4[uiDir];
        }

        while (realCoord.m_byData4[uiDir] >= _constIntegers[ECI_Lx + uiDir])
        {
            realCoord.m_byData4[uiDir] = realCoord.m_byData4[uiDir] - _constIntegers[ECI_Lx + uiDir];
            signchange = signchange * bc.m_byData4[uiDir];
        }
    }

    const UINT uiSiteIndex = _deviceGetSiteIndex(realCoord);
    for (BYTE byDir = 0; byDir < _DC_Dir; ++byDir)
    {
        pDeviceData[idxAll * _DC_Dir + byDir] = SIndex(uiSiteIndex);
        pDeviceData[idxAll * _DC_Dir + byDir].m_byDir = byDir;

        //Bound should never have anti-periodic boundary condition?
        pDeviceData[idxAll * _DC_Dir + byDir].m_byTag = signchange < 0 ? _kDaggerOrOpposite : 0;
    }
}

#pragma endregion

CBoundaryConditionTorusSquare::CBoundaryConditionTorusSquare() : CBoundaryCondition()
{
    for (UINT i = 0; i < kMaxFieldCount; ++i)
    {
        m_FieldBC[i].x = 1;
        m_FieldBC[i].y = 1;
        m_FieldBC[i].z = 1;
        m_FieldBC[i].w = -1;
    }
    m_FieldBC[0].w = 1;
    m_FieldBC[1].w = 1;
}

//void CBoundaryConditionTorusSquare::SetFieldSpecificBc(BYTE byFieldId, const SBoundCondition& bc)
//{
//    assert(byFieldId < kMaxFieldCount);
//    m_FieldBC[byFieldId] = bc.m_sPeriodic;
//}

void CBoundaryConditionTorusSquare::BakeEdgePoints(BYTE byFieldId, const SSmallInt4* deviceMappingTable, SIndex* deviceBuffer) const
{
    uint4 biggerLattice;
    biggerLattice.x = _HC_Lx + 2 * CIndexData::kCacheIndexEdge;
    biggerLattice.y = _HC_Ly + 2 * CIndexData::kCacheIndexEdge;
    biggerLattice.z = _HC_Lz + 2 * CIndexData::kCacheIndexEdge;
    biggerLattice.w = _HC_Lt + 2 * CIndexData::kCacheIndexEdge;
    uint3 biggerLatticeMod;

    const UINT uiVolumn = biggerLattice.x * biggerLattice.y * biggerLattice.z * biggerLattice.w;
    const UINT threadPerSite = CIndexSquare::GetDecompose(uiVolumn);
    dim3 threads(threadPerSite, 1, 1);
    dim3 blocks(uiVolumn / threadPerSite, 1, 1);
    biggerLatticeMod.x = biggerLattice.y * biggerLattice.z * biggerLattice.w;
    biggerLatticeMod.y = biggerLattice.z * biggerLattice.w;
    biggerLatticeMod.z = biggerLattice.w;

    _kernalBakeEdgeTorusBoundary << <blocks, threads >> > (m_FieldBC[byFieldId], deviceMappingTable, deviceBuffer, biggerLatticeMod);
}

//void CBoundaryConditionTorusSquare::BakeBondInfo(const SSmallInt4*, BYTE* deviceTable, BYTE byFieldId) const
//{
//    uint4 biggerLattice;
//    biggerLattice.x = _HC_Lx + 2 * CIndexData::kCacheIndexEdge;
//    biggerLattice.y = _HC_Ly + 2 * CIndexData::kCacheIndexEdge;
//    biggerLattice.z = _HC_Lz + 2 * CIndexData::kCacheIndexEdge;
//    biggerLattice.w = _HC_Lt + 2 * CIndexData::kCacheIndexEdge;
//
//    const UINT uiVolumn = biggerLattice.x * biggerLattice.y * biggerLattice.z * biggerLattice.w;
//    const UINT threadPerSite = CIndexSquare::GetDecompose(uiVolumn);
//    dim3 threads(threadPerSite, 1, 1);
//    dim3 blocks(uiVolumn / threadPerSite, 1, 1);
//
//    _kernalBakeBondInfo_Torus << <blocks, threads >> > (deviceTable);
//}

void CBoundaryConditionTorusSquare::BakeBondGlue(BYTE byFieldId, const SSmallInt4* deviceMappingTable, SIndex* deviceBuffer) const
{
    uint4 biggerLattice;
    biggerLattice.x = _HC_Lx + 2 * CIndexData::kCacheIndexEdge;
    biggerLattice.y = _HC_Ly + 2 * CIndexData::kCacheIndexEdge;
    biggerLattice.z = _HC_Lz + 2 * CIndexData::kCacheIndexEdge;
    biggerLattice.w = _HC_Lt + 2 * CIndexData::kCacheIndexEdge;
    uint3 biggerLatticeMod;

    const UINT uiVolumn = biggerLattice.x * biggerLattice.y * biggerLattice.z * biggerLattice.w;
    const UINT threadPerSite = CIndexSquare::GetDecompose(uiVolumn);
    dim3 threads(threadPerSite, 1, 1);
    dim3 blocks(uiVolumn / threadPerSite, 1, 1);
    biggerLatticeMod.x = biggerLattice.y * biggerLattice.z * biggerLattice.w;
    biggerLatticeMod.y = biggerLattice.z * biggerLattice.w;
    biggerLatticeMod.z = biggerLattice.w;

    _kernalBakeBoundGlueTorusBoundary << <blocks, threads >> > (m_FieldBC[byFieldId], deviceMappingTable, deviceBuffer, biggerLatticeMod);
}

__END_NAMESPACE


//=============================================================================
// END OF FILE
//=============================================================================
