#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CActionGaugePlaquetteRotating.cu
// 
// DESCRIPTION:
// This is the class for rotating su3
//
// REVISION:
//  [05/07/2019 nbale]
//=============================================================================
#include "CLGLib_Private.h"


__BEGIN_NAMESPACE

__CLGIMPLEMENT_CLASS(CActionGaugePlaquetteRotating)


#pragma region kernels

/**
* This is slower, just for testing
* directly calculate Retr[1 - \hat{U}]
*/
__global__ void _CLG_LAUNCH_BOUND
_kernelAdd4PlaqutteTermSU3_Test(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    Real betaOverN, Real fOmegaSq, 
    Real* results)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    if (__idx->m_pDeviceIndexPositionToSIndex[1][uiBigIdx].IsDirichlet())
    {
        results[uiSiteIndex] = F(0.0);
        return;
    }

    Real fXSq = (sSite4.x - sCenterSite.x);
    fXSq = fXSq * fXSq;
    Real fYSq = (sSite4.y - sCenterSite.y);
    fYSq = fYSq * fYSq;

    //======================================================
    //4-plaqutte terms
    //Omega^2 x^2 Retr[1 - U_1,4]
    Real fU14 = fOmegaSq * fXSq * _device4PlaqutteTerm(pDeviceData, 0, 3, uiBigIdx);

    //Omega^2 y^2 Retr[1 - U_2,4]
    Real fU24 = fOmegaSq * fYSq * _device4PlaqutteTerm(pDeviceData, 1, 3, uiBigIdx);

    //Omega^2 (x^2 + y^2) Retr[1 - U_3,4]
    Real fU34 = fOmegaSq * (fXSq + fYSq) * _device4PlaqutteTerm(pDeviceData, 2, 3, uiBigIdx);

    results[uiSiteIndex] = (fU14 + fU24 + fU34) * betaOverN;
}

/**
* Using plaqutte and (f(n)+f(n+mu)+f(n+nu)+f(n+mu+nu))/4 
*/
__global__ void _CLG_LAUNCH_BOUND
_kernelAdd4PlaqutteTermSU3(
    const deviceSU3 * __restrict__ pDeviceData,
    const SIndex* __restrict__ pCachedPlaqutte,
    SSmallInt4 sCenterSite,
    Real betaOverN, Real fOmegaSq,
    Real* results)
{
    //intokernalInt4;
    SSmallInt4 sSite4;
    UINT _ixy = (threadIdx.x + blockIdx.x * blockDim.x);
    UINT _iz_idx = (threadIdx.y + blockIdx.y * blockDim.y);

    sSite4.x = static_cast<SBYTE> (_ixy / _DC_Lx);
    sSite4.y = static_cast<SBYTE> (_ixy % _DC_Lx);
    sSite4.z = static_cast<SBYTE>(_iz_idx / 3);
    sSite4.w = static_cast<SBYTE>(threadIdx.z + blockIdx.z * blockDim.z);
    UINT uiSiteIndex = _ixy * _DC_GridDimZT + sSite4.z * _DC_Lt + sSite4.w;
    BYTE idx0 = _iz_idx % 3;

    UINT uiN = __idx->_deviceGetBigIndex(sSite4);
    UINT plaqLength = __idx->m_pSmallData[CIndexData::kPlaqLengthIdx];
    UINT plaqCountAll = __idx->m_pSmallData[CIndexData::kPlaqPerSiteIdx] * plaqLength;
    
    //i=0: 12
    //  1: 13
    //  2: 14
    //  3: 23
    //  4: 24
    //  5: 34
    //0->2, 1->4, 2->5
    BYTE idx = (2 == idx0) ? 5 : ((idx0 + 1) * 2);

    //Real resThisThread = F(0.0);

    //========================================
    //find plaqutte 1-4, or 2-4, or 3-4
    SIndex first = pCachedPlaqutte[idx * plaqLength + uiSiteIndex * plaqCountAll];
    deviceSU3 toAdd(_deviceGetGaugeBCSU3(pDeviceData, first));
    if (first.NeedToDagger())
    {
        toAdd.Dagger();
    }
    for (BYTE j = 1; j < plaqLength; ++j)
    {
        first = pCachedPlaqutte[idx * plaqLength + j + uiSiteIndex * plaqCountAll];
        deviceSU3 toMul(_deviceGetGaugeBCSU3(pDeviceData, first));
        if (first.NeedToDagger())
        {
            toAdd.MulDagger(toMul);
        }
        else
        {
            toAdd.Mul(toMul);
        }
    }

    atomicAdd(&results[uiSiteIndex], betaOverN * fOmegaSq * (F(3.0) - toAdd.ReTr()) * _deviceFi(sCenterSite, uiN, idx0, idx0, 3));

}


/**
* Split into 3 functions to avoid max-register problem
*/
__global__ void _CLG_LAUNCH_BOUND
_kernelAddChairTermSU3_Term12(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    Real betaOverN, Real fOmega,
    Real* results)
{
    intokernalInt4;

    UINT uiN = __idx->_deviceGetBigIndex(sSite4);

    if (__idx->m_pDeviceIndexPositionToSIndex[1][uiN].IsDirichlet())
    {
        results[uiSiteIndex] = F(0.0);
        return;
    }

    betaOverN = F(0.125) * betaOverN;
    Real fXOmega = (sSite4.x - sCenterSite.x) * fOmega;

    //===============
    //+x Omega V412
    Real fV412 = fXOmega * _deviceChairTerm(pDeviceData, 3, 0, 1, uiN);

    //===============
    //+x Omega V432
    Real fV432 = fXOmega * _deviceChairTerm(pDeviceData, 3, 2, 1, uiN);

    results[uiSiteIndex] = (fV412 + fV432) * betaOverN;
}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddChairTermSU3_Term34(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    Real betaOverN, Real fOmega,
    Real* results)
{
    intokernalInt4;

    UINT uiN = __idx->_deviceGetBigIndex(sSite4);

    if (__idx->m_pDeviceIndexPositionToSIndex[1][uiN].IsDirichlet())
    {
        results[uiSiteIndex] = F(0.0);
        return;
    }

    betaOverN = F(0.125) * betaOverN;
    Real fYOmega = -(sSite4.y - sCenterSite.y) * fOmega;

    //===============
    //-y Omega V421
    Real fV421 = fYOmega * _deviceChairTerm(pDeviceData, 3, 1, 0, uiN);

    //===============
    //-y Omega V431
    Real fV431 = fYOmega * _deviceChairTerm(pDeviceData, 3, 2, 0, uiN);

    results[uiSiteIndex] = (fV421 + fV431) * betaOverN;
}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddChairTermSU3_Term5(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    Real betaOverN, Real fOmegaSq,
    Real* results)
{
    intokernalInt4;

    UINT uiN = __idx->_deviceGetBigIndex(sSite4);

    if (__idx->m_pDeviceIndexPositionToSIndex[1][uiN].IsDirichlet())
    {
        results[uiSiteIndex] = F(0.0);
        return;
    }

    betaOverN = F(0.125) * betaOverN;
    Real fXYOmega2 = (sSite4.x - sCenterSite.x) * (sSite4.y - sCenterSite.y) * fOmegaSq;

    //===============
    //+Omega^2 xy V142
    Real fV142 = fXYOmega2 * _deviceChairTerm(pDeviceData, 0, 3, 1, uiN);

    results[uiSiteIndex] = fV142 * betaOverN;
}

/**
* 
*/
__global__ void _CLG_LAUNCH_BOUND
_kernelAddForce4PlaqutteTermSU3_XYZ(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmegaSq)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(-0.5);
    deviceSU3 plaqSum = deviceSU3::makeSU3Zero();

    for (UINT idir = 0; idir < 3; ++idir)
    {
        if (__idx->_deviceIsBondOnSurface(uiBigIdx, idir))
        {
            continue;
        }
        UINT linkIndex = _deviceGetLinkIndex(uiSiteIndex, idir);

        //mu = idir, nu = 4, i = mu
        deviceSU3 stap(_deviceStapleTerm123(pDeviceData, sCenterSite, sSite4, fOmegaSq, uiBigIdx, idir, 3, idir));
        deviceSU3 force(pDeviceData[linkIndex]);
        force.MulDagger(stap);
        force.Ta();
        force.MulReal(betaOverN);
        pForceData[linkIndex].Add(force);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForce4PlaqutteTermSU3_T(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmegaSq)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);
    //UINT uiDir = _DC_Dir;

    betaOverN = betaOverN * F(-0.5);
    deviceSU3 plaqSum = deviceSU3::makeSU3Zero();

    BYTE idir = 3;
    if (__idx->_deviceIsBondOnSurface(uiBigIdx, idir))
    {
        return;
    }
    UINT linkIndex = _deviceGetLinkIndex(uiSiteIndex, idir);

    //mu = idir, nu = i = sum _1-3
    deviceSU3 stap(_deviceStapleTerm4(pDeviceData, sCenterSite, sSite4, fOmegaSq, uiBigIdx, idir, 0));
    stap.Add(_deviceStapleTerm4(pDeviceData, sCenterSite, sSite4, fOmegaSq, uiBigIdx, idir, 1));
    stap.Add(_deviceStapleTerm123(pDeviceData, sCenterSite, sSite4, fOmegaSq, uiBigIdx, idir, 2, 2));
    deviceSU3 force(pDeviceData[linkIndex]);
    force.MulDagger(stap);
    force.Ta();
    force.MulReal(betaOverN);
    pForceData[linkIndex].Add(force);
}

/**
* Split to 15 functions to avoid max-regcount
*/
__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term1_1(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmega)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmega * F(0.125);

    //===============
    //+x Omega V412
    //add force for mu=4
    UINT uiLink4 = _deviceGetLinkIndex(uiSiteIndex, 3);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 3))
    {
        deviceSU3 staple_term1_4 = _deviceStapleChairTerm1(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            3, 0, 1, 0);
        deviceSU3 force4(pDeviceData[uiLink4]);
        force4.MulDagger(staple_term1_4);
        force4.Ta();
        force4.MulReal(betaOverN);
        pForceData[uiLink4].Add(force4);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term1_2(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmega)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmega * F(0.125);

    //===============
    //+x Omega V412
    //add force for mu=4
    UINT uiLink2 = _deviceGetLinkIndex(uiSiteIndex, 1);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 1))
    {
        deviceSU3 staple_term1_2 = _deviceStapleChairTerm1(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            1, 0, 3, 0);
        deviceSU3 force2(pDeviceData[uiLink2]);
        force2.MulDagger(staple_term1_2);
        force2.Ta();
        force2.MulReal(betaOverN);
        pForceData[uiLink2].Add(force2);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term1_3(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmega)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmega * F(0.125);

    //===============
    //+x Omega V412
    //add force for mu=4
    UINT uiLink1 = _deviceGetLinkIndex(uiSiteIndex, 0);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 0))
    {
        deviceSU3 staple_term1_1 = _deviceStapleChairTerm2(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            3, 0, 1, 0);
        deviceSU3 force1(pDeviceData[uiLink1]);
        force1.MulDagger(staple_term1_1);
        force1.Ta();
        force1.MulReal(betaOverN);
        pForceData[uiLink1].Add(force1);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term2_1(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmega)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmega * F(0.125);

    //===============
    //+x Omega V432
    //add force for mu=4
    UINT uiLink4 = _deviceGetLinkIndex(uiSiteIndex, 3);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 3))
    {
        deviceSU3 staple_term2_4 = _deviceStapleChairTerm1(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            3, 2, 1, 0);
        deviceSU3 force4(pDeviceData[uiLink4]);
        force4.MulDagger(staple_term2_4);
        force4.Ta();
        force4.MulReal(betaOverN);
        pForceData[uiLink4].Add(force4);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term2_2(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmega)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmega * F(0.125);

    //===============
    //+x Omega V432
    //add force for mu=4
    UINT uiLink2 = _deviceGetLinkIndex(uiSiteIndex, 1);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 1))
    {
        deviceSU3 staple_term2_2 = _deviceStapleChairTerm1(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            1, 2, 3, 0);
        deviceSU3 force2(pDeviceData[uiLink2]);
        force2.MulDagger(staple_term2_2);
        force2.Ta();
        force2.MulReal(betaOverN);
        pForceData[uiLink2].Add(force2);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term2_3(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmega)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmega * F(0.125);

    //===============
    //+x Omega V432
    //add force for mu=4
    UINT uiLink3 = _deviceGetLinkIndex(uiSiteIndex, 2);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 2))
    {
        deviceSU3 staple_term2_3 = _deviceStapleChairTerm2(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            3, 2, 1, 0);
        deviceSU3 force3(pDeviceData[uiLink3]);
        force3.MulDagger(staple_term2_3);
        force3.Ta();
        force3.MulReal(betaOverN);
        pForceData[uiLink3].Add(force3);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term3_1(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmega)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmega * F(0.125);

    //===============
    //-y Omega V421
    //add force for mu=4
    UINT uiLink4 = _deviceGetLinkIndex(uiSiteIndex, 3);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 3))
    {
        deviceSU3 staple_term3_4 = _deviceStapleChairTerm1(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            3, 1, 0, 1);
        deviceSU3 force4(pDeviceData[uiLink4]);
        force4.MulDagger(staple_term3_4);
        force4.Ta();
        force4.MulReal(betaOverN);
        pForceData[uiLink4].Add(force4);
    }

}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term3_2(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmega)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmega * F(0.125);

    //===============
    //-y Omega V421
    //add force for mu=4
    UINT uiLink1 = _deviceGetLinkIndex(uiSiteIndex, 0);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 0))
    {
        deviceSU3 staple_term3_1 = _deviceStapleChairTerm1(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            0, 1, 3, 1);
        deviceSU3 force1(pDeviceData[uiLink1]);
        force1.MulDagger(staple_term3_1);
        force1.Ta();
        force1.MulReal(betaOverN);
        pForceData[uiLink1].Add(force1);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term3_3(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmega)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmega * F(0.125);

    //===============
    //-y Omega V421
    //add force for mu=4
    UINT uiLink2 = _deviceGetLinkIndex(uiSiteIndex, 1);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 1))
    {
        deviceSU3 staple_term3_2 = _deviceStapleChairTerm2(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            3, 1, 0, 1);
        deviceSU3 force2(pDeviceData[uiLink2]);
        force2.MulDagger(staple_term3_2);
        force2.Ta();
        force2.MulReal(betaOverN);
        pForceData[uiLink2].Add(force2);
    }

}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term4_1(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmega)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmega * F(0.125);

    //===============
    //-y Omega V431
    //add force for mu=4
    UINT uiLink4 = _deviceGetLinkIndex(uiSiteIndex, 3);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 3))
    {
        deviceSU3 staple_term4_4 = _deviceStapleChairTerm1(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            3, 2, 0, 1);
        deviceSU3 force4(pDeviceData[uiLink4]);
        force4.MulDagger(staple_term4_4);
        force4.Ta();
        force4.MulReal(betaOverN);
        pForceData[uiLink4].Add(force4);
    }

}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term4_2(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmega)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmega * F(0.125);

    //===============
    //-y Omega V431
    //add force for mu=4
    UINT uiLink1 = _deviceGetLinkIndex(uiSiteIndex, 0);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 0))
    {
        deviceSU3 staple_term4_1 = _deviceStapleChairTerm1(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            0, 2, 3, 1);
        deviceSU3 force1(pDeviceData[uiLink1]);
        force1.MulDagger(staple_term4_1);
        force1.Ta();
        force1.MulReal(betaOverN);
        pForceData[uiLink1].Add(force1);
    }

}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term4_3(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmega)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmega * F(0.125);

    //===============
    //-y Omega V431
    //add force for mu=4
    UINT uiLink3 = _deviceGetLinkIndex(uiSiteIndex, 2);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 2))
    {
        deviceSU3 staple_term4_3 = _deviceStapleChairTerm2(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            3, 2, 0, 1);
        deviceSU3 force3(pDeviceData[uiLink3]);
        force3.MulDagger(staple_term4_3);
        force3.Ta();
        force3.MulReal(betaOverN);
        pForceData[uiLink3].Add(force3);
    }

}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term5_1(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmegaSq)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmegaSq * F(0.125);

    //===============
    //+Omega^2 xy V142
    UINT uiLink1 = _deviceGetLinkIndex(uiSiteIndex, 0);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 0))
    {
        deviceSU3 staple_term5_1 = _deviceStapleChairTerm1(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            0, 3, 1, 2);
        deviceSU3 force1(pDeviceData[uiLink1]);
        force1.MulDagger(staple_term5_1);
        force1.Ta();
        force1.MulReal(betaOverN);
        pForceData[uiLink1].Add(force1);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term5_2(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmegaSq)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmegaSq * F(0.125);

    //===============
    //+Omega^2 xy V142
    UINT uiLink2 = _deviceGetLinkIndex(uiSiteIndex, 1);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 1))
    {
        deviceSU3 staple_term5_2 = _deviceStapleChairTerm1(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            1, 3, 0, 2);
        deviceSU3 force2(pDeviceData[uiLink2]);
        force2.MulDagger(staple_term5_2);
        force2.Ta();
        force2.MulReal(betaOverN);
        pForceData[uiLink2].Add(force2);
    }

}

__global__ void _CLG_LAUNCH_BOUND
_kernelAddForceChairTermSU3_Term5_3(
    const deviceSU3 * __restrict__ pDeviceData,
    SSmallInt4 sCenterSite,
    deviceSU3 *pForceData,
    Real betaOverN, Real fOmegaSq)
{
    intokernalInt4;

    UINT uiBigIdx = __idx->_deviceGetBigIndex(sSite4);

    betaOverN = betaOverN * F(0.5) * fOmegaSq * F(0.125);

    //===============
    //+Omega^2 xy V142
    UINT uiLink4 = _deviceGetLinkIndex(uiSiteIndex, 3);

    if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 3))
    {
        deviceSU3 staple_term5_4 = _deviceStapleChairTerm2(pDeviceData, sCenterSite, sSite4, uiSiteIndex, uiBigIdx,
            0, 3, 1, 2);
        deviceSU3 force4(pDeviceData[uiLink4]);
        force4.MulDagger(staple_term5_4);
        force4.Ta();
        force4.MulReal(betaOverN);
        pForceData[uiLink4].Add(force4);
    }

}

#pragma endregion


CActionGaugePlaquetteRotating::CActionGaugePlaquetteRotating()
    : CAction()
    , m_uiPlaqutteCount(0)
    , m_fLastEnergy(F(0.0))
    , m_fNewEnergy(F(0.0))
    , m_fBetaOverN(F(0.1))
    , m_fOmega(F(0.0))
{
}

void CActionGaugePlaquetteRotating::PrepareForHMC(const CFieldGauge* pGauge, UINT uiUpdateIterate)
{
    if (0 == uiUpdateIterate)
    {
        m_fLastEnergy = Energy(FALSE, pGauge, NULL);
    }
}

void CActionGaugePlaquetteRotating::OnFinishTrajectory(UBOOL bAccepted)
{
    if (bAccepted)
    {
        m_fLastEnergy = m_fNewEnergy;
    }
}

void CActionGaugePlaquetteRotating::Initial(class CLatticeData* pOwner, const CParameters& param, BYTE byId)
{
    m_pOwner = pOwner;
    m_byActionId = byId;
    Real fBeta = 0.1f;
    param.FetchValueReal(_T("Beta"), fBeta);
    CCommonData::m_fBeta = fBeta;
    if (NULL != pOwner->m_pGaugeField && EFT_GaugeSU3 == pOwner->m_pGaugeField->GetFieldType())
    {
        fBeta = fBeta / F(3.0);
    }
    m_fBetaOverN = fBeta;
    m_uiPlaqutteCount = _HC_Volume * (_HC_Dir - 1) * (_HC_Dir - 2);

    Real fOmega = 0.1f;
    param.FetchValueReal(_T("Omega"), fOmega);
    m_fOmega = fOmega;
    CCommonData::m_fOmega = fOmega;

    TArray<INT> centerArray;
    param.FetchValueArrayINT(_T("Center"), centerArray);
    if (centerArray.Num() > 3)
    {
        m_sCenter.x = static_cast<SBYTE>(centerArray[0]);
        m_sCenter.y = static_cast<SBYTE>(centerArray[1]);
        m_sCenter.z = static_cast<SBYTE>(centerArray[2]);
        m_sCenter.w = static_cast<SBYTE>(centerArray[3]);
    }
    CCommonData::m_sCenter = m_sCenter;
}

void CActionGaugePlaquetteRotating::SetBeta(Real fBeta)
{
    CCommonData::m_fBeta = fBeta;
    if (NULL != m_pOwner->m_pGaugeField && EFT_GaugeSU3 == m_pOwner->m_pGaugeField->GetFieldType())
    {
        fBeta = fBeta / F(3.0);
    }
    m_fBetaOverN = fBeta;
}

UBOOL CActionGaugePlaquetteRotating::CalculateForceOnGauge(const CFieldGauge * pGauge, class CFieldGauge * pForce, class CFieldGauge * pStaple, ESolverPhase ePhase) const
{
    pGauge->CalculateForceAndStaple(pForce, pStaple, m_fBetaOverN);

    const CFieldGaugeSU3* pGaugeSU3 = dynamic_cast<const CFieldGaugeSU3*>(pGauge);
    CFieldGaugeSU3* pForceSU3 = dynamic_cast<CFieldGaugeSU3*>(pForce);
    if (NULL == pGaugeSU3 || NULL == pForceSU3)
    {
        appCrucial(_T("CActionGaugePlaquetteRotating only work with SU3 now.\n"));
        return TRUE;
    }

    preparethread;

    _kernelAddForce4PlaqutteTermSU3_XYZ << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega * m_fOmega);

    _kernelAddForce4PlaqutteTermSU3_T << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega * m_fOmega);

    _kernelAddForceChairTermSU3_Term1_1 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega);

    _kernelAddForceChairTermSU3_Term1_2 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega);

    _kernelAddForceChairTermSU3_Term1_3 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega);

    _kernelAddForceChairTermSU3_Term2_1 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega);

    _kernelAddForceChairTermSU3_Term2_2 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega);

    _kernelAddForceChairTermSU3_Term2_3 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega);

    _kernelAddForceChairTermSU3_Term3_1 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega);

    _kernelAddForceChairTermSU3_Term3_2 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega);

    _kernelAddForceChairTermSU3_Term3_3 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega);

    _kernelAddForceChairTermSU3_Term4_1 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega);

    _kernelAddForceChairTermSU3_Term4_2 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega);

    _kernelAddForceChairTermSU3_Term4_3 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega);

    _kernelAddForceChairTermSU3_Term5_1 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega * m_fOmega);

    _kernelAddForceChairTermSU3_Term5_2 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega * m_fOmega);

    _kernelAddForceChairTermSU3_Term5_3 << <block, threads >> >(pGaugeSU3->m_pDeviceData, m_sCenter,
        pForceSU3->m_pDeviceData, m_fBetaOverN, m_fOmega * m_fOmega);

    checkCudaErrors(hipDeviceSynchronize());
    return TRUE;
}

/**
* The implementation depends on the type of gauge field
*/
Real CActionGaugePlaquetteRotating::Energy(UBOOL bBeforeEvolution, const class CFieldGauge* pGauge, const class CFieldGauge* pStable)
{
    if (bBeforeEvolution)
    {
        return m_fLastEnergy;
    }
    m_fNewEnergy = pGauge->CalculatePlaqutteEnergy(m_fBetaOverN);

    const CFieldGaugeSU3D* pGaugeSU3 = dynamic_cast<const CFieldGaugeSU3D*>(pGauge);
    if (NULL == pGaugeSU3)
    {
        appCrucial(_T("CActionGaugePlaquetteRotating only work with SU3-Dirichlet now.\n"));
        return m_fNewEnergy;
    }

    preparethread;

    //======== this is only for test ================
    //_kernelAdd4PlaqutteTermSU3_Test << <block, threads >> > (
    //    pGaugeSU3->m_pDeviceData,
    //    m_sCenter,
    //    m_fBetaOverN,
    //    m_fOmega * m_fOmega,
    //    _D_RealThreadBuffer);

    appGetCudaHelper()->ThreadBufferZero(_D_RealThreadBuffer);

    dim3 block2 = block;
    block2.y = block.y * 3;
    _kernelAdd4PlaqutteTermSU3 << <block2, threads >> > (
            pGaugeSU3->m_pDeviceData, 
            appGetLattice()->m_pIndexCache->m_pPlaqutteCache,
            m_sCenter,
            m_fBetaOverN,
            m_fOmega * m_fOmega,
            _D_RealThreadBuffer);

    m_fNewEnergy += appGetCudaHelper()->ThreadBufferSum(_D_RealThreadBuffer);


    _kernelAddChairTermSU3_Term12 << <block, threads >> > (
        pGaugeSU3->m_pDeviceData,
        m_sCenter,
        m_fBetaOverN,
        m_fOmega,
        _D_RealThreadBuffer);

    m_fNewEnergy += appGetCudaHelper()->ThreadBufferSum(_D_RealThreadBuffer);

    _kernelAddChairTermSU3_Term34 << <block, threads >> > (
        pGaugeSU3->m_pDeviceData,
        m_sCenter,
        m_fBetaOverN,
        m_fOmega,
        _D_RealThreadBuffer);

    m_fNewEnergy += appGetCudaHelper()->ThreadBufferSum(_D_RealThreadBuffer);

    _kernelAddChairTermSU3_Term5 << <block, threads >> > (
        pGaugeSU3->m_pDeviceData,
        m_sCenter,
        m_fBetaOverN,
        m_fOmega * m_fOmega,
        _D_RealThreadBuffer);

    m_fNewEnergy += appGetCudaHelper()->ThreadBufferSum(_D_RealThreadBuffer);

    return m_fNewEnergy;
}

//Real CActionGaugePlaquetteRotating::GetEnergyPerPlaqutte() const
//{
//    return m_pOwner->m_pGaugeField->CalculatePlaqutteEnergy(m_fBetaOverN) / m_uiPlaqutteCount;
//}

void CActionGaugePlaquetteRotating::SetOmega(Real fOmega) 
{ 
    m_fOmega = fOmega; 
    CCommonData::m_fOmega = fOmega;
}

void CActionGaugePlaquetteRotating::SetCenter(const SSmallInt4 &newCenter) 
{
    m_sCenter = newCenter;
    CCommonData::m_sCenter = newCenter;
}

CCString CActionGaugePlaquetteRotating::GetInfos(const CCString &tab) const
{
    CCString sRet;
    sRet = tab + _T("Name : CActionGaugePlaquetteRotating\n");
    sRet = sRet + tab + _T("Beta : ") + appFloatToString(CCommonData::m_fBeta) + _T("\n");
    sRet = sRet + tab + _T("Omega : ") + appFloatToString(m_fOmega) + _T("\n");
    CCString sCenter;
    sCenter.Format(_T("Center: [%d, %d, %d, %d]\n")
        , static_cast<INT>(m_sCenter.x)
        , static_cast<INT>(m_sCenter.y)
        , static_cast<INT>(m_sCenter.z)
        , static_cast<INT>(m_sCenter.w));
    sRet = sRet + tab + sCenter;
    return sRet;
}

__END_NAMESPACE


//=============================================================================
// END OF FILE
//=============================================================================