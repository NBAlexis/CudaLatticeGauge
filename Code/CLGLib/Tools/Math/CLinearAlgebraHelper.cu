#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CLinearAlgebraHelper.h
// 
// DESCRIPTION:
//
//
// REVISION:
//  [03/14/2019 nbale]
//=============================================================================
#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

CLinearAlgebraHelper::CLinearAlgebraHelper(UINT uiDim)
    : m_uiDim(uiDim)
    , m_pDeviceIntBuffer(NULL)
    , m_pDeviceFloatBuffer(NULL)
    , m_pDeviceComplexBuffer1(NULL)
    , m_pDeviceComplexBuffer2(NULL)
{
    if (uiDim > _kMaxSmallDim)
    {
        appGeneral(_T("CLinearAlgebraHelper only support dim <= 32 !!\n"));
        m_uiDim = 0;
        return;
    }

    checkCudaErrors(hipMalloc((void**)&m_pDeviceIntBuffer, sizeof(INT) * m_uiDim));
    checkCudaErrors(hipMalloc((void**)&m_pDeviceFloatBuffer, sizeof(Real) * m_uiDim));
    checkCudaErrors(hipMalloc((void**)&m_pDeviceComplexBuffer1, sizeof(CLGComplex) * m_uiDim));
    checkCudaErrors(hipMalloc((void**)&m_pDeviceComplexBuffer2, sizeof(CLGComplex) * m_uiDim));

    //5 is enough
    AddTempMatrix(5);
}

CLinearAlgebraHelper::~CLinearAlgebraHelper()
{
    if (NULL != m_pDeviceIntBuffer)
    {
        checkCudaErrors(hipFree(m_pDeviceIntBuffer));
    }
    if (NULL != m_pDeviceFloatBuffer)
    {
        checkCudaErrors(hipFree(m_pDeviceFloatBuffer));
    }
    if (NULL != m_pDeviceComplexBuffer1)
    {
        checkCudaErrors(hipFree(m_pDeviceComplexBuffer1));
    }
    if (NULL != m_pDeviceComplexBuffer2)
    {
        checkCudaErrors(hipFree(m_pDeviceComplexBuffer2));
    }

    for (int i = 0; i < m_lstTmpMatrix.Num(); ++i)
    {
        checkCudaErrors(hipFree(m_lstTmpMatrix[i].m_pMatrix));
    }
}

void CLinearAlgebraHelper::TestSmallMatrix()
{
    const INT testDim1 = 30;
    const INT testDim2 = 15;
    CLinearAlgebraHelper* pHelper = new CLinearAlgebraHelper(testDim1);
    
    CLGComplex mij[testDim1 * testDim2]; //Test thin QR, mij dagger mij
    CLGComplex hij[testDim1 * testDim1]; //Test EigenValue hij block mult iij, gev
    CLGComplex iij[testDim1 * testDim1]; //block multiply
    CLGComplex aij[testDim1 * testDim1]; //test gev
    CLGComplex bij[testDim1 * testDim1]; //test gev
    CLGComplex tij[testDim1 * testDim1]; //test triangular
    CLGComplex hesij[testDim1 * testDim1]; //test hessenberg
    CLGComplex heij[(testDim1 - 1) * testDim1];
    CLGComplex yij[testDim1];
    CLGComplex res1[testDim1 * testDim1];
    CLGComplex res2[testDim1 * testDim1];
    for (INT x = 0; x < testDim1; ++x)
    {
        for (INT y = 0; y < testDim2; ++y)
        {
            mij[x * testDim2 + y].x = (rand() % 101 - 50) / 50.0f;
            mij[x * testDim2 + y].y = (rand() % 101 - 50) / 50.0f;
        }

        for (INT y = 0; y < testDim1; ++y)
        {
            hij[x * testDim1 + y].x = (rand() % 101 - 50) / 50.0f;
            hij[x * testDim1 + y].y = (rand() % 101 - 50) / 50.0f;
            aij[x * testDim1 + y].x = (rand() % 101 - 50) / 50.0f;
            aij[x * testDim1 + y].y = (rand() % 101 - 50) / 50.0f;
            bij[x * testDim1 + y].x = (rand() % 101 - 50) / 50.0f;
            bij[x * testDim1 + y].y = (rand() % 101 - 50) / 50.0f;
            if (y >= x)
            {
                tij[x * testDim1 + y].x = (rand() % 101 - 50) / 50.0f;
                tij[x * testDim1 + y].y = (rand() % 101 - 50) / 50.0f;
            }
            else
            {
                tij[x * testDim1 + y] = _make_cuComplex(F(0.0), F(0.0));
            }

            if (y >= x - 1)
            {
                hesij[x * testDim1 + y].x = (rand() % 101 - 50) / 50.0f;
                hesij[x * testDim1 + y].y = (rand() % 101 - 50) / 50.0f;
            }
            else
            {
                hesij[x * testDim1 + y] = _make_cuComplex(F(0.0), F(0.0));
            }

            if (x < testDim2 && y < testDim2)
            {
                iij[x * testDim1 + y].x = (rand() % 101 - 50) / 50.0f;
                iij[x * testDim1 + y].y = (rand() % 101 - 50) / 50.0f;
            }
            else
            {
                if (y == x)
                {
                    iij[x * testDim1 + y] = _make_cuComplex(F(1.0), F(0.0));
                }
                else
                {
                    iij[x * testDim1 + y] = _make_cuComplex(F(0.0), F(0.0));
                }
            }
            if (y < testDim1 - 1)
            {
                if (y >= x - 1)
                {
                    heij[x * (testDim1 - 1) + y].x = (rand() % 101 - 50) / 50.0f;
                    heij[x * (testDim1 - 1) + y].y = (rand() % 101 - 50) / 50.0f;
                }
                else
                {
                    heij[x * (testDim1 - 1) + y] = _make_cuComplex(F(0.0), F(0.0));
                }
            }
        }

        if (x < (testDim1 - 1))
        {
            yij[x] = _make_cuComplex((rand() % 101 - 50) / 50.0f, (rand() % 101 - 50) / 50.0f);
        }
        else
        {
            yij[x] = _make_cuComplex(F(0.0), F(0.0));
        }
    }

    appGeneral(_T("\n(* ============= Copy the output below to Mathematica ============== *)\n"));

    appGeneral(_T("\nm=\n"));
    PrintMatrix(mij, testDim1, testDim2);
    appGeneral(_T("\nh=\n"));
    PrintMatrix(hij, testDim1, testDim1);
    appGeneral(_T("\nb=\n"));
    PrintMatrix(bij, testDim1, testDim1);
    appGeneral(_T("\na=\n"));
    PrintMatrix(aij, testDim1, testDim1);
    appGeneral(_T("\nii=\n"));
    PrintMatrix(iij, testDim1, testDim1);
    appGeneral(_T("\nt=\n"));
    PrintMatrix(tij, testDim1, testDim1);
    appGeneral(_T("\nhe=\n"));
    PrintMatrix(heij, testDim1, testDim1 - 1);
    appGeneral(_T("\nyy=\n"));
    PrintMatrix(yij, testDim1, 1);
    appGeneral(_T("\nhes=\n"));
    PrintMatrix(hesij, testDim1, testDim1);

    //transpose
    pHelper->TransposeHost(mij, testDim1, testDim2);
    appGeneral(_T("\ntm=\n"));
    PrintMatrix(mij, testDim2, testDim1);
    pHelper->TransposeHost(mij, testDim2, testDim1);

    //matrix multiply
    pHelper->SmallMatrixMultHost(res1, mij, mij, testDim2, testDim1, testDim2, TRUE, FALSE);
    appGeneral(_T("\nmm=\n"));
    PrintMatrix(res1, testDim2, testDim2);

    //block multiply
    pHelper->BlockMatrixMultHost(res1, hij, iij, testDim1, 0, testDim2, FALSE, TRUE, FALSE);
    appGeneral(_T("\nhdaggeri=\n"));
    PrintMatrix(res1, testDim1, testDim1);

    //Solve Y
    pHelper->ThinQRFactorizationHost(res1, res2, mij, testDim1, testDim2);
    pHelper->SolveYHost(mij, tij, testDim2, testDim1);
    appGeneral(_T("\ninversetm=\n"));
    PrintMatrix(mij, testDim1, testDim2);

    //QR factor m is changed in Solve Y, so put this in fromt of Solve Y
    //pHelper->ThinQRFactorizationHost(res1, res2, mij, testDim1, testDim2);
    appGeneral(_T("\nq=\n"));
    PrintMatrix(res1, testDim1, testDim2);
    appGeneral(_T("\nr=\n"));
    PrintMatrix(res2, testDim2, testDim2);

    //Henssenberg
    pHelper->RotateHenssenbergHost(heij, yij, testDim1 - 1);
    appGeneral(_T("\nrhe=\n"));
    PrintMatrix(heij, testDim1 - 1, testDim1 - 1);
    appGeneral(_T("\nryy=\n"));
    PrintMatrix(yij, testDim1 - 1, 1);
    appGeneral(_T("\nrheresidue=%f %s %f I;\n"), 
        yij[testDim1 - 1].x,
        yij[testDim1 - 1].y < 0 ? _T("") : _T("+"),
        yij[testDim1 - 1].y);

    //EV
    pHelper->EigenValueProblemHost(hij, res1, res2, testDim1, testDim2);
    appGeneral(_T("\nev1=\n"));
    PrintMatrix(res1, 1, testDim2);
    appGeneral(_T("\nvv1=\n"));
    PrintMatrix(res2, testDim2, testDim1);

    //GEV
    pHelper->GeneralizedEigenValueProblemHost(aij, bij, res1, res2, testDim1, testDim2);
    appGeneral(_T("\nev2=\n"));
    PrintMatrix(res1, 1, testDim2);
    appGeneral(_T("\nvv2=\n"));
    PrintMatrix(res2, testDim2, testDim1);

    //TEV
    pHelper->UpperTriangularEigenVectorsHost(tij, res1, res2, testDim1, testDim2);
    appGeneral(_T("\nev3=\n"));
    PrintMatrix(res1, 1, testDim2);
    appGeneral(_T("\nvv3=\n"));
    PrintMatrix(res2, testDim2, testDim1);

    //HEV
    pHelper->EigenValueProblemHessenbergHost(hesij, res1, res2, testDim1, testDim2);
    appGeneral(_T("\nev4=\n"));
    PrintMatrix(res1, 1, testDim2);
    appGeneral(_T("\nvv4=\n"));
    PrintMatrix(res2, testDim2, testDim1);

    appGeneral(_T("\nPrint[\"Test matrix transpose\"]\n"));
    appGeneral(_T("\nMax[Transpose[m]-tm //Flatten//Abs]\n"));
    appGeneral(_T("\nPrint[\"Test matrix multiply\"]\n"));
    appGeneral(_T("\nMax[Conjugate[Transpose[m]].m-mm //Flatten//Abs]\n"));
    appGeneral(_T("\nMax[Conjugate[Transpose[h]].ii-hdaggeri //Flatten//Abs]\n"));
    appGeneral(_T("\nPrint[\"Test backward substitution\"]\n"));
    appGeneral(_T("\nMax[Inverse[t].m-inversetm //Flatten//Abs]/Max[Inverse[t].m // Flatten // Abs]\n"));

    appGeneral(_T("\nPrint[\"Test QR factorization\"]\n"));
    appGeneral(_T("\nMax[q.r-m //Flatten//Abs]\n"));
    appGeneral(_T("Tr[Conjugate[Transpose[q]].q] / %d - 1\n"), testDim2);

    appGeneral(_T("\nPrint[\"Test Henssenberg rotation\"]\n"));
    appGeneral(_T("\nNorm[he.(Inverse[rhe].ryy) - yy] - Abs[rheresidue]\n"));

    appGeneral(_T("\nPrint[\"Test Eigen Value\"]\n"));

    appGeneral(_T("Eigenvalues[h][[Table[%d - i, {i, 1, %d}]]] - ev1[[1]] // Abs // Max\n"), testDim1 + 1, testDim2);
    for (INT i = 0; i < testDim2; ++i)
    {
        appGeneral(_T("Max[Eigensystem[h][[2]][[%d]]/Eigensystem[h][[2]][[%d]][[%d]] - vv1[[%d]] / vv1[[%d]][[%d]] //Abs]\n"),
            testDim1 - i, testDim1 - i, testDim1, i + 1, i + 1, testDim1);
    }

    appGeneral(_T("\nPrint[\"Test Generalized Eigen Value\"]\n"));

    appGeneral(_T("Eigenvalues[Inverse[b].a][[Table[%d - i, {i, 1, %d}]]] - ev2[[1]] // Abs // Max\n"), testDim1 + 1, testDim2);
    for (INT i = 0; i < testDim2; ++i)
    {
        appGeneral(_T("Max[Eigensystem[Inverse[b].a][[2]][[%d]]/Eigensystem[Inverse[b].a][[2]][[%d]][[%d]] - vv2[[%d]] / vv2[[%d]][[%d]] //Abs]\n"),
            testDim1 - i, testDim1 - i, testDim1, i + 1, i + 1, testDim1);
    }

    appGeneral(_T("\nPrint[\"Test Upper triangular Eigen Value\"]\n"));

    appGeneral(_T("Eigenvalues[t][[Table[%d - i, {i, 1, %d}]]] - ev3[[1]] // Abs // Max\n"), testDim1 + 1, testDim2);
    for (INT i = 0; i < testDim2; ++i)
    {
        appGeneral(_T("Abs[Conjugate[vv3[[%d]]].Normalize[Eigensystem[t][[2]][[%d]]]] - 1\n"),
            i + 1, testDim1 - i);
    }

    appGeneral(_T("\nPrint[\"Test Hessenberg Eigen Value\"]\n"));

    appGeneral(_T("Eigenvalues[hes][[Table[%d - i, {i, 1, %d}]]] - ev4[[1]] // Abs // Max\n"), testDim1 + 1, testDim2);
    for (INT i = 0; i < testDim2; ++i)
    {
        appGeneral(_T("Abs[Conjugate[vv4[[%d]]].Normalize[Eigensystem[hes][[2]][[%d]]]] - 1\n"),
            i + 1, testDim1 - i);
    }

    appGeneral(_T("\n\n(*============= Please copy those results to Mathematica to check, all should be nearly zero ============ *)\n"));
}

#pragma region Common

#pragma region Initial

__global__ void _CLG_LAUNCH_BOUND
_kernelInitialZero(CLGComplex* R, UINT dy)
{
    R[threadIdx.x * dy + threadIdx.y] = _make_cuComplex(F(0.0), F(0.0));
}

__global__ void _CLG_LAUNCH_BOUND
_kernelInitialOne(CLGComplex* R, UINT dy)
{
    UINT i = threadIdx.x;
    UINT j = threadIdx.y;
    if (i == j)
    {
        R[threadIdx.x * dy + threadIdx.y] = _make_cuComplex(F(1.0), F(0.0));
    }
    else
    {
        R[threadIdx.x * dy + threadIdx.y] = _make_cuComplex(F(0.0), F(0.0));
    }
}

void CLinearAlgebraHelper::InitialZero(CLGComplex* deviceMatrix, UINT dx, UINT dy)
{
    dim3 block(1, 1, 1);
    dim3 thread(dx, dy, 1);
    _kernelInitialZero << <block, thread >> > (deviceMatrix, dy);
}

void CLinearAlgebraHelper::InitialOne(CLGComplex* deviceMatrix, UINT dx)
{
    dim3 block(1, 1, 1);
    dim3 thread(dx, dx, 1);
    _kernelInitialOne << <block, thread >> > (deviceMatrix, dx);
}

#pragma endregion

#pragma region Multiply

/**
* This is tested to be faster than for, even with launch bound
* If Left = X*Y, Right = Y*Z
* Res = X*Z, block(Y,1,1) thread(X,Z,1)
* leftDim = Y, midDim = Z
*/
__global__ void _kernelSmallMatrixMult_NN(CLGComplex* res,
    const CLGComplex* __restrict__ left,
    const CLGComplex* __restrict__ right,
    UINT leftDim, UINT midDim)
{
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;
    UINT n = blockIdx.x;

    //left is dx x n, right is n x dy matrix
    CLGComplex toAdd = _cuCmulf(left[x * leftDim + n], right[n * midDim + y]);

    atomicAdd(&res[x * midDim + y].x, toAdd.x);
    atomicAdd(&res[x * midDim + y].y, toAdd.y);
}

/**
* Left * Right^+
* If Left = X*Y, Right = Z*Y (Right^+ = Y*Z)
* Res = X*Z, block(Y,1,1) thread(X,Z,1)
* leftDim = Y, midDim = Z
*/
__global__ void _kernelSmallMatrixMult_ND(CLGComplex* res,
    const CLGComplex* __restrict__ left,
    const CLGComplex* __restrict__ right,
    UINT leftDim, UINT midDim)
{
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;
    UINT n = blockIdx.x;

    //left is dx x n, right is n x dy matrix
    //left = dx dy
    //mid = dx dz
    //right = dz dy (right dagger = dy dz)
    //n->0->dy
    //x->0->dx
    //y->0->dz
    //leftDim = dy
    //rightDim = dy
    //midDim = dz
    CLGComplex toAdd = _cuCmulf(left[x * leftDim + n], _cuConjf(right[y * leftDim + n]));

    atomicAdd(&res[x * midDim + y].x, toAdd.x);
    atomicAdd(&res[x * midDim + y].y, toAdd.y);
}

/**
* Left^+ * Right
* If Left = Y*X (Left^+ = X*Y), Right = Y*Z
* Res = X*Z, block(Y,1,1) thread(X,Z,1)
* leftDim = X, midDim = Z
*/
__global__ void _kernelSmallMatrixMult_DN(CLGComplex* res,
    const CLGComplex* __restrict__ left,
    const CLGComplex* __restrict__ right,
    UINT leftDim, UINT midDim)
{
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;
    UINT n = blockIdx.x;

    CLGComplex toAdd = _cuCmulf(_cuConjf(left[n * leftDim + x]), right[n * midDim + y]);

    atomicAdd(&res[x * midDim + y].x, toAdd.x);
    atomicAdd(&res[x * midDim + y].y, toAdd.y);
}


void CLinearAlgebraHelper::SmallMatrixMult(
    CLGComplex * deviceRes, 
    const CLGComplex* left, 
    const CLGComplex* right, 
    UINT dLeft, UINT dMid, UINT dRight, 
    UBOOL bLeftDagger, UBOOL bRightDagger)
{
    InitialZero(deviceRes, dLeft, dRight);
    dim3 block(dMid, 1, 1);
    dim3 thread(dLeft, dRight, 1);
    if (bLeftDagger)
    {
        _kernelSmallMatrixMult_DN << <block, thread >> >(deviceRes, left, right, dLeft, dRight);
    }
    else if (bRightDagger)
    {
        _kernelSmallMatrixMult_ND<<<block, thread >>>(deviceRes, left, right, dMid, dRight);
    }
    else
    {
        _kernelSmallMatrixMult_NN << <block, thread >> > (deviceRes, left, right, dMid, dRight);
    }
}

#pragma endregion

#pragma region Block Multply

/**
* left = 1 0 0
*        0 U 0
*        0 0 1
*
* right = A1 A2 A3
*         B1 B2 B3
*         C1 C2 C3
*
* res = A1  A2  A3
*       UB1 UB2 UB3
*       C1  C2  C3
*
* if U is dy x dy
* block  = dy, 1, 1
* thread = dx, dx, 1
* Assume res is zeroed
* Y Dir ----->
*/
__global__ void _kernelMatrixBlockMult_LNN(CLGComplex* res, const CLGComplex* __restrict__ left, const CLGComplex* __restrict__ right, UINT iStart, UINT iEnd, UINT dm)
{
    UINT n = blockIdx.x;
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;

    if (x >= iStart && x < iEnd)
    {
        UINT mid = n + iStart;
        CLGComplex toAdd = _cuCmulf(left[x * dm + mid], right[mid * dm + y]);
        atomicAdd(&res[x * dm + y].x, toAdd.x);
        atomicAdd(&res[x * dm + y].y, toAdd.y);
    }
    else
    {
        if (0 == n)
        {
            res[x * dm + y] = right[x * dm + y];
        }
    }
}

__global__ void _kernelMatrixBlockMult_LDN(CLGComplex* res, const CLGComplex* __restrict__ left, const CLGComplex* __restrict__ right, UINT iStart, UINT iEnd, UINT dm)
{
    UINT n = blockIdx.x;
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;

    if (x >= iStart && x < iEnd)
    {
        UINT mid = n + iStart;
        CLGComplex toAdd = _cuCmulf(_cuConjf(left[mid * dm + x]), right[mid * dm + y]);
        atomicAdd(&res[x * dm + y].x, toAdd.x);
        atomicAdd(&res[x * dm + y].y, toAdd.y);
    }
    else
    {
        if (0 == n)
        {
            res[x * dm + y] = right[x * dm + y];
        }
    }
}

__global__ void _kernelMatrixBlockMult_LND(CLGComplex* res, const CLGComplex* __restrict__ left, const CLGComplex* __restrict__ right, UINT iStart, UINT iEnd, UINT dm)
{
    UINT n = blockIdx.x;
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;

    if (x >= iStart && x < iEnd)
    {
        UINT mid = n + iStart;
        CLGComplex toAdd = _cuCmulf(left[x * dm + mid], _cuConjf(right[y * dm + mid]));
        atomicAdd(&res[x * dm + y].x, toAdd.x);
        atomicAdd(&res[x * dm + y].y, toAdd.y);
    }
    else
    {
        if (0 == n)
        {
            res[x * dm + y] = _cuConjf(right[y * dm + x]);
        }
    }
}

/**
* left = A1 A2 A3
*        B1 B2 B3
*        C1 C2 C3
*
* right = 1 0 0
*         0 U 0
*         0 0 1
*
* res = A1 A2U A3
*       B1 B2U B3
*       C1 C2U C3
*
* if U is dy x dy
* block  = dy, 1, 1
* thread = dx, dx, 1
* Assume res is zeroed
* Y Dir ----->
*/
__global__ void _kernelMatrixBlockMult_RNN(CLGComplex* res, const CLGComplex* __restrict__ left, const CLGComplex* __restrict__ right, UINT iStart, UINT iEnd, UINT dm)
{
    UINT n = blockIdx.x;
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;

    if (y >= iStart && y < iEnd)
    {
        UINT mid = n + iStart;
        CLGComplex toAdd = _cuCmulf(left[x * dm + mid], right[mid * dm + y]);
        atomicAdd(&res[x * dm + y].x, toAdd.x);
        atomicAdd(&res[x * dm + y].y, toAdd.y);
    }
    else
    {
        if (0 == n)
        {
            res[x * dm + y] = left[x * dm + y];
        }
    }
}

__global__ void _kernelMatrixBlockMult_RDN(CLGComplex* res, const CLGComplex* __restrict__ left, const CLGComplex* __restrict__ right, UINT iStart, UINT iEnd, UINT dm)
{
    UINT n = blockIdx.x;
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;

    if (y >= iStart && y < iEnd)
    {
        UINT mid = n + iStart;
        CLGComplex toAdd = _cuCmulf(_cuConjf(left[mid * dm + x]), right[mid * dm + y]);
        atomicAdd(&res[x * dm + y].x, toAdd.x);
        atomicAdd(&res[x * dm + y].y, toAdd.y);
    }
    else
    {
        if (0 == n)
        {
            res[x * dm + y] = _cuConjf(left[y * dm + x]);
        }
    }
}

__global__ void _kernelMatrixBlockMult_RND(CLGComplex* res, const CLGComplex* __restrict__ left, const CLGComplex* __restrict__ right, UINT iStart, UINT iEnd, UINT dm)
{
    UINT n = blockIdx.x;
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;

    if (y >= iStart && y < iEnd)
    {
        UINT mid = n + iStart;
        CLGComplex toAdd = _cuCmulf(left[x * dm + mid], _cuConjf(right[y * dm + mid]));
        atomicAdd(&res[x * dm + y].x, toAdd.x);
        atomicAdd(&res[x * dm + y].y, toAdd.y);
    }
    else
    {
        if (0 == n)
        {
            res[x * dm + y] = left[x * dm + y];
        }
    }
}

void CLinearAlgebraHelper::BlockMatrixMult(
    CLGComplex * deviceRes,
    const CLGComplex* left,
    const CLGComplex* right,
    UINT dDim, UINT uiStart, UINT uiEnd,
    UBOOL bLeft, UBOOL bLeftDagger, UBOOL bRightDagger)
{
    InitialZero(deviceRes, dDim, dDim);
    dim3 block(uiEnd - uiStart, 1, 1);
    dim3 thread(dDim, dDim, 1);

    if (bLeft)
    {
        if (bLeftDagger)
        {
            _kernelMatrixBlockMult_LDN << <block, thread >> >(deviceRes, left, right, uiStart, uiEnd, dDim);
        }
        else if (bRightDagger)
        {
            _kernelMatrixBlockMult_LND << <block, thread >> >(deviceRes, left, right, uiStart, uiEnd, dDim);
        }
        else
        {
            _kernelMatrixBlockMult_LNN << <block, thread >> > (deviceRes, left, right, uiStart, uiEnd, dDim);
        }
    }
    else
    {
        if (bLeftDagger)
        {
            _kernelMatrixBlockMult_RDN << <block, thread >> >(deviceRes, left, right, uiStart, uiEnd, dDim);
        }
        else if (bRightDagger)
        {
            _kernelMatrixBlockMult_RND << <block, thread >> >(deviceRes, left, right, uiStart, uiEnd, dDim);
        }
        else
        {
            _kernelMatrixBlockMult_RNN << <block, thread >> > (deviceRes, left, right, uiStart, uiEnd, dDim);
        }
    }
}

#pragma endregion

#pragma region Add, Minus etc

/**
* M=M+cI, c is device buffer
*/
__global__ void _CLG_LAUNCH_BOUND
_kernelMatrixAddConstant(CLGComplex* m, CLGComplex* c, UINT dy)
{
    UINT i = threadIdx.x;
    m[i * dy + i] = _cuCaddf(m[i * dy + i], c[0]);
}

__global__ void _CLG_LAUNCH_BOUND
_kernelMatrixTranspose(const CLGComplex* __restrict__ m, CLGComplex* tmpM, UINT dx, UINT dy)
{
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;

    tmpM[y * dx + x] = m[x * dy + y];
}

void CLinearAlgebraHelper::Transpose(CLGComplex* deviceMatrix, UINT dx, UINT dy)
{
    if (dx > m_uiDim || dy > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpMRes = GetTmpMatrix();
    CLGComplex* tmpMRes = sTmpMRes.m_pMatrix;

    dim3 block(1, 1, 1);
    dim3 thread(dx, dy, 1);
    _kernelMatrixTranspose << <block, thread >> > (deviceMatrix, tmpMRes, dx, dy);
    checkCudaErrors(hipMemcpy(deviceMatrix, tmpMRes, sizeof(CLGComplex) * dx * dy, hipMemcpyDeviceToDevice));

    sTmpMRes.Free();
}

void CLinearAlgebraHelper::TransposeHost(CLGComplex* hostMatrix, UINT dx, UINT dy)
{
    if (dx > m_uiDim || dy > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpMRes = GetTmpMatrix();
    CLGComplex* tmpMRes = sTmpMRes.m_pMatrix;
    checkCudaErrors(hipMemcpy(tmpMRes, hostMatrix, sizeof(CLGComplex) * dx * dy, hipMemcpyHostToDevice));
    Transpose(tmpMRes, dx, dy);
    checkCudaErrors(hipMemcpy(hostMatrix, tmpMRes, sizeof(CLGComplex) * dx * dy, hipMemcpyDeviceToHost));

    sTmpMRes.Free();
}

#pragma endregion

#pragma region Block Copy

/**
* thread.xy = lx,ly
*/
__global__ void _CLG_LAUNCH_BOUND
_kernelCopyMatrixXY(
    CLGComplex* mtr,
    const CLGComplex* __restrict__ orignal,
    UINT newdy, UINT olddy)
{
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;

    mtr[x * newdy + y] = orignal[x * olddy + y];
}

void CLinearAlgebraHelper::BlockCopy(CLGComplex* deviceDest, const CLGComplex* deviceSrc,
    UINT lengthX, UINT lengthY, UINT dimDest, UINT dimSrc)
{
    dim3 block(1, 1, 1);
    dim3 thread(lengthX, lengthY, 1);
    _kernelCopyMatrixXY << <block, thread >> > (deviceDest, deviceSrc, dimDest, dimSrc);
}

#pragma endregion

void CLinearAlgebraHelper::PrintMatrix(const CLGComplex* mtr, UINT dx, UINT dy)
{
    appGeneral("\n{");
    for (UINT i = 0; i < dx; ++i)
    {
        for (UINT j = 0; j < dy; ++j)
        {
            appGeneral("%s%1.8f %s %1.8f I%s ",
                0 == j ? "{" : "",
                mtr[i * dy + j].x,
                mtr[i * dy + j].y < F(0.0) ? "" : "+",
                mtr[i * dy + j].y,
                dy - 1 == j ? "}" : ",");
        }
        if (i == dx - 1)
        {
            appGeneral("};\n");
        }
        else
        {
            appGeneral(",\n");
        }
    }
}

#pragma endregion

#pragma region QR Decomposition

__global__ void _CLG_LAUNCH_BOUND
_kernelOneStepHouseHolderQR(
    CLGComplex* Q,
    const CLGComplex* __restrict__ R,
    UINT i, UINT dy)
{
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;

    __shared__ Real length;
    __shared__ Real lengthu;
    __shared__ CLGComplex u[CLinearAlgebraHelper::_kMaxSmallDim];

    if (0 == x && 0 == y)
    {
        length = F(0.0);
        lengthu = F(0.0);
    }

    __syncthreads();

    if (0 == y && x >= i)
    {
        atomicAdd(&length, R[x * dy + i].x * R[x * dy + i].x + R[x * dy + i].y * R[x * dy + i].y);
    }

    __syncthreads();

    if (0 == x && 0 == y)
    {
        length = _sqrt(length);
    }

    __syncthreads();

    if (0 == y && x >= i)
    {
        u[x] = R[x * dy + i];

        if (x == i)
        {
            Real fuxL = __div(F(1.0), _cuCabsf(u[x]));
            Real fCos = u[x].x * fuxL;
            Real fSin = u[x].y * fuxL;
            u[x] = _cuCaddf(u[x], _make_cuComplex(length * fCos, length * fSin));
        }
        atomicAdd(&lengthu, u[x].x * u[x].x + u[x].y * u[x].y);
    }

    __syncthreads();

    if (0 == x && 0 == y)
    {
        lengthu = __div(F(1.0), _sqrt(lengthu * F(0.5)));
    }

    __syncthreads();

    if (0 == y && x >= i)
    {
        u[x].x = u[x].x * lengthu;
        u[x].y = u[x].y * lengthu;
    }

    __syncthreads();

    //uk = A[i + 1->n, i] - |A[i+1]|
    if (x < i || y < i)
    {
        if (x == y)
        {
            Q[x * dy + y] = _make_cuComplex(F(1.0), F(0.0));
        }
        else
        {
            Q[x * dy + y] = _make_cuComplex(F(0.0), F(0.0));
        }
    }
    else
    {
        Q[x * dy + y] = _cuCmulf(_cuConjf(u[y]), u[x]);
        Q[x * dy + y].x = -Q[x * dy + y].x;
        Q[x * dy + y].y = -Q[x * dy + y].y;
        if (x == y)
        {
            Q[x * dy + y].x = Q[x * dy + y].x + F(1.0);
        }
    }
}

void CLinearAlgebraHelper::QRFactorization(
    CLGComplex* Q, 
    CLGComplex* R, 
    const CLGComplex* T, 
    UINT dy)
{
    if (dy > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }
    dim3 block(1, 1, 1);
    dim3 thread1(dy, dy, 1);
    STmpMatrix sTmpQ = GetTmpMatrix();
    CLGComplex* tmpQ = sTmpQ.m_pMatrix;
    STmpMatrix sTmpM = GetTmpMatrix();
    CLGComplex* tmpM = sTmpM.m_pMatrix;
    checkCudaErrors(hipMemcpy(R, T, sizeof(CLGComplex) * dy * dy, hipMemcpyDeviceToDevice));
    InitialOne(Q, dy);
    for (UINT i = 0; i < dy - 1; ++i)
    {
        _kernelOneStepHouseHolderQR << <block, thread1 >> > (tmpQ, R, i, dy);

        //left is block
        BlockMatrixMult(tmpM, tmpQ, R, dy, i, dy, TRUE, FALSE, FALSE);

        checkCudaErrors(hipMemcpy(R, tmpM, sizeof(CLGComplex) * dy * dy, hipMemcpyDeviceToDevice));

        //right is block and right is dagger
        BlockMatrixMult(tmpM, Q, tmpQ, dy, i, dy, FALSE, FALSE, TRUE);

        checkCudaErrors(hipMemcpy(Q, tmpM, sizeof(CLGComplex) * dy * dy, hipMemcpyDeviceToDevice));
    }
    sTmpQ.Free();
    sTmpM.Free();
}

void CLinearAlgebraHelper::ThinQRFactorization(
    CLGComplex* Q,
    CLGComplex* R,
    const CLGComplex* T,
    UINT dx, UINT dy)
{
    if (dy > m_uiDim || dx > m_uiDim || dy > dx)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d, or it is not thin!\n"), m_uiDim);
        return;
    }

    dim3 thread2(dx, dx, 1);

    STmpMatrix sTmpR = GetTmpMatrix();
    CLGComplex* tmpR = sTmpR.m_pMatrix;
    STmpMatrix sTmpQ = GetTmpMatrix();
    CLGComplex* tmpQ = sTmpQ.m_pMatrix;
    STmpMatrix sTmpQ2 = GetTmpMatrix();
    CLGComplex* tmpQ2 = sTmpQ2.m_pMatrix;
    STmpMatrix sTmpM = GetTmpMatrix();
    CLGComplex* tmpM = sTmpM.m_pMatrix;

    InitialZero(tmpR, dx, dx);
    BlockCopy(tmpR, T, dx, dy, dx, dy);
    InitialOne(tmpQ2, dx);
    dim3 block(1, 1, 1);
    for (UINT i = 0; i < dx - 1; ++i)
    {
        _kernelOneStepHouseHolderQR << <block, thread2 >> > (tmpQ, tmpR, i, dx);

        //m = q x r
        BlockMatrixMult(tmpM, tmpQ, tmpR, dx, i, dx, TRUE, FALSE, FALSE);

        //r = m
        checkCudaErrors(hipMemcpy(tmpR, tmpM, sizeof(CLGComplex) * dx * dx, hipMemcpyDeviceToDevice));

        //m = q2 * q
        BlockMatrixMult(tmpM, tmpQ2, tmpQ, dx, i, dx, FALSE, FALSE, TRUE);

        //q2 = m
        checkCudaErrors(hipMemcpy(tmpQ2, tmpM, sizeof(CLGComplex) * dx * dx, hipMemcpyDeviceToDevice));
    }
    
    //r=r, q=q2
    BlockCopy(R, tmpR, dy, dy, dy, dx);
    BlockCopy(Q, tmpQ2, dx, dy, dy, dx);
    sTmpQ.Free();
    sTmpM.Free();
    sTmpQ2.Free();
    sTmpR.Free();
}

#pragma endregion

#pragma region QR Hensenberg

__global__ void _CLG_LAUNCH_BOUND
_kernelOneStepHouseHolder(
    CLGComplex* U, 
    const CLGComplex* __restrict__ A, 
    UINT i, UINT dx)
{
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;

    __shared__ Real length;
    __shared__ Real lengthu;
    __shared__ CLGComplex u[CLinearAlgebraHelper::_kMaxSmallDim];
    if (0 == x && 0 == y)
    {
        length = F(0.0);
        lengthu = F(0.0);
    }

    __syncthreads();

    if (0 == y && x > i)
    {
        atomicAdd(&length, A[x * dx + i].x * A[x * dx + i].x + A[x * dx + i].y * A[x * dx + i].y);
    }

    __syncthreads();

    if (0 == x && 0 == y)
    {
        length = _sqrt(length);
    }

    __syncthreads();

    if (0 == y && x > i)
    {
        u[x] = A[x * dx + i];

        if (x == i + 1)
        {
            Real fuxL = __div(F(1.0), _sqrt(_cuCabsf(u[x])));
            Real fCos = u[x].x * fuxL;
            Real fSin = u[x].y * fuxL;
            u[x] = _cuCaddf(u[x], _make_cuComplex(length * fCos, length * fSin));
        }
        atomicAdd(&lengthu, u[x].x * u[x].x + u[x].y * u[x].y);
    }

    __syncthreads();

    if (0 == x && 0 == y)
    {
        lengthu = __div(F(1.0), _sqrt(lengthu * F(0.5)));
    }

    __syncthreads();

    if (0 == y && x > i)
    {
        u[x].x = u[x].x * lengthu;
        u[x].y = u[x].y * lengthu;
    }

    __syncthreads();

    //uk = A[i + 1->n, i] - |A[i+1]|
    if (x <= i || y <= i)
    {
        if (x == y)
        {
            U[x * dx + y] = _make_cuComplex(F(1.0), F(0.0));
        }
        else
        {
            U[x * dx + y] = _make_cuComplex(F(0.0), F(0.0));
        }
    }
    else
    {
        U[x * dx + y] = _cuCmulf(_cuConjf(u[y]), u[x]);
        U[x * dx + y].x = -U[x * dx + y].x;
        U[x * dx + y].y = -U[x * dx + y].y;
        if (x == y)
        {
            U[x * dx + y].x = U[x * dx + y].x + F(1.0);
        }
    }
}

void CLinearAlgebraHelper::Henssenberg(CLGComplex* T, UINT dx)
{
    if (dx > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }
    dim3 block(1, 1, 1);
    dim3 thread1(dx, dx, 1);
    STmpMatrix sTmpU = GetTmpMatrix();
    CLGComplex* tmpU = sTmpU.m_pMatrix;
    STmpMatrix sTmpM = GetTmpMatrix();
    CLGComplex* tmpM = sTmpM.m_pMatrix;

    for (UINT i = 0; i < dx - 2; ++i)
    {
        _kernelOneStepHouseHolder << <block, thread1 >> > (tmpU, T, i, dx);
        BlockMatrixMult(tmpM, tmpU, T, dx, i + 1, dx, TRUE, FALSE, FALSE);
        BlockMatrixMult(T, tmpM, tmpU, dx, i + 1, dx, FALSE, FALSE, TRUE);
    }
    sTmpU.Free();
    sTmpM.Free();
}

#pragma endregion

#pragma region Back Shift

__global__ void _CLG_LAUNCH_BOUND
_kernelOneLineReduceBS(
    CLGComplex* y, 
    const CLGComplex* __restrict__ R, 
    UINT i, UINT dk, UINT dx)
{
    UINT j = threadIdx.x + i + 1; //j=i+1 to dx
    UINT n = threadIdx.y;

    if (j < dx)
    {
        CLGComplex toAdd = _cuCmulf(R[i * dx + j], y[j * dk + n]);
        atomicAdd(&y[i * dk + n].x, -toAdd.x);
        atomicAdd(&y[i * dk + n].y, -toAdd.y);
    }

    __syncthreads();

    //i + 1 == j means threadIdx = 0
    if (i + 1 == j)
    {
        y[i * dk + n] = _cuCdivf(y[i * dk + n], R[i * dx + i]);
    }
}

void CLinearAlgebraHelper::SolveY(CLGComplex* deviceY, const CLGComplex* deviceR, UINT dk, UINT dx)
{
    dim3 block(1, 1, 1);
    for (INT i = dx - 1; i >= 0; --i)
    {
        if (i == static_cast<INT>(dx - 1))
        {
            dim3 thread(1, dk, 1);
            _kernelOneLineReduceBS << <block, thread >> > (deviceY, deviceR, i, dk, dx);
        }
        else
        {
            dim3 thread(dx - i - 1, dk, 1);
            _kernelOneLineReduceBS << <block, thread >> > (deviceY, deviceR, i, dk, dx);
        }
    }
}


#pragma endregion

#pragma region Shift QR Iteration

__global__ void _CLG_LAUNCH_BOUND
_kernelWilkinsonShift(CLGComplex* m, CLGComplex* c, UINT dim)
{
    UINT i = threadIdx.x;
    if (0 == i)
    {
        //d
        c[0] = m[dim * dim - 1];

        //bc
        CLGComplex omega = _cuCmulf(m[dim * dim - dim - 1], m[dim * dim - 2]);

        Real fOmegaSq = __cuCabsSqf(omega);
        if (fOmegaSq > F(0.0000001))
        {
            //(d-a)/2
            CLGComplex xi = _make_cuComplex(
                F(0.5) * (c[0].x - m[dim * dim - dim - 2].x),
                F(0.5) * (c[0].y - m[dim * dim - dim - 2].y));

            //sqrt(((d-a)/2)^2 + bc)
            CLGComplex eta = __cuCsqrtf(_cuCaddf(_cuCmulf(xi, xi), omega));
            if (xi.x * eta.x + xi.y * eta.y < F(0.0))
            {
                c[0] = _cuCsubf(c[0], _cuCdivf(omega, _cuCsubf(eta, xi)));
            }
            else
            {
                c[0] = _cuCaddf(c[0], _cuCdivf(omega, _cuCaddf(eta, xi)));
            }
        }
    }

    __syncthreads();

    m[i * dim + i] = _cuCsubf(m[i * dim + i], c[0]);
}

__global__ void _CLG_LAUNCH_BOUND_SINGLE
_kernelCheckMatrix(CLGComplex* mtr, INT* decomp, UINT dx, Real fCrit)
{
    decomp[0] = dx;
    for (INT i = dx - 2; i >= 0; --i)
    {
        if (__cuCabsSqf(mtr[(i + 1) * dx + i]) < 
            fCrit * (__cuCabsSqf(mtr[(i + 1) * dx + i + 1]) + __cuCabsSqf(mtr[i * dx + i])))
        {
            mtr[(i + 1) * dx + i].x = F(0.0);
            mtr[(i + 1) * dx + i].y = F(0.0);

            if (decomp[0] == i + 2)
            {
                decomp[0] = i + 1;
            }
        }
    }
}

void CLinearAlgebraHelper::QRIterate(CLGComplex* T, UINT dx, Real fCrit, UINT iCrit)
{
    if (dx > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpQ = GetTmpMatrix();
    CLGComplex* tmpQ = sTmpQ.m_pMatrix;
    STmpMatrix sTmpR = GetTmpMatrix();
    CLGComplex* tmpR = sTmpR.m_pMatrix;
    STmpMatrix sTmpT = GetTmpMatrix();
    CLGComplex* tmpT = sTmpT.m_pMatrix;

    CLGComplex* tmpShift = m_pDeviceComplexBuffer1;
    INT* tmpDecomp = m_pDeviceIntBuffer;
    
    dim3 block(1, 1, 1);
    dim3 thread1(dx, dx, 1);
    INT endindex[1];

    checkCudaErrors(hipMemcpy(tmpT, T, sizeof(CLGComplex) * dx * dx, hipMemcpyDeviceToDevice));

    UINT iLastDim = dx;
    for (UINT i = 0; i < iCrit; ++i)
    {
        //find decomp
        _kernelCheckMatrix << <1, 1 >> > (tmpT, tmpDecomp, iLastDim, fCrit);
        checkCudaErrors(hipMemcpy(endindex, tmpDecomp, sizeof(INT), hipMemcpyDeviceToHost));

        if (endindex[0] < static_cast<INT>(iLastDim))
        {
            //copy matrix
            BlockCopy(T, tmpT, iLastDim, iLastDim, dx, iLastDim);
            if (1 == endindex[0])
            {
                //finished
                sTmpQ.Free();
                sTmpR.Free();
                sTmpT.Free();

                return;
            }

            iLastDim = endindex[0];
            //must do the copy, because [x * dim + y] the "dim" is changed.
            BlockCopy(tmpT, T, iLastDim, iLastDim, iLastDim, dx);
        }

        //shift
        //T = T - sigma I, tmpDeviceFloat[0] = sigma
        dim3 thread(iLastDim, 1, 1);
        _kernelWilkinsonShift << <block, thread >> > (tmpT, tmpShift, iLastDim);

        //QR decompose
        QRFactorization(tmpQ, tmpR, tmpT, iLastDim);

        //Update H
        //T = R Q + sigma I
        SmallMatrixMult(tmpT, tmpR, tmpQ, iLastDim, iLastDim, iLastDim, FALSE, FALSE);
        _kernelMatrixAddConstant << <block, thread >> > (tmpT, tmpShift, iLastDim);
    }

    sTmpQ.Free();
    sTmpR.Free();
    sTmpT.Free();
}

#pragma endregion

#pragma region Eigen Problem

__global__ void _CLG_LAUNCH_BOUND
_kernelSortEigenValues(const CLGComplex* __restrict__ R,
    CLGComplex* outV, Real* tmpF, INT* tmpO, UINT k, UINT dx)
{
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;

    if (0 == x)
    {
        tmpF[y] = R[y * dx + y].x * R[y * dx + y].x + R[y * dx + y].y * R[y * dx + y].y;
        tmpO[y] = 0;
    }

    __syncthreads();

    if (x != y)
    {
        if (tmpF[x] < tmpF[y])
        {
            atomicAdd(&tmpO[y], 1);
        }
    }

    __syncthreads();

    if (0 == x)
    {
        if (tmpO[y] < k)
        {
            outV[tmpO[y]] = R[y * dx + y];
        }
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelDaggerVector(CLGComplex* y, const CLGComplex* __restrict__ Q, UINT dx)
{
    UINT j = threadIdx.x;
    y[j] = hipConjf(Q[j * dx]);
}

__global__ void _CLG_LAUNCH_BOUND
_kernelInverseIterateShift(CLGComplex* A, const CLGComplex* __restrict__ outV, UINT k, UINT dx)
{
    UINT x = threadIdx.x;
    A[x * dx + x] = hipCsubf(A[x * dx + x], outV[k]);
}

__global__ void _CLG_LAUNCH_BOUND
_kernelNormVectors(CLGComplex* v, UINT dx)
{
    UINT x = threadIdx.y;
    UINT y = threadIdx.x;
    __shared__ Real fAmp[CLinearAlgebraHelper::_kMaxSmallDim];
    if (0 == y)
    {
        fAmp[x] = F(0.0);
    }

    __syncthreads();

    atomicAdd(&fAmp[x], v[x * dx + y].x * v[x * dx + y].x + v[x * dx + y].y * v[x * dx + y].y);

    __syncthreads();

    if (0 == y)
    {
        fAmp[x] = __div(F(1.0), _sqrt(fAmp[x]));
    }

    v[x * dx + y].x = v[x * dx + y].x * fAmp[x];
    v[x * dx + y].y = v[x * dx + y].y * fAmp[x];
}

__global__ void _CLG_LAUNCH_BOUND
_kernelErrorCheck(Real* outE, CLGComplex* v, const CLGComplex* __restrict__ A, UINT dx)
{
    UINT x = threadIdx.x;
    UINT y = threadIdx.y;

    __shared__ Real length;
    __shared__ CLGComplex afterMult[CLinearAlgebraHelper::_kMaxSmallDim];

    if (0 == x && 0 == y)
    {
        length = F(0.0);
    }

    __syncthreads();

    if (0 == x)
    {
        atomicAdd(&length, v[y].x * v[y].x + v[y].y * v[y].y);
        afterMult[y] = _make_cuComplex(F(0.0), F(0.0));
    }

    __syncthreads();

    if (0 == x && 0 == y)
    {
        length = __div(F(1.0), _sqrt(length));
    }

    __syncthreads();

    if (0 == x)
    {
        v[y].x = v[y].x * length;
        v[y].y = v[y].y * length;
    }

    __syncthreads();

    hipComplex toAdd = hipCmulf(A[x * dx + y], v[y]);
    atomicAdd(&afterMult[x].x, toAdd.x);
    atomicAdd(&afterMult[x].y, toAdd.y);

    __syncthreads();

    if (0 == x)
    {
        atomicAdd(outE, afterMult[y].x * afterMult[y].x + afterMult[y].y * afterMult[y].y);
    }
}

void CLinearAlgebraHelper::EigenValueProblem(
    CLGComplex* H,
    CLGComplex* outEigenValue,
    CLGComplex* outEigenVector,
    UINT dm, UINT dk,
    Real fEigenCrit,
    UINT iMaxEigenIterate,
    Real fCrit,
    UINT iMaxIterate)
{
    if (dm > m_uiDim || dk > dm)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d! or required eigen vector number larger than dimension!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpH = GetTmpMatrix();
    CLGComplex* tmpH = sTmpH.m_pMatrix;

    CLGComplex* tmpVector = m_pDeviceComplexBuffer1;

    //preserve H for solve eigen vectors
    checkCudaErrors(hipMemcpy(tmpH, H, sizeof(CLGComplex) * dm * dm, hipMemcpyDeviceToDevice));

    Henssenberg(tmpH, dm);
    QRIterate(tmpH, dm, fCrit, iMaxIterate);

    STmpMatrix sTmpQ = GetTmpMatrix();
    CLGComplex* tmpQ = sTmpQ.m_pMatrix;
    STmpMatrix sTmpR = GetTmpMatrix();
    CLGComplex* tmpR = sTmpR.m_pMatrix;

    dim3 block(1, 1, 1);
    dim3 thread1(dm, dm, 1);
    dim3 thread2(dm, 1, 1);

    _kernelSortEigenValues << <block, thread1 >> > (tmpH, outEigenValue, m_pDeviceFloatBuffer, m_pDeviceIntBuffer, dk, dm);

    for (UINT i = 0; i < dk; ++i)
    {
        //Inverse Iterate
        checkCudaErrors(hipMemcpy(tmpH, H, sizeof(CLGComplex) * dm * dm, hipMemcpyDeviceToDevice));
        _kernelInverseIterateShift << <block, thread2 >> > (tmpH, outEigenValue, i, dm);

        QRFactorization(tmpQ, tmpR, tmpH, dm);

        //q=tmpM2, r=tmpM3
        _kernelDaggerVector << <block, thread2 >> > (tmpVector, tmpQ, dm);
        SolveY(tmpVector, tmpR, 1, dm);

        // Sometimes One Iteration is NOT enough!
        Real fErr[1];
        Real* tmpF = m_pDeviceFloatBuffer;
        STmpMatrix sTmpM = GetTmpMatrix();
        CLGComplex* tmpM = sTmpM.m_pMatrix;
        for (UINT j = 0; j < iMaxEigenIterate; ++j)
        {
            fErr[0] = F(0.0);
            checkCudaErrors(hipMemcpy(tmpF, fErr, sizeof(Real), hipMemcpyHostToDevice));

            _kernelErrorCheck << <block, thread1 >> > (tmpF, tmpVector, tmpH, dm);

            checkCudaErrors(hipMemcpy(fErr, tmpF, sizeof(float), hipMemcpyDeviceToHost));

            if (j == iMaxEigenIterate - 1)
            {
                appParanoiac(_T("(* Eigen vector(%d-iterate %d) error now = %1.12f *)\n"), i + 1, j, fErr[0]);
            }

            if (fErr[0] < fEigenCrit)
            {
                break;
            }
            SmallMatrixMult(tmpM, tmpQ, tmpVector, dm, dm, 1, TRUE, FALSE);
            SolveY(tmpM, tmpR, 1, dm);
            checkCudaErrors(hipMemcpy(tmpVector, tmpM, sizeof(CLGComplex) * dm, hipMemcpyDeviceToDevice));
        }

        checkCudaErrors(hipMemcpy(outEigenVector + dm * i, tmpVector, sizeof(CLGComplex) * dm, hipMemcpyDeviceToDevice));
    }

    //It is normalized in _kernelErrorCheck
    //dim3 thread3(dm, dk, 1);
    //_kernelNormVectors << <block, thread3 >> > (outEigenVector, dm);

    sTmpH.Free();
    sTmpQ.Free();
    sTmpR.Free();
}

void CLinearAlgebraHelper::EigenValueProblemHessenberg(
    CLGComplex* H, CLGComplex* outEigenValue, CLGComplex* outEigenVector,
    UINT dm, UINT dk, Real fEigenCrit, UINT iMaxEigenIterate,
    Real fQRCrit, UINT iMaxIterate)
{
    if (dm > m_uiDim || dk > dm)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d! or required eigen vector number larger than dimension!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpH = GetTmpMatrix();
    CLGComplex* tmpH = sTmpH.m_pMatrix;

    CLGComplex* tmpVector = m_pDeviceComplexBuffer1;

    //preserve H for solve eigen vectors
    checkCudaErrors(hipMemcpy(tmpH, H, sizeof(CLGComplex) * dm * dm, hipMemcpyDeviceToDevice));

    QRIterate(tmpH, dm, fQRCrit, iMaxIterate);

    STmpMatrix sTmpQ = GetTmpMatrix();
    CLGComplex* tmpQ = sTmpQ.m_pMatrix;
    STmpMatrix sTmpR = GetTmpMatrix();
    CLGComplex* tmpR = sTmpR.m_pMatrix;

    dim3 block(1, 1, 1);
    dim3 thread1(dm, dm, 1);
    dim3 thread2(dm, 1, 1);

    _kernelSortEigenValues << <block, thread1 >> > (tmpH, outEigenValue, m_pDeviceFloatBuffer, m_pDeviceIntBuffer, dk, dm);

    for (UINT i = 0; i < dk; ++i)
    {
        //Inverse Iterate
        checkCudaErrors(hipMemcpy(tmpH, H, sizeof(CLGComplex) * dm * dm, hipMemcpyDeviceToDevice));
        _kernelInverseIterateShift << <block, thread2 >> > (tmpH, outEigenValue, i, dm);

        QRFactorization(tmpQ, tmpR, tmpH, dm);

        //q=tmpM2, r=tmpM3
        _kernelDaggerVector << <block, thread2 >> > (tmpVector, tmpQ, dm);
        SolveY(tmpVector, tmpR, 1, dm);

        // Sometimes One Iteration is NOT enough!
        Real fErr[1];
        Real* tmpF = m_pDeviceFloatBuffer;
        STmpMatrix sTmpM = GetTmpMatrix();
        CLGComplex* tmpM = sTmpM.m_pMatrix;
        for (UINT j = 0; j < iMaxEigenIterate; ++j)
        {
            fErr[0] = F(0.0);
            checkCudaErrors(hipMemcpy(tmpF, fErr, sizeof(Real), hipMemcpyHostToDevice));

            _kernelErrorCheck << <block, thread1 >> > (tmpF, tmpVector, tmpH, dm);

            checkCudaErrors(hipMemcpy(fErr, tmpF, sizeof(float), hipMemcpyDeviceToHost));

            if (j == iMaxEigenIterate - 1)
            {
                appParanoiac(_T("(* Eigen vector(%d-iterate %d) error now = %1.12f *)\n"), i + 1, j, fErr[0]);
            }

            if (fErr[0] < fEigenCrit)
            {
                break;
            }
            SmallMatrixMult(tmpM, tmpQ, tmpVector, dm, dm, 1, TRUE, FALSE);
            SolveY(tmpM, tmpR, 1, dm);
            checkCudaErrors(hipMemcpy(tmpVector, tmpM, sizeof(CLGComplex) * dm, hipMemcpyDeviceToDevice));
        }

        checkCudaErrors(hipMemcpy(outEigenVector + dm * i, tmpVector, sizeof(CLGComplex) * dm, hipMemcpyDeviceToDevice));
    }

    //It is normalized in _kernelErrorCheck
    //dim3 thread3(dm, dk, 1);
    //_kernelNormVectors << <block, thread3 >> > (outEigenVector, dm);

    sTmpH.Free();
    sTmpQ.Free();
    sTmpR.Free();
}

__global__ void _CLG_LAUNCH_BOUND
_kernelExchangeOrders(INT* orders)
{
    UINT x = threadIdx.x; //0 to dm
    UINT y = threadIdx.y; //0 to dk
    __shared__ INT outOrders[CLinearAlgebraHelper::_kMaxSmallDim];

    if (y == orders[x])
    {
        //if 0 == orders[3], outOrders[3] = 0
        outOrders[y] = x;
    }

    __syncthreads();

    if (0 == x)
    {
        orders[y] = outOrders[y];
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelInitialE1Vector(CLGComplex* v, UINT dm, UINT x)
{
    UINT y = threadIdx.x;
    if (0 == y)
    {
        v[x * dm + y] = _make_cuComplex(F(1.0), F(0.0));
    }
    else
    {
        v[x * dm + y] = _make_cuComplex(F(0.0), F(0.0));
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelCreateBackshiftProblem(
    const CLGComplex* __restrict__ triangular, const CLGComplex* __restrict__ eigenValue,
    UINT i, UINT iOrder, //the k row, iOrder = k - 1
    UINT dm,
    CLGComplex* resultMatrixR, CLGComplex* resultVecotr)
{
    UINT x = threadIdx.x; //0 -> iOrder - 1
    UINT y = threadIdx.y; //0 -> iOrder - 1

    if (y < x)
    {
        resultMatrixR[x * iOrder + y] = _make_cuComplex(F(0.0), F(0.0));
    }
    else if (y == x)
    {
        resultMatrixR[x * iOrder + y] = _cuCsubf(triangular[x * dm + y], eigenValue[i]);
    }
    else
    {
        resultMatrixR[x * iOrder + y] = triangular[x * dm + y];
    }

    if (0 == y)
    {
        resultVecotr[x].x = -triangular[x * dm + iOrder].x;
        resultVecotr[x].y = -triangular[x * dm + iOrder].y;
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelFinalNorm(
    const CLGComplex* __restrict__ triangular,
    UINT iOrder, //the k row, iOrder = k - 1
    UINT dm,
    CLGComplex* resultVecotr)
{
    UINT x = threadIdx.x;
    __shared__ Real fLength;
    if (1 == iOrder)
    {
        if (0 == x)
        {
            //norm of
            //- r[0,1]/(r[0,0] - r[1,1])
            CLGComplex a = _cuCdivf(triangular[1], _cuCsubf(triangular[dm + 1], triangular[0]));
            fLength = __div(F(1.0), _sqrt(__cuCabsSqf(a) + F(1.0)));
            resultVecotr[0] = a;
            resultVecotr[0].x = resultVecotr[0].x * fLength;
            resultVecotr[0].y = resultVecotr[0].y * fLength;
            resultVecotr[1] = _make_cuComplex(fLength, F(0.0));
        }
        else if (x > 1)
        {
            resultVecotr[x] = _make_cuComplex(F(0.0), F(0.0));
        }
    }
    else
    {
        if (0 == x)
        {
            fLength = F(0.0);
        }

        __syncthreads();

        if (x < iOrder)
        {
            atomicAdd(&fLength, __cuCabsSqf(resultVecotr[x]));
        }

        __syncthreads();

        if (0 == x)
        {
            fLength = __div(F(1.0), _sqrt(fLength + F(1.0)));
        }

        __syncthreads();

        if (x < iOrder)
        {
            resultVecotr[x].x = resultVecotr[x].x * fLength;
            resultVecotr[x].y = resultVecotr[x].y * fLength;
        }
        else if (x == iOrder)
        {
            resultVecotr[x] = _make_cuComplex(fLength, F(0.0));
        }
        else
        {
            resultVecotr[x] = _make_cuComplex(F(0.0), F(0.0));
        }
    }
}

void CLinearAlgebraHelper::UpperTriangularEigenVectors(
    const CLGComplex* upperTriangular, CLGComplex* outEigenValue, CLGComplex* outEigenVector,
    UINT dm, UINT dk)
{
    if (dm > m_uiDim || dk > dm)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d! or required eigen vector number larger than dimension!\n"), m_uiDim);
        return;
    }

    dim3 block(1, 1, 1);
    dim3 thread1(dm, dm, 1);
    dim3 thread2(dm, 1, 1);
    dim3 thread3(dm, dk, 1);

    _kernelSortEigenValues << <block, thread1 >> > (upperTriangular, outEigenValue, m_pDeviceFloatBuffer, m_pDeviceIntBuffer, dk, dm);
    _kernelExchangeOrders << <block, thread3 >>> (m_pDeviceIntBuffer);
    INT orders[_kMaxSmallDim];
    checkCudaErrors(hipMemcpy(orders, m_pDeviceIntBuffer, sizeof(INT) * dk, hipMemcpyDeviceToHost));

    CLGComplex* tmpVector = m_pDeviceComplexBuffer1;
    STmpMatrix sTmpR = GetTmpMatrix();
    CLGComplex* tmpR = sTmpR.m_pMatrix;

    for (UINT i = 0; i < dk; ++i)
    {
        if (0 == orders[i])
        {
            //usually it is not 0
            _kernelInitialE1Vector << <block, thread2 >> > (outEigenVector, dm, i);
        }
        else if (1 == orders[i])
        {
            _kernelFinalNorm << <block, thread2 >> > (upperTriangular, 1, dm, tmpVector);
            checkCudaErrors(hipMemcpy(outEigenVector + i * dm, tmpVector, sizeof(CLGComplex) * dm, hipMemcpyDeviceToDevice));
        }
        else
        {
            //create back shift problem
            UINT toSolveDim = orders[i];
            //if it is the number 2 eigen-value, it is the 3rd eigen-value
            //when 3rd eigen-value, we need a 2x2 matrix.
            dim3 thread4(toSolveDim, toSolveDim, 1);
            _kernelCreateBackshiftProblem << <block, thread4 >> > (upperTriangular, outEigenValue, 
                i, toSolveDim, dm, tmpR, tmpVector);

            //solve back shift
            SolveY(tmpVector, tmpR, 1, toSolveDim);

            _kernelFinalNorm << <block, thread2 >> > (upperTriangular, toSolveDim, dm, tmpVector);

            checkCudaErrors(hipMemcpy(outEigenVector + i * dm, tmpVector, sizeof(CLGComplex) * dm, hipMemcpyDeviceToDevice));
        }
    }

    sTmpR.Free();
}

#pragma endregion

#pragma region Generalized Eigen Problem

void CLinearAlgebraHelper::GeneralizedEigenValueProblem(
    CLGComplex* A,
    CLGComplex* B,
    CLGComplex* outEigenValue,
    CLGComplex* outEigenVector,
    UINT dm, UINT dk,
    Real fEigenCrit,
    UINT iMaxEigenIterate,
    Real fCrit,
    UINT iMaxIterate)
{
    if (dm > m_uiDim || dk > dm)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d! or required eigen vector number larger than dimension!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpQ = GetTmpMatrix();
    CLGComplex* tmpQ = sTmpQ.m_pMatrix;
    STmpMatrix sTmpR = GetTmpMatrix();
    CLGComplex* tmpR = sTmpR.m_pMatrix;

    QRFactorization(tmpQ, tmpR, B, dm);
    SmallMatrixMult(B, tmpQ, A, dm, dm, dm, TRUE, FALSE);
    SolveY(B, tmpR, dm, dm);

    sTmpQ.Free();
    sTmpR.Free();

    EigenValueProblem(B, outEigenValue, outEigenVector, dm, dk, fEigenCrit, iMaxEigenIterate, fCrit, iMaxIterate);
}

#pragma endregion

#pragma region Givens

/**
* Left Given
*
* AX=B
* A'X=GAX=GB
* where A'[i-1, i] is zeroed.
* A is assumed to be a Henssenberg
*
* j from 0 to n-3
* i from n-1 to j+1
*
*
* left:
*
* h00* h10*   h00 h01  =  +  +
* -h10 h00    h10 h11     0  +
*
* right:
* h00 h01   h11 h10*   = +  +
* h10 h11  -h10 h11*     0  +
*
* A = dm x dmA, B = dm x 1
*
* thread.x = dmA - j
* thread.y = dm
*
*/
__global__ void _CLG_LAUNCH_BOUND
_kernelLeftGivenHessenberg(UINT i, UINT j,
    CLGComplex* A, CLGComplex* g, UINT dm)
{
    //A is Henssenberg, so no need to calculate all 
    //UINT affectedDimA = dm - j;
    UINT x = threadIdx.x;
    
    __shared__ CLGComplex lineAi[CLinearAlgebraHelper::_kMaxSmallDim];
    __shared__ CLGComplex lineAj[CLinearAlgebraHelper::_kMaxSmallDim];
    __shared__ CLGComplex c0;
    __shared__ CLGComplex s0;
    __shared__ CLGComplex c0h;
    __shared__ CLGComplex s0h;

    lineAi[x] = A[(i - 1) * dm + j + x];
    lineAj[x] = A[i * dm + j + x];

    if (0 == x)
    {
        CLGComplex h00 = A[(i - 1) * dm + j];
        CLGComplex h10 = A[i * dm + j];
        Real fDemon = __div(F(1.0), _sqrt(h00.x * h00.x + h00.y * h00.y + h10.x * h10.x + h10.y * h10.y));
        c0.x = h00.x * fDemon;
        c0.y = h00.y * fDemon;
        s0.x = h10.x * fDemon;
        s0.y = h10.y * fDemon;
        c0h = _cuConjf(c0);
        s0h = _cuConjf(s0);

        //  c0h s0h
        //  -s0 c0
        CLGComplex g_im1 = g[i - 1];
        g[i - 1] = _cuCaddf(_cuCmulf(c0h, g[i - 1]), _cuCmulf(s0h, g[i]));
        g[i] = _cuCsubf(_cuCmulf(c0, g[i]), _cuCmulf(s0, g_im1));
    }

    __syncthreads();

    A[(i - 1) * dm + x + j] = _cuCaddf(_cuCmulf(c0h, lineAi[x]), _cuCmulf(s0h, lineAj[x]));
    A[i * dm + x + j] = _cuCsubf(_cuCmulf(c0, lineAj[x]), _cuCmulf(s0, lineAi[x]));
}

void CLinearAlgebraHelper::RotateHenssenberg(CLGComplex* H, CLGComplex* Y, UINT dm)
{
    dim3 block(1, 1, 1);
    for (UINT i = 0; i < dm; ++i)
    {
        dim3 thread(dm - i, 1, 1);
        _kernelLeftGivenHessenberg << <block, thread >> > (i + 1, i, H, Y, dm);
    }
}

#pragma endregion

#pragma region Host Functions

void CLinearAlgebraHelper::InitialZeroHost(CLGComplex* hostMatrix, UINT dx, UINT dy)
{
    if (dx > m_uiDim || dy > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpM = GetTmpMatrix();
    CLGComplex* tmpM = sTmpM.m_pMatrix;
    checkCudaErrors(hipMemcpy(tmpM, hostMatrix, sizeof(CLGComplex) * dx * dy, hipMemcpyHostToDevice));
    InitialZero(tmpM, dx, dy);
    checkCudaErrors(hipMemcpy(hostMatrix, tmpM, sizeof(CLGComplex) * dx * dy, hipMemcpyDeviceToHost));
    sTmpM.Free();
}

void CLinearAlgebraHelper::InitialOneHost(CLGComplex* hostMatrix, UINT dx)
{
    if (dx > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpM = GetTmpMatrix();
    CLGComplex* tmpM = sTmpM.m_pMatrix;
    checkCudaErrors(hipMemcpy(tmpM, hostMatrix, sizeof(CLGComplex) * dx * dx, hipMemcpyHostToDevice));
    InitialOne(tmpM, dx);
    checkCudaErrors(hipMemcpy(hostMatrix, tmpM, sizeof(CLGComplex) * dx * dx, hipMemcpyDeviceToHost));
    sTmpM.Free();
}

void CLinearAlgebraHelper::SmallMatrixMultHost(
    CLGComplex * hostRes,
    const CLGComplex* left,
    const CLGComplex* right,
    UINT dLeft, UINT dMid, UINT dRight,
    UBOOL bLeftDagger, UBOOL bRightDagger)
{
    if (dLeft > m_uiDim || dMid > m_uiDim || dRight > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpMRes = GetTmpMatrix();
    CLGComplex* tmpMRes = sTmpMRes.m_pMatrix;
    STmpMatrix sTmpMLeft = GetTmpMatrix();
    CLGComplex* tmpMLeft = sTmpMLeft.m_pMatrix;
    STmpMatrix sTmpMRight = GetTmpMatrix();
    CLGComplex* tmpMRight = sTmpMRight.m_pMatrix;

    checkCudaErrors(hipMemcpy(tmpMLeft, left, sizeof(CLGComplex) * dLeft * dMid, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(tmpMRight, right, sizeof(CLGComplex) * dMid * dRight, hipMemcpyHostToDevice));

    SmallMatrixMult(
        tmpMRes, tmpMLeft, tmpMRight,
        dLeft, dMid, dRight,
        bLeftDagger, bRightDagger);

    checkCudaErrors(hipMemcpy(hostRes, tmpMRes, sizeof(CLGComplex) * dLeft * dRight, hipMemcpyDeviceToHost));
    sTmpMRes.Free();
    sTmpMLeft.Free();
    sTmpMRight.Free();
}

void CLinearAlgebraHelper::BlockMatrixMultHost(
    CLGComplex * hostRes,
    const CLGComplex* left,
    const CLGComplex* right,
    UINT dDim, UINT uiStart, UINT uiEnd,
    UBOOL bLeft, UBOOL bLeftDagger, UBOOL bRightDagger)
{
    if (dDim > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpMRes = GetTmpMatrix();
    CLGComplex* tmpMRes = sTmpMRes.m_pMatrix;
    STmpMatrix sTmpMLeft = GetTmpMatrix();
    CLGComplex* tmpMLeft = sTmpMLeft.m_pMatrix;
    STmpMatrix sTmpMRight = GetTmpMatrix();
    CLGComplex* tmpMRight = sTmpMRight.m_pMatrix;

    checkCudaErrors(hipMemcpy(tmpMLeft, left, sizeof(CLGComplex) * dDim * dDim, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(tmpMRight, right, sizeof(CLGComplex) * dDim * dDim, hipMemcpyHostToDevice));

    BlockMatrixMult(
        tmpMRes, tmpMLeft, tmpMRight,
        dDim, uiStart, uiEnd,
        bLeft, bLeftDagger, bRightDagger);

    checkCudaErrors(hipMemcpy(hostRes, tmpMRes, sizeof(CLGComplex) * dDim * dDim, hipMemcpyDeviceToHost));
    sTmpMRes.Free();
    sTmpMLeft.Free();
    sTmpMRight.Free();
}

void CLinearAlgebraHelper::BlockCopyHost(CLGComplex* hostDest, const CLGComplex* hostSrc,
    UINT lengthX, UINT lengthY, UINT dimDest, UINT dimSrc)
{
    if (lengthX > m_uiDim || lengthY > m_uiDim || dimDest > m_uiDim || dimSrc > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpMRes = GetTmpMatrix();
    CLGComplex* tmpMRes = sTmpMRes.m_pMatrix;
    STmpMatrix sTmpMSrc = GetTmpMatrix();
    CLGComplex* tmpMSrc = sTmpMSrc.m_pMatrix;

    checkCudaErrors(hipMemcpy(tmpMSrc, hostSrc, sizeof(CLGComplex) * dimSrc * lengthY, hipMemcpyHostToDevice));
    //copy the unchanged elements
    checkCudaErrors(hipMemcpy(tmpMRes, hostDest, sizeof(CLGComplex) * dimDest * lengthY, hipMemcpyHostToDevice));

    BlockCopy(tmpMRes, tmpMSrc, lengthX, lengthY, dimDest, dimSrc);

    checkCudaErrors(hipMemcpy(hostDest, tmpMRes, sizeof(CLGComplex) * dimDest * lengthY, hipMemcpyDeviceToHost));

    sTmpMRes.Free();
    sTmpMSrc.Free();
}

void CLinearAlgebraHelper::QRFactorizationHost(CLGComplex* Q, CLGComplex* R, const CLGComplex* T, UINT uiDim)
{
    if (uiDim > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpQ = GetTmpMatrix();
    CLGComplex* tmpQ = sTmpQ.m_pMatrix;
    STmpMatrix sTmpR = GetTmpMatrix();
    CLGComplex* tmpR = sTmpR.m_pMatrix;
    STmpMatrix sTmpT = GetTmpMatrix();
    CLGComplex* tmpT = sTmpT.m_pMatrix;

    checkCudaErrors(hipMemcpy(tmpT, T, sizeof(CLGComplex) * uiDim * uiDim, hipMemcpyHostToDevice));

    QRFactorization(tmpQ, tmpR, tmpT, uiDim);

    checkCudaErrors(hipMemcpy(Q, tmpQ, sizeof(CLGComplex) * uiDim * uiDim, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(R, tmpR, sizeof(CLGComplex) * uiDim * uiDim, hipMemcpyDeviceToHost));

    sTmpQ.Free();
    sTmpR.Free();
    sTmpT.Free();
}

void CLinearAlgebraHelper::ThinQRFactorizationHost(CLGComplex* Q, CLGComplex* R, const CLGComplex* T, UINT dx, UINT dy)
{
    if (dx > m_uiDim || dy > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpQ = GetTmpMatrix();
    CLGComplex* tmpQ = sTmpQ.m_pMatrix;
    STmpMatrix sTmpR = GetTmpMatrix();
    CLGComplex* tmpR = sTmpR.m_pMatrix;
    STmpMatrix sTmpT = GetTmpMatrix();
    CLGComplex* tmpT = sTmpT.m_pMatrix;

    checkCudaErrors(hipMemcpy(tmpT, T, sizeof(CLGComplex) * dx * dy, hipMemcpyHostToDevice));

    ThinQRFactorization(tmpQ, tmpR, tmpT, dx, dy);

    checkCudaErrors(hipMemcpy(Q, tmpQ, sizeof(CLGComplex) * dx * dy, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(R, tmpR, sizeof(CLGComplex) * dy * dy, hipMemcpyDeviceToHost));

    sTmpQ.Free();
    sTmpR.Free();
    sTmpT.Free();
}

void CLinearAlgebraHelper::SolveYHost(CLGComplex* Y, const CLGComplex* R, UINT dk, UINT dx)
{
    if (dx > m_uiDim || dk > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpY = GetTmpMatrix();
    CLGComplex* tmpY = sTmpY.m_pMatrix;
    STmpMatrix sTmpR = GetTmpMatrix();
    CLGComplex* tmpR = sTmpR.m_pMatrix;

    checkCudaErrors(hipMemcpy(tmpR, R, sizeof(CLGComplex) * dx * dx, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(tmpY, Y, sizeof(CLGComplex) * dx * dk, hipMemcpyHostToDevice));

    SolveY(tmpY, tmpR, dk, dx);

    checkCudaErrors(hipMemcpy(Y, tmpY, sizeof(CLGComplex) * dx * dk, hipMemcpyDeviceToHost));

    sTmpY.Free();
    sTmpR.Free();
}

void CLinearAlgebraHelper::UpperTriangularEigenVectorsHost(
    const CLGComplex* upperTriangular, CLGComplex* outEigenValue, CLGComplex* outEigenVector,
    UINT dm, UINT dk)
{
    if (dm > m_uiDim || dk > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpT = GetTmpMatrix();
    CLGComplex* tmpT = sTmpT.m_pMatrix;
    STmpMatrix sTmpE = GetTmpMatrix();
    CLGComplex* tmpE = sTmpE.m_pMatrix;
    STmpMatrix sTmpV = GetTmpMatrix();
    CLGComplex* tmpV = sTmpV.m_pMatrix;

    checkCudaErrors(hipMemcpy(tmpT, upperTriangular, sizeof(CLGComplex) * dm * dm, hipMemcpyHostToDevice));

    UpperTriangularEigenVectors(tmpT, tmpE, tmpV, dm, dk);

    checkCudaErrors(hipMemcpy(outEigenValue, tmpE, sizeof(CLGComplex) * dk, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(outEigenVector, tmpV, sizeof(CLGComplex) * dm * dk, hipMemcpyDeviceToHost));

    sTmpT.Free();
    sTmpE.Free();
    sTmpV.Free();
}

void CLinearAlgebraHelper::EigenValueProblemHost(CLGComplex* H, CLGComplex* outEigenValue, CLGComplex* outEigenVector,
    UINT dm, UINT dk, Real fEigenCrit, UINT iMaxEigenIter, Real fQRCrit, UINT iMaxIterate)
{
    if (dm > m_uiDim || dk > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpT = GetTmpMatrix();
    CLGComplex* tmpT = sTmpT.m_pMatrix;
    STmpMatrix sTmpE = GetTmpMatrix();
    CLGComplex* tmpE = sTmpE.m_pMatrix;
    STmpMatrix sTmpV = GetTmpMatrix();
    CLGComplex* tmpV = sTmpV.m_pMatrix;

    checkCudaErrors(hipMemcpy(tmpT, H, sizeof(CLGComplex) * dm * dm, hipMemcpyHostToDevice));

    EigenValueProblem(tmpT, tmpE, tmpV, dm, dk, fEigenCrit, iMaxEigenIter, fQRCrit, iMaxIterate);

    checkCudaErrors(hipMemcpy(outEigenValue, tmpE, sizeof(CLGComplex) * dk, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(outEigenVector, tmpV, sizeof(CLGComplex) * dm * dk, hipMemcpyDeviceToHost));

    sTmpT.Free();
    sTmpE.Free();
    sTmpV.Free();
}

void CLinearAlgebraHelper::EigenValueProblemHessenbergHost(CLGComplex* H, CLGComplex* outEigenValue, CLGComplex* outEigenVector,
    UINT dm, UINT dk, Real fEigenCrit, UINT iMaxEigenIter, Real fQRCrit, UINT iMaxIterate)
{
    if (dm > m_uiDim || dk > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpT = GetTmpMatrix();
    CLGComplex* tmpT = sTmpT.m_pMatrix;
    STmpMatrix sTmpE = GetTmpMatrix();
    CLGComplex* tmpE = sTmpE.m_pMatrix;
    STmpMatrix sTmpV = GetTmpMatrix();
    CLGComplex* tmpV = sTmpV.m_pMatrix;

    checkCudaErrors(hipMemcpy(tmpT, H, sizeof(CLGComplex) * dm * dm, hipMemcpyHostToDevice));

    EigenValueProblemHessenberg(tmpT, tmpE, tmpV, dm, dk, fEigenCrit, iMaxEigenIter, fQRCrit, iMaxIterate);

    checkCudaErrors(hipMemcpy(outEigenValue, tmpE, sizeof(CLGComplex) * dk, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(outEigenVector, tmpV, sizeof(CLGComplex) * dm * dk, hipMemcpyDeviceToHost));

    sTmpT.Free();
    sTmpE.Free();
    sTmpV.Free();
}

void CLinearAlgebraHelper::GeneralizedEigenValueProblemHost(
    CLGComplex* A, CLGComplex* B,
    CLGComplex* outEigenValue,
    CLGComplex* outEigenVector,
    UINT dm, UINT dk, Real fEigenCrit, UINT iMaxEigenIter, Real fQRCrit, UINT iMaxIterate)
{
    if (dm > m_uiDim || dk > m_uiDim)
    {
        appCrucial(_T("Cannot deal with matrix larger than %d!\n"), m_uiDim);
        return;
    }

    STmpMatrix sTmpA = GetTmpMatrix();
    CLGComplex* tmpA = sTmpA.m_pMatrix;
    STmpMatrix sTmpB = GetTmpMatrix();
    CLGComplex* tmpB = sTmpB.m_pMatrix;
    STmpMatrix sTmpE = GetTmpMatrix();
    CLGComplex* tmpE = sTmpE.m_pMatrix;
    STmpMatrix sTmpV = GetTmpMatrix();
    CLGComplex* tmpV = sTmpV.m_pMatrix;

    checkCudaErrors(hipMemcpy(tmpA, A, sizeof(CLGComplex) * dm * dm, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(tmpB, B, sizeof(CLGComplex) * dm * dm, hipMemcpyHostToDevice));

    GeneralizedEigenValueProblem(tmpA, tmpB, tmpE, tmpV, dm, dk, fEigenCrit, iMaxEigenIter, fQRCrit, iMaxIterate);

    checkCudaErrors(hipMemcpy(outEigenValue, tmpE, sizeof(CLGComplex) * dk, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(outEigenVector, tmpV, sizeof(CLGComplex) * dm * dk, hipMemcpyDeviceToHost));

    sTmpA.Free();
    sTmpB.Free();
    sTmpE.Free();
    sTmpV.Free();
}

void CLinearAlgebraHelper::RotateHenssenbergHost(CLGComplex* H, CLGComplex* Ye1, UINT dmH)
{
    STmpMatrix sTmpH = GetTmpMatrix();
    CLGComplex* tmpH = sTmpH.m_pMatrix;
    STmpMatrix sTmpY = GetTmpMatrix();
    CLGComplex* tmpY = sTmpY.m_pMatrix;

    checkCudaErrors(hipMemcpy(tmpH, H, sizeof(CLGComplex) * (dmH + 1) * dmH, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(tmpY, Ye1, sizeof(CLGComplex) * (dmH + 1), hipMemcpyHostToDevice));

    RotateHenssenberg(tmpH, tmpY, dmH);

    checkCudaErrors(hipMemcpy(H, tmpH, sizeof(CLGComplex) * (dmH + 1) * dmH, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(Ye1, tmpY, sizeof(CLGComplex) * (dmH + 1), hipMemcpyDeviceToHost));

    sTmpH.Free();
    sTmpY.Free();
}

#pragma endregion

__END_NAMESPACE

//=============================================================================
// END OF FILE
//=============================================================================
