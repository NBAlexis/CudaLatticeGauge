#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : Random.h
// 
// DESCRIPTION:
//
//
// REVISION:
//  [12/6/2018 nbale]
//=============================================================================
#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

__global__ void _CLG_LAUNCH_BOUND
_kernalAllocateSeedTable(UINT* pDevicePtr)
{
    intokernaldir;

    UINT uiSeed = _DC_Seed;

    for (UINT idir = 0; idir < uiDir + 1; ++idir)
    {
        UINT fatIndex = _deviceGetFatIndex(uiSiteIndex, idir);
        CRandom::_deviceAsignSeeds(pDevicePtr, uiSeed, fatIndex);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernalInitialXORWOW(hiprandState * states)
{
    UINT uiSiteIndex = ((threadIdx.x + blockIdx.x * blockDim.x) * blockDim.y * gridDim.y * blockDim.z * gridDim.z + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.z * gridDim.z + (threadIdx.z + blockIdx.z * blockDim.z));

    UINT uiSeed = _DC_Seed;
    UINT uiDir = _DC_Dir;
    for (UINT idir = 0; idir < uiDir + 1; ++idir)
    {
        UINT fatIndex = _deviceGetFatIndex(uiSiteIndex, idir);
        hiprand_init(uiSeed, fatIndex, 0, &states[fatIndex]);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernalInitialPhilox(hiprandStatePhilox4_32_10_t * states)
{
    UINT uiSiteIndex = ((threadIdx.x + blockIdx.x * blockDim.x) * blockDim.y * gridDim.y * blockDim.z * gridDim.z + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.z * gridDim.z + (threadIdx.z + blockIdx.z * blockDim.z));
    UINT uiSeed = _DC_Seed;
    UINT uiDir = _DC_Dir;
    for (UINT idir = 0; idir < uiDir + 1; ++idir)
    {
        UINT fatIndex = _deviceGetFatIndex(uiSiteIndex, idir);
        hiprand_init(uiSeed, fatIndex, 0, &states[fatIndex]);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernalInitialMRG(hiprandStateMRG32k3a  * states)
{
    UINT uiSiteIndex = ((threadIdx.x + blockIdx.x * blockDim.x) * blockDim.y * gridDim.y * blockDim.z * gridDim.z + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.z * gridDim.z + (threadIdx.z + blockIdx.z * blockDim.z));
    UINT uiSeed = _DC_Seed;
    UINT uiDir = _DC_Dir;
    for (UINT idir = 0; idir < uiDir + 1; ++idir)
    {
        UINT fatIndex = _deviceGetFatIndex(uiSiteIndex, idir);
        hiprand_init(uiSeed, fatIndex, 0, &states[fatIndex]);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernalInitialSobel32(hiprandStateSobol32* states, hiprandDirectionVectors32_t* dirs)
{
    intokernal;
    hiprand_init(dirs[uiSiteIndex], _DC_Seed % 16, &states[uiSiteIndex]);
}

__global__ void _CLG_LAUNCH_BOUND
_kernalInitialScrambledSobel32(hiprandStateScrambledSobol32* states, UINT* consts, hiprandDirectionVectors32_t* dirs)
{
    intokernal;
    hiprand_init(dirs[uiSiteIndex], consts[uiSiteIndex], _DC_Seed % __SOBEL_OFFSET_MAX, &states[uiSiteIndex]);
}

CRandom::~CRandom()
{

    switch (m_eRandomType)
    {
    case ER_Schrage:
        {
            checkCudaErrors(hipFree(m_pDeviceSeedTable));
        }
        break;
    case ER_MRG32K3A:
        {
            CURAND_CALL(hiprandDestroyGenerator(m_HGen));
            checkCudaErrors(hipFree(m_deviceBuffer));
            checkCudaErrors(hipFree(m_pDeviceRandStatesMRG));
        }
        break;
    case ER_PHILOX4_32_10:
        {
            CURAND_CALL(hiprandDestroyGenerator(m_HGen));
            checkCudaErrors(hipFree(m_deviceBuffer));
            checkCudaErrors(hipFree(m_pDeviceRandStatesPhilox));
        }
        break;
    case ER_QUASI_SOBOL32:
        {
            CURAND_CALL(hiprandDestroyGenerator(m_HGen));
            checkCudaErrors(hipFree(m_deviceBuffer));
            checkCudaErrors(hipFree(m_pDeviceRandStatesSobol32));
            checkCudaErrors(hipFree(m_pDeviceSobolDirVec));
        }
        break;
    case ER_SCRAMBLED_SOBOL32:
        {
            CURAND_CALL(hiprandDestroyGenerator(m_HGen));
            checkCudaErrors(hipFree(m_deviceBuffer));
            checkCudaErrors(hipFree(m_pDeviceRandStatesScrambledSobol32));
            checkCudaErrors(hipFree(m_pDeviceSobolDirVec));
            checkCudaErrors(hipFree(m_pDeviceSobelConsts));
        }
        break;
    case ER_XORWOW:
        default:
        {
            CURAND_CALL(hiprandDestroyGenerator(m_HGen));
            checkCudaErrors(hipFree(m_deviceBuffer));
            checkCudaErrors(hipFree(m_pDeviceRandStatesXORWOW));
        }
        break;
    }
}

//Initial XORWOW only support 512 threads per block
void CRandom::InitialStatesXORWOW(UINT )
{
    checkCudaErrors(hipMalloc((void **)&m_pDeviceRandStatesXORWOW, sizeof(hiprandState) * _HC_Volumn * (_HC_Dir + 1)));
    TArray<UINT> deviceConstraints = CCudaHelper::GetMaxThreadCountAndThreadPerblock();
    deviceConstraints[0] = 512;
    TArray<UINT> latticeDim;
    latticeDim.AddItem(_HC_Lx * _HC_Ly);
    latticeDim.AddItem(_HC_Lz);
    latticeDim.AddItem(_HC_Lt);
    TArray <UINT> decomp = _getDecompose(deviceConstraints, latticeDim);
    dim3 block(decomp[0], decomp[1], decomp[2]);
    dim3 threads(decomp[3], decomp[4], decomp[5]);
    _kernalInitialXORWOW << <block, threads >> > (m_pDeviceRandStatesXORWOW);
}

//Initial Philox only support 256 threads per block
void CRandom::InitialStatesPhilox(UINT )
{
    checkCudaErrors(hipMalloc((void **)&m_pDeviceRandStatesPhilox, sizeof(hiprandStatePhilox4_32_10_t) * _HC_Volumn * (_HC_Dir + 1)));

    TArray<UINT> deviceConstraints = CCudaHelper::GetMaxThreadCountAndThreadPerblock();
    deviceConstraints[0] = 256;
    TArray<UINT> latticeDim;
    latticeDim.AddItem(_HC_Lx * _HC_Ly);
    latticeDim.AddItem(_HC_Lz);
    latticeDim.AddItem(_HC_Lt);
    TArray <UINT> decomp = _getDecompose(deviceConstraints, latticeDim);
    dim3 block(decomp[0], decomp[1], decomp[2]);
    dim3 threads(decomp[3], decomp[4], decomp[5]);

    _kernalInitialPhilox << <block, threads >> > (m_pDeviceRandStatesPhilox);
}

//Initial MRG only support 256 threads per block
void CRandom::InitialStatesMRG(UINT )
{
    checkCudaErrors(hipMalloc((void **)&m_pDeviceRandStatesMRG, sizeof(hiprandStateMRG32k3a) * _HC_Volumn * (_HC_Dir + 1)));
    TArray<UINT> deviceConstraints = CCudaHelper::GetMaxThreadCountAndThreadPerblock();
    deviceConstraints[0] = 256;
    TArray<UINT> latticeDim;
    latticeDim.AddItem(_HC_Lx * _HC_Ly);
    latticeDim.AddItem(_HC_Lz);
    latticeDim.AddItem(_HC_Lt);
    TArray <UINT> decomp = _getDecompose(deviceConstraints, latticeDim);
    dim3 block(decomp[0], decomp[1], decomp[2]);
    dim3 threads(decomp[3], decomp[4], decomp[5]);
    _kernalInitialMRG << <block, threads >> > (m_pDeviceRandStatesMRG);
}

void CRandom::InitialStatesSobol32(UINT )
{
    //support only 20000 dimensions, so using _HC_Volumn instead
    m_uiFatIdDivide = _HC_Dir + 1;
    checkCudaErrors(hipMalloc((void **)&m_pDeviceRandStatesSobol32, 
        sizeof(hiprandStateSobol32) * _HC_Volumn));
    checkCudaErrors(hipMalloc((void **)&m_pDeviceSobolDirVec, 
        sizeof(hiprandDirectionVectors32_t) * _HC_Volumn));

    //int[32]
    hiprandDirectionVectors32_t *hostVectors32;
    CURAND_CALL(hiprandGetDirectionVectors32(&hostVectors32, HIPRAND_DIRECTION_VECTORS_32_JOEKUO6));
    checkCudaErrors(hipMemcpy(m_pDeviceSobolDirVec, hostVectors32, 
        _HC_Volumn * sizeof(hiprandDirectionVectors32_t),
        hipMemcpyHostToDevice));

    preparethread;
    _kernalInitialSobel32 << <block, threads >> > (m_pDeviceRandStatesSobol32, m_pDeviceSobolDirVec);
}

void CRandom::InitialStatesScrambledSobol32(UINT )
{
    m_uiFatIdDivide = _HC_Dir + 1;
    checkCudaErrors(hipMalloc((void **)&m_pDeviceRandStatesScrambledSobol32, 
        sizeof(hiprandStateScrambledSobol32) * _HC_Volumn));
    checkCudaErrors(hipMalloc((void **)&m_pDeviceSobolDirVec, 
        sizeof(hiprandDirectionVectors32_t) * _HC_Volumn));
    checkCudaErrors(hipMalloc((void **)&m_pDeviceSobelConsts, 
        sizeof(UINT) * _HC_Volumn));

    hiprandDirectionVectors32_t *hostVectors32;
    CURAND_CALL(hiprandGetDirectionVectors32(&hostVectors32, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6));
    checkCudaErrors(hipMemcpy(
        m_pDeviceSobolDirVec, 
        hostVectors32, 
        _HC_Volumn * sizeof(hiprandDirectionVectors32_t),
        hipMemcpyHostToDevice));

    UINT * hostScrambleConstants32;
    CURAND_CALL(hiprandGetScrambleConstants32(&hostScrambleConstants32));
    checkCudaErrors(hipMemcpy(
        m_pDeviceSobelConsts, 
        hostScrambleConstants32, 
        _HC_Volumn * sizeof(UINT), 
        hipMemcpyHostToDevice));

    preparethread;
    _kernalInitialScrambledSobel32 << <block, threads >> > (m_pDeviceRandStatesScrambledSobol32, m_pDeviceSobelConsts, m_pDeviceSobolDirVec);
}

void CRandom::InitialTableSchrage(UINT )
{
    checkCudaErrors(hipMalloc((void **)&m_pDeviceSeedTable, sizeof(UINT) * _HC_Volumn * (_HC_Dir + 1)));
    preparethread;
    _kernalAllocateSeedTable << <block, threads >> > (m_pDeviceSeedTable);
}

Real GetRandomReal()
{
    return appGetLattice()->m_pRandom->GetRandomF();
}

#pragma region Test

__global__ void _CLG_LAUNCH_BOUND
_kernelMCPi(UINT* output, UINT lengthyz, UINT lengthz, UINT uiLoop, UINT uithreadCount)
{
    __shared__ UINT sData1[1024];
    __shared__ UINT sData2[1024];
    UINT uiToAdd = 0;
    UINT uiToAdd2 = 0;
    //We have a very large grid, but for a block, it is always smaller (or equval to volumn)
    UINT fatIndex = threadIdx.x * lengthyz + threadIdx.y * lengthz + threadIdx.z;
    for (UINT i = 0; i < uiLoop; ++i)
    {
        Real x = _deviceRandomF(fatIndex) * 2.0f - 1.0f;
        Real y = _deviceRandomF(fatIndex) * 2.0f - 1.0f;
        if (x * x + y * y < 1.0f)
        {
            ++uiToAdd;
        }
        ++uiToAdd2;
    }
    sData1[fatIndex] = uiToAdd;
    sData2[fatIndex] = uiToAdd2;

    __syncthreads();
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
    {
        UINT all1 = 0;
        UINT all2 = 0;
        for (UINT i = 0; i < uithreadCount; ++i)
        {
            all1 += sData1[i];
            all2 += sData2[i];
        }
        //printf("how many?= %d\n", all1);
        atomicAdd(output, all1);
        atomicAdd(output + 1, all2);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelMCE(Real* output, UINT lengthyz, UINT lengthz, UINT uiLoop, UINT uithreadCount)
{
    __shared__ Real sData1[1024];
    __shared__ Real sData2[1024];
    Real fToAdd = 0;
    Real fToAdd2 = 0;
    UINT fatIndex = threadIdx.x * lengthyz + threadIdx.y * lengthz + threadIdx.z;
    for (UINT i = 0; i < uiLoop; ++i)
    {
        _Complex c = _deviceRandomGaussC(fatIndex);
        fToAdd += (c.x + c.y);
        fToAdd2 += (c.x * c.x + c.y * c.y);
    }
    sData1[fatIndex] = fToAdd;
    sData2[fatIndex] = fToAdd2;

    __syncthreads();
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
    {
        Real all1 = 0;
        Real all2 = 0;
        for (UINT i = 0; i < uithreadCount; ++i)
        {
            all1 += sData1[i];
            all2 += sData2[i];
        }
        //printf("how many?= %d\n", all1);
        atomicAdd(output, all1);
        atomicAdd(output + 1, all2);
    }
}

Real CLGAPI CalculatePi(const TArray<UINT> & decompose)
{
    dim3 blocknumber(decompose[0], decompose[1], decompose[2]);
    dim3 threadnumber(decompose[3], decompose[4], decompose[5]);
    UINT threadCount = decompose[3] * decompose[4] * decompose[5];
    UINT lengthyz = decompose[4] * decompose[5];
    UINT lengthz = decompose[5];
    UINT total = decompose[0] * decompose[1] * decompose[2] * decompose[3] * decompose[4] * decompose[5] * decompose[6];
    UINT uiLoop = decompose[6];

    UINT outPutHost[2];
    outPutHost[0] = 0;
    outPutHost[1] = 0;

    UINT *outPut;
    checkCudaErrors(hipMalloc((void**)&outPut, sizeof(UINT) * 2));
    checkCudaErrors(hipMemcpy(outPut, outPutHost, sizeof(UINT) * 2, hipMemcpyHostToDevice));

    _kernelMCPi << <blocknumber, threadnumber >> > (outPut, lengthyz, lengthz, uiLoop, threadCount);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipMemcpy(outPutHost, outPut, sizeof(UINT) * 2, hipMemcpyDeviceToHost));

    appParanoiac(_T("==== results: %d / %d \n"), outPutHost[0], outPutHost[1]);

    return 4.0f * outPutHost[0] / (Real)(total);
}

Real CLGAPI CalculateE(const TArray<UINT> & decompose)
{
    dim3 blocknumber(decompose[0], decompose[1], decompose[2]);
    dim3 threadnumber(decompose[3], decompose[4], decompose[5]);
    UINT threadCount = decompose[3] * decompose[4] * decompose[5];
    UINT lengthyz = decompose[4] * decompose[5];
    UINT lengthz = decompose[5];
    UINT total = decompose[0] * decompose[1] * decompose[2] * decompose[3] * decompose[4] * decompose[5] * decompose[6];
    UINT uiLoop = decompose[6];

    Real outPutHost[2];
    outPutHost[0] = 0.0F;
    outPutHost[1] = 0.0F;

    Real *outPut;
    checkCudaErrors(hipMalloc((void**)&outPut, sizeof(Real) * 2));
    checkCudaErrors(hipMemcpy(outPut, outPutHost, sizeof(Real) * 2, hipMemcpyHostToDevice));

    _kernelMCE << <blocknumber, threadnumber >> > (outPut, lengthyz, lengthz, uiLoop, threadCount);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipMemcpy(outPutHost, outPut, sizeof(Real) * 2, hipMemcpyDeviceToHost));

    Real fAv = outPutHost[0] / (2.0f * total);
    Real fBv = outPutHost[1] / (2.0f * total) - fAv * fAv;

    return _hostsqrt(fBv);
}

#pragma endregion

__END_NAMESPACE

//=============================================================================
// END OF FILE
//=============================================================================
