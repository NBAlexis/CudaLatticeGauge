#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CMeasurePolyakovXY.cu
// 
// DESCRIPTION:
//
//
// REVISION:
//  [05/29/2019 nbale]
//=============================================================================

#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

__CLGIMPLEMENT_CLASS(CMeasurePolyakovXY)

#pragma region kernles 

__global__ void
_CLG_LAUNCH_BOUND 
_kernelPolyakovLoopOfSite(
    const deviceSU3* __restrict__ pDeviceBuffer,
    UINT uiT,
    deviceSU3* res)
{
    UINT uiXYZ = (threadIdx.x + blockIdx.x * blockDim.x) * _DC_Lz + (threadIdx.y + blockIdx.y * blockDim.y);
    const UINT uiSiteIndex = uiXYZ * _DC_Lt + uiT;
    UINT uiLinkIdx = _deviceGetLinkIndex(uiSiteIndex, _DC_Dir - 1);
    //(uiSiteIndex + 1) * _DC_Dir - 1;//uiSiteIndex * _DC_Dir + (_DC_Dir - 1);
    //if (0 == uiXYZ)
    //{
    //    printf("t=%d, site=%d, linkidx=%d\n", uiT, uiSiteIndex, uiLinkIdx);
    //}

    const SSmallInt4 site4 = __deviceSiteIndexToInt4(uiSiteIndex);
    const UINT uiBigIdx = __idx->_deviceGetBigIndex(site4);

    if (0 == uiT)
    {
        if (__idx->_deviceIsBondOnSurface(uiBigIdx, _DC_Dir - 1))
        {
            res[uiXYZ] = deviceSU3::makeSU3Zero();
        }
        else
        {
            res[uiXYZ] = pDeviceBuffer[uiLinkIdx];
        }
    }
    else
    {
        if (__idx->_deviceIsBondOnSurface(uiBigIdx, _DC_Dir - 1))
        {
            res[uiXYZ] = deviceSU3::makeSU3Zero();
        }
        else
        {
            res[uiXYZ].Mul(pDeviceBuffer[uiLinkIdx]);
        }
    }
}

/**
 * Before call me, set block dim thread dim.y = 1
 */
__global__ void
_CLG_LAUNCH_BOUND
_kernelPolyakovLoopOfSiteZ(
    const deviceSU3* __restrict__ pDeviceBuffer,
    deviceSU3* res)
{
    UINT uiXYT = (threadIdx.x + blockIdx.x * blockDim.x) * _DC_Lt + (threadIdx.z + blockIdx.z * blockDim.z);

    for (UINT z = 0; z < _DC_Lz; ++z)
    {
        const UINT uiSiteIndex = (threadIdx.x + blockIdx.x * blockDim.x) * _DC_GridDimZT + z * _DC_Lt + (threadIdx.z + blockIdx.z * blockDim.z);
        UINT uiLinkIdx = _deviceGetLinkIndex(uiSiteIndex, 2);
        const SSmallInt4 site4 = __deviceSiteIndexToInt4(uiSiteIndex);
        const UINT uiBigIdx = __idx->_deviceGetBigIndex(site4);

        if (0 == z)
        {
            if (__idx->_deviceIsBondOnSurface(uiBigIdx, 2))
            {
                res[uiXYT] = deviceSU3::makeSU3Zero();
            }
            else
            {
                res[uiXYT] = pDeviceBuffer[uiLinkIdx];
            }
        }
        else
        {
            if (!__idx->_deviceIsBondOnSurface(uiBigIdx, 2))
            {
                res[uiXYT].Mul(pDeviceBuffer[uiLinkIdx]);
            }
        }
    }
}

__global__ void
_CLG_LAUNCH_BOUND
_kernelPolyakovTraceOfSiteXY(
    const deviceSU3* __restrict__ resXYZ,
    CLGComplex* resXY,
    CLGComplex* resZ,
    Real* resXYAbs,
    Real* resZAbs)
{
    UINT uiXY = threadIdx.x + blockIdx.x * blockDim.x;
    UINT uiZ = threadIdx.y + blockIdx.y * blockDim.y;
    UINT uiXYZ = uiXY * _DC_Lz + uiZ;
    const CLGComplex trres = resXYZ[uiXYZ].Tr();
    atomicAdd(&resXY[uiXY].x, trres.x);
    atomicAdd(&resXY[uiXY].y, trres.y);
    atomicAdd(&resXYAbs[uiXY], _cuCabsf(trres));
    //printf("trres= %f, %f, ||= %f\n", trres.x, trres.y, _cuCabsf(trres));
    if (NULL != resZ)
    {
        atomicAdd(&resZ[uiZ].x, trres.x);
        atomicAdd(&resZ[uiZ].y, trres.y);
    }
    if (NULL != resZAbs)
    {
        atomicAdd(&resZAbs[uiZ], _cuCabsf(trres));
    }
}

__global__ void
_CLG_LAUNCH_BOUND
_kernelInitialZSlice(CLGComplex* resZ, Real* resZAbs)
{
    resZ[threadIdx.x + blockIdx.x * blockDim.x] = _zeroc;
    resZAbs[threadIdx.x + blockIdx.x * blockDim.x] = F(0.0);
}

__global__ void
_CLG_LAUNCH_BOUND
_kernelPolyakovZTraceOfSiteXY(
    const deviceSU3* __restrict__ resXYT,
    CLGComplex* resXY)
{
    UINT uiXY = threadIdx.x + blockIdx.x * blockDim.x;
    UINT uiXYT = uiXY * _DC_Lt + (threadIdx.z + blockIdx.z * blockDim.z);
    const CLGComplex trres = resXYT[uiXYT].Tr();
    atomicAdd(&resXY[uiXY].x, trres.x);
    atomicAdd(&resXY[uiXY].y, trres.y);
}


#pragma endregion

CLGAPI void _PolyakovAtSite(const deviceSU3* __restrict__ pDeviceBuffer, deviceSU3* pRes)
{
    dim3 block1(_HC_DecompX, _HC_DecompY, 1);
    dim3 threads1(_HC_DecompLx, _HC_DecompLy, 1);
    for (UINT uiT = 0; uiT < _HC_Lt; ++uiT)
    {
        _kernelPolyakovLoopOfSite << <block1, threads1 >> >(pDeviceBuffer, uiT, pRes);
    }
}

CMeasurePolyakovXY::~CMeasurePolyakovXY()
{
    if (NULL != m_pXYHostLoopDensity)
    {
        free(m_pXYHostLoopDensity);
    }

    if (NULL != m_pTmpDeviceSum)
    {
        checkCudaErrors(hipFree(m_pTmpDeviceSum));
    }

    if (NULL != m_pXYDeviceLoopDensity)
    {
        checkCudaErrors(hipFree(m_pXYDeviceLoopDensity));
    }

    if (NULL != m_pTmpLoop)
    {
        checkCudaErrors(hipFree(m_pTmpLoop));
    }

    if (NULL != m_pTmpLoopZ)
    {
        checkCudaErrors(hipFree(m_pTmpLoopZ));
    }

    if (NULL != m_pDistributionR)
    {
        checkCudaErrors(hipFree(m_pDistributionR));
    }

    if (NULL != m_pDistributionP)
    {
        checkCudaErrors(hipFree(m_pDistributionP));
    }

    if (NULL != m_pDistributionPAbs)
    {
        checkCudaErrors(hipFree(m_pDistributionPAbs));
    }

    if (NULL != m_pHostDistributionR)
    {
        free(m_pHostDistributionR);
    }

    if (NULL != m_pHostDistributionP)
    {
        free(m_pHostDistributionP);
    }

    if (NULL != m_pHostDistributionPAbs)
    {
        free(m_pHostDistributionPAbs);
    }

    if (NULL != m_pZDeviceLoopDensity)
    {
        checkCudaErrors(hipFree(m_pZDeviceLoopDensity));
    }

    if (NULL != m_pZHostLoopDensity)
    {
        free(m_pZHostLoopDensity);
    }

    if (NULL != m_pXYDeviceLoopDensityAbs)
    {
        checkCudaErrors(hipFree(m_pXYDeviceLoopDensityAbs));
    }

    if (NULL != m_pXYHostLoopDensityAbs)
    {
        free(m_pXYHostLoopDensityAbs);
    }

    if (NULL != m_pZDeviceLoopDensityAbs)
    {
        checkCudaErrors(hipFree(m_pZDeviceLoopDensityAbs));
    }

    if (NULL != m_pZHostLoopDensityAbs)
    {
        free(m_pZHostLoopDensityAbs);
    }
}

void CMeasurePolyakovXY::Initial(CMeasurementManager* pOwner, CLatticeData* pLatticeData, const CParameters& param, BYTE byId)
{
    CMeasure::Initial(pOwner, pLatticeData, param, byId);

    m_pXYHostLoopDensity = (CLGComplex*)malloc(sizeof(CLGComplex) * _HC_Lx * _HC_Ly);
    m_pXYHostLoopDensityAbs = (Real*)malloc(sizeof(Real) * _HC_Lx * _HC_Ly);
    checkCudaErrors(hipMalloc((void**)&m_pTmpDeviceSum, sizeof(CLGComplex)));
    checkCudaErrors(hipMalloc((void**)&m_pXYDeviceLoopDensity, sizeof(CLGComplex) * _HC_Lx * _HC_Ly));
    checkCudaErrors(hipMalloc((void**)&m_pXYDeviceLoopDensityAbs, sizeof(Real) * _HC_Lx * _HC_Ly));
    checkCudaErrors(hipMalloc((void**)&m_pTmpLoop, sizeof(deviceSU3) * _HC_Lx * _HC_Ly * _HC_Lz));
    Reset();

    INT iValue = 1;
    param.FetchValueINT(_T("FieldId"), iValue);
    m_byFieldId = static_cast<BYTE>(iValue);

    iValue = 1;
    param.FetchValueINT(_T("ShowResult"), iValue);
    m_bShowResult = iValue != 0;

    iValue = 0;
    param.FetchValueINT(_T("MeasureZ"), iValue);
    m_bMeasureLoopZ = iValue != 0;

    iValue = 0;
    param.FetchValueINT(_T("ZSlice"), iValue);
    m_bMeasureZSlice = iValue != 0;
    if (m_bMeasureZSlice)
    {
        checkCudaErrors(hipMalloc((void**)&m_pZDeviceLoopDensity, sizeof(CLGComplex) * _HC_Lz));
        checkCudaErrors(hipMalloc((void**)&m_pZDeviceLoopDensityAbs, sizeof(Real) * _HC_Lz));
        m_pZHostLoopDensity = (CLGComplex*)malloc(sizeof(CLGComplex) * _HC_Lz);
        m_pZHostLoopDensityAbs = (Real*)malloc(sizeof(Real) * _HC_Lz);
    }

    m_bMeasureDistribution = TRUE;

    iValue = 0;
    param.FetchValueINT(_T("ShiftCenter"), iValue);
    m_bShiftCenter = iValue != 0;

    //assuming the center is really at center
    SetMaxAndEdge(&m_uiMaxR, &m_uiEdgeR, m_bShiftCenter);
    checkCudaErrors(hipMalloc((void**)&m_pDistributionR, sizeof(UINT) * (m_uiMaxR + 1)));
    checkCudaErrors(hipMalloc((void**)&m_pDistributionP, sizeof(CLGComplex) * (m_uiMaxR + 1)));
    checkCudaErrors(hipMalloc((void**)&m_pDistributionPAbs, sizeof(Real) * (m_uiMaxR + 1)));

    m_pHostDistributionR = (UINT*)malloc(sizeof(UINT) * (m_uiMaxR + 1));
    m_pHostDistributionP = (CLGComplex*)malloc(sizeof(CLGComplex) * (m_uiMaxR + 1));
    m_pHostDistributionPAbs = (Real*)malloc(sizeof(Real) * (m_uiMaxR + 1));

    if (m_bMeasureLoopZ)
    {
        checkCudaErrors(hipMalloc((void**)&m_pTmpLoopZ, sizeof(deviceSU3) * _HC_Lx * _HC_Ly * _HC_Lt));
    }
}

void CMeasurePolyakovXY::OnConfigurationAccepted(const class CFieldGauge* pAcceptGauge, const class CFieldGauge* pCorrespondingStaple)
{
    if (NULL == pAcceptGauge || EFT_GaugeSU3 != pAcceptGauge->GetFieldType())
    {
        appCrucial(_T("CMeasureMesonCorrelator only implemented with gauge SU3!\n"));
        return;
    }
    const CFieldGaugeSU3* pGaugeSU3 = dynamic_cast<const CFieldGaugeSU3*>(pAcceptGauge);

    dim3 block1(_HC_DecompX, _HC_DecompY, 1); 
    dim3 threads1(_HC_DecompLx, _HC_DecompLy, 1);
    for (UINT uiT = 0; uiT < _HC_Lt; ++uiT)
    {
        _kernelPolyakovLoopOfSite << <block1, threads1 >> > (pGaugeSU3->m_pDeviceData, uiT, m_pTmpLoop);
    }
    _ZeroXYPlaneC(m_pXYDeviceLoopDensity);
    _ZeroXYPlane(m_pXYDeviceLoopDensityAbs);
    if (m_bMeasureZSlice)
    {
        dim3 blockz(_HC_DecompY, 1, 1);
        dim3 threadz(_HC_DecompLy, 1, 1);
        _kernelInitialZSlice << <blockz , threadz >> > (m_pZDeviceLoopDensity, m_pZDeviceLoopDensityAbs);
    }
    _kernelPolyakovTraceOfSiteXY << <block1, threads1 >> > (
        m_pTmpLoop, m_pXYDeviceLoopDensity, m_pZDeviceLoopDensity, 
        m_pXYDeviceLoopDensityAbs, m_pZDeviceLoopDensityAbs);
    checkCudaErrors(hipMemcpy(m_pXYHostLoopDensity, m_pXYDeviceLoopDensity, sizeof(CLGComplex) * _HC_Lx * _HC_Ly, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(m_pXYHostLoopDensityAbs, m_pXYDeviceLoopDensityAbs, sizeof(Real) * _HC_Lx * _HC_Ly, hipMemcpyDeviceToHost));
    if (m_bMeasureZSlice)
    {
        const Real fFactor = F(1.0) / static_cast<Real>(_HC_Lx * _HC_Ly);
        checkCudaErrors(hipMemcpy(m_pZHostLoopDensity, m_pZDeviceLoopDensity, sizeof(CLGComplex) * _HC_Lz, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(m_pZHostLoopDensityAbs, m_pZDeviceLoopDensityAbs, sizeof(Real) * _HC_Lz, hipMemcpyDeviceToHost));
        for (UINT i = 0; i < _HC_Lz; ++i)
        {
            m_lstPZSlice.AddItem(cuCmulf_cr(m_pZHostLoopDensity[i], fFactor));
            m_lstPZSliceAbs.AddItem(m_pZHostLoopDensityAbs [i] / fFactor);
        }
    }
    for (UINT i = CCommonData::m_sCenter.x; i < _HC_Lx; ++i)
    {
        m_lstLoopDensity.AddItem(m_pXYHostLoopDensity[i * _HC_Ly + CCommonData::m_sCenter.y]);
    }

    TransformFromXYDataToRDataOnce_C(
        m_bShiftCenter,
        m_pXYDeviceLoopDensity,
        m_pDistributionR,
        m_pDistributionP,
        m_pHostDistributionR,
        m_pHostDistributionP,
        m_uiMaxR,
        m_uiEdgeR,
        TRUE,
        m_byFieldId,
        m_lstP,
        &m_lstLoopInner,
        m_lstLoop,
        m_lstR,
        m_uiConfigurationCount,
        F(1.0) / static_cast<Real>(_HC_Lz)
    );

    TransformFromXYDataToRDataOnce_R(
        m_bShiftCenter,
        m_pXYDeviceLoopDensityAbs,
        m_pDistributionR,
        m_pDistributionPAbs,
        m_pHostDistributionR,
        m_pHostDistributionPAbs,
        m_uiMaxR,
        m_uiEdgeR,
        FALSE,
        m_byFieldId,
        m_lstPAbs,
        &m_lstLoopAbsInner,
        m_lstLoopAbs,
        m_lstR,
        m_uiConfigurationCount,
        F(1.0) / static_cast<Real>(_HC_Lz)
    );

    if (m_bShowResult)
    {
        appDetailed(_T("\n\n ==================== Polyakov Loop (%d con)============================ \n\n"), m_uiConfigurationCount);
    }

    if (m_bShowResult)
    {
        appSetLogDate(FALSE);
        appGeneral(_T("Loop is "));
        LogGeneralComplex(m_lstLoop[m_lstLoop.GetCount() - 1]);
        appGeneral(_T(" Abs is %f\n"), m_lstLoopAbs[m_lstLoopAbs.GetCount() - 1]);
        appSetLogDate(TRUE);
    }

    if (m_bShowResult)
    {
        for (UINT i = 1; i < _HC_Lx; ++i)
        {
            appDetailed(_T("{"));
            for (UINT j = 1; j < _HC_Ly; ++j)
            {
                appDetailed(_T("%1.12f %s %1.12f I%s"),
                    m_pXYHostLoopDensity[i * _HC_Ly + j].x,
                    m_pXYHostLoopDensity[i * _HC_Ly + j].y < F(0.0) ? _T("-") : _T("+"),
                    appAbs(m_pXYHostLoopDensity[i * _HC_Ly + j].y),
                    (j == _HC_Ly - 1) ? _T("},\n") : _T(",   ")
                );
            }
        }
    }

    if (m_bShowResult)
    {
        appGeneral(_T("\n"));
    }

    if (m_bShowResult)
    {
        appDetailed(_T("\n=====================================================\n"), m_uiConfigurationCount);
    }

    if (m_bMeasureLoopZ)
    {
        dim3 block3(_HC_DecompX, 1, _HC_DecompZ);
        dim3 threads3(_HC_DecompLx, 1, _HC_DecompLz);
        _kernelPolyakovLoopOfSiteZ << <block3, threads3 >> > (pGaugeSU3->m_pDeviceData, m_pTmpLoopZ);
        _ZeroXYPlaneC(m_pXYDeviceLoopDensity);
        _kernelPolyakovZTraceOfSiteXY << <block3, threads3 >> > (m_pTmpLoopZ, m_pXYDeviceLoopDensity);

        checkCudaErrors(hipMemcpy(m_pXYHostLoopDensity, m_pXYDeviceLoopDensity, sizeof(CLGComplex) * _HC_Lx * _HC_Ly, hipMemcpyDeviceToHost));
        for (UINT i = CCommonData::m_sCenter.x; i < _HC_Lx; ++i)
        {
            m_lstLoopZDensity.AddItem(m_pXYHostLoopDensity[
                i * _HC_Ly + CCommonData::m_sCenter.y]);
        }

        TransformFromXYDataToRDataOnce_C(
            m_bShiftCenter,
            m_pXYDeviceLoopDensity,
            m_pDistributionR,
            m_pDistributionP,
            m_pHostDistributionR,
            m_pHostDistributionP,
            m_uiMaxR,
            m_uiEdgeR,
            FALSE,
            m_byFieldId,
            m_lstPZ,
            &m_lstLoopZInner,
            m_lstLoopZ,
            m_lstR,
            m_uiConfigurationCount,
            F(1.0) / static_cast<Real>(_HC_Lt)
        );

        if (m_bShowResult)
        {
            appDetailed(_T("\n\n ==================== Polyakov LoopZ (%d con)============================ \n\n"), m_uiConfigurationCount);
        }

        if (m_bShowResult)
        {
            appSetLogDate(FALSE);
            appGeneral(_T("Loop Z is "));
            LogGeneralComplex(m_lstLoopZ[m_lstLoopZ.GetCount() - 1]);
            appGeneral(_T("\n"));
            appSetLogDate(TRUE);
            //appGeneral(_T("Loop is %f + %f I\n"), res[0].x, res[0].y);
        }
    }

    ++m_uiConfigurationCount;
}

void CMeasurePolyakovXY::Average(UINT )
{
    //nothing to do
}

void CMeasurePolyakovXY::Report()
{
    UINT uiHalf = (_HC_Lx + 1) / 2;
    assert(m_uiConfigurationCount == static_cast<UINT>(m_lstLoop.Num()));
    assert(static_cast<UINT>(m_uiConfigurationCount * uiHalf)
        == static_cast<UINT>(m_lstLoopDensity.Num()));

    appSetLogDate(FALSE);
    CLGComplex tmpChargeSum = _make_cuComplex(F(0.0), F(0.0));
    m_lstAverageLoopDensity.RemoveAll();

    appGeneral(_T("\n\n==========================================================================\n"));
    appGeneral(_T("==================== Polyakov Loop (%d con)============================\n"), m_uiConfigurationCount);

    appGeneral(_T("\n ----------- Loop ------------- \n"));

    appGeneral(_T("{"));
    for (UINT i = 0; i < m_uiConfigurationCount; ++i)
    {
        tmpChargeSum.x += m_lstLoop[i].x;
        tmpChargeSum.y += m_lstLoop[i].y;
        LogGeneralComplex(m_lstLoop[i]);
    }
    appGeneral(_T("}\n"));

    tmpChargeSum.x = tmpChargeSum.x / m_uiConfigurationCount;
    tmpChargeSum.y = tmpChargeSum.y / m_uiConfigurationCount;
    m_cAverageLoop = tmpChargeSum;
    appGeneral(_T("\n ----------- average Loop |<P>| = %2.12f arg(P) = %2.12f ------------- \n"), _cuCabsf(tmpChargeSum), __cuCargf(tmpChargeSum));

    appGeneral(_T("\n ----------- Loop density ------------- \n"));

    appGeneral(_T("{\n"));
    for (UINT k = 0; k < m_uiConfigurationCount; ++k)
    {
        appGeneral(_T("{"));
        for (UINT i = 0; i < uiHalf; ++i)
        {
            LogGeneralComplex(m_lstLoopDensity[k * uiHalf + i]);

            if (0 == k)
            {
                m_lstAverageLoopDensity.AddItem(m_lstLoopDensity[k * uiHalf + i]);
            }
            else
            {
                m_lstAverageLoopDensity[i] = _cuCaddf(m_lstAverageLoopDensity[i], m_lstLoopDensity[k * uiHalf + i]);
            }

            if (k == m_uiConfigurationCount - 1)
            {
                m_lstAverageLoopDensity[i].x = m_lstAverageLoopDensity[i].x / m_uiConfigurationCount;
                m_lstAverageLoopDensity[i].y = m_lstAverageLoopDensity[i].y / m_uiConfigurationCount;
            }
        }
        appGeneral(_T("}\n"));
    }
    appGeneral(_T("}\n"));

    appGeneral(_T("\n==========================================================================\n"));
    appGeneral(_T("==========================================================================\n\n"));
    appSetLogDate(TRUE);
}

void CMeasurePolyakovXY::Reset()
{
    m_uiConfigurationCount = 0;
    m_lstLoop.RemoveAll();
    m_lstLoopInner.RemoveAll();
    m_lstLoopAbs.RemoveAll();
    m_lstLoopAbsInner.RemoveAll();
    m_lstLoopDensity.RemoveAll();
    m_lstLoopZ.RemoveAll();
    m_lstLoopZInner.RemoveAll();
    m_lstLoopZDensity.RemoveAll();

    m_lstR.RemoveAll();
    m_lstP.RemoveAll();
    m_lstPAbs.RemoveAll();
    m_lstPZ.RemoveAll();
    m_lstPZSlice.RemoveAll();
    m_lstPZSliceAbs.RemoveAll();
}

__END_NAMESPACE

//=============================================================================
// END OF FILE
//=============================================================================