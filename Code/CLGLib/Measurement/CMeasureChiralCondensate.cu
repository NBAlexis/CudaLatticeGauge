#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CMeasureChiralCondensate.cpp
// 
// DESCRIPTION:
//
//
// REVISION:
//  [06/13/2019 nbale]
//=============================================================================

#include "CLGLib_Private.h"
#include "Data/Field/WilsonDirac/CFieldFermionWilsonSquareSU3.h"
#include "CMeasureChiralCondensate.h"

__BEGIN_NAMESPACE

__CLGIMPLEMENT_CLASS(CMeasureChiralCondensate)

//Function ptr is a good idea, but exceeds the regcount...
//__constant__ _deviceMeasureCondensateFunc _cMeasureCondFuncs[CMeasureChiralCondensate::_kCondMeasureCount] =
//{
//    _deviceMeasureChiral,
//    _deviceMeasureGamma1,
//    _deviceMeasureGamma2,
//    _deviceMeasureGamma3,
//    _deviceMeasureGamma4,
//    _deviceMeasureGamma5,
//    _deviceMeasureGamma45,
//    _deviceMeasureGammaX,
//    _deviceMeasureGammaY
//};

#pragma region kernels

/**
 * -4*kappa is multiplied outside
 */
__global__ void
_CLG_LAUNCH_BOUND
_kernelDotMeasureAll(
    BYTE byMeasureIndex,
#if !_CLG_DOUBLEFLOAT
    DOUBLE fOmega,
#else
    Real fOmega,
#endif
    const deviceWilsonVectorSU3* __restrict__ pMe,
    const deviceWilsonVectorSU3* __restrict__ pOther, 
    CLGComplex* resultXYPlan,
#if !_CLG_DOUBLEFLOAT
    hipDoubleComplex* result
#else
    CLGComplex* result
#endif
)
{
    intokernalInt4;

    deviceWilsonVectorSU3 right(pOther[uiSiteIndex]);
    switch (byMeasureIndex)
    {
    case 1:
    case 2:
    case 3:
    case 4:
    case 5:
        {
            right = __chiralGamma[byMeasureIndex].MulWilsonC(right);
        }
        break;
    case 6:
        {
            right = __chiralGamma[GAMMA45].MulWilsonC(right);
        }
        break;
    case 7:
        {
            const Real fYOmega = static_cast<Real>(sSite4.y - _DC_Centery)* fOmega;
            deviceWilsonVectorSU3 toAdd(__chiralGamma[GAMMA4].MulWilsonC(right));
            toAdd.MulReal(fYOmega);
            right = __chiralGamma[GAMMA1].MulWilsonC(right);
            right.Add(toAdd);
        }
        break;
    case 8:
        {
            const Real fXOmega = static_cast<Real>(sSite4.x - _DC_Centerx)* fOmega;
            deviceWilsonVectorSU3 toAdd = __chiralGamma[GAMMA4].MulWilsonC(right);
            toAdd.MulReal(fXOmega);
            right = __chiralGamma[GAMMA2].MulWilsonC(right);
            right.Sub(toAdd);
        }
        break;
    case 0:
    default:
        break;
    }
    
#if !_CLG_DOUBLEFLOAT
    result[uiSiteIndex] = _cToDouble(pMe[uiSiteIndex].ConjugateDotC(right));
    atomicAdd(&resultXYPlan[_ixy].x, static_cast<Real>(result[uiSiteIndex].x));
    atomicAdd(&resultXYPlan[_ixy].y, static_cast<Real>(result[uiSiteIndex].y));
#else
    result[uiSiteIndex] = pMe[uiSiteIndex].ConjugateDotC(right);
    atomicAdd(&resultXYPlan[_ixy].x, result[uiSiteIndex].x);
    atomicAdd(&resultXYPlan[_ixy].y, result[uiSiteIndex].y);
#endif
}

__global__ void
_CLG_LAUNCH_BOUND
_kernelChiralCondensateInitialDistR(UINT* pCount)
{
    pCount[threadIdx.x] = 0;
}

__global__ void
_CLG_LAUNCH_BOUND
_kernelChiralCondensateInitialDistCond(CLGComplex* pCond)
{
    pCond[threadIdx.x] = _zeroc;
}

__global__ void
_CLG_LAUNCH_BOUND
_kernelChiralCondensateMeasureDist(
    const CLGComplex* __restrict__ CondXY,
    UINT uiMax, BYTE byFieldId, UBOOL bCalcR,
    UINT* counter, 
    CLGComplex* CondR
)
{
    UINT uiXY = (threadIdx.x + blockIdx.x * blockDim.x);
    INT uiX = static_cast<INT>(uiXY / _DC_Ly);
    INT uiY = static_cast<INT>(uiXY % _DC_Ly);
    UINT uiC = (_DC_Centerx - uiX) * (_DC_Centerx - uiX)
        + (_DC_Centery - uiY) * (_DC_Centery - uiY);

    SSmallInt4 sSite4;
    sSite4.z = _DC_Centerz;
    sSite4.w = _DC_Centert;
    sSite4.x = static_cast<SBYTE>(uiX);
    sSite4.y = static_cast<SBYTE>(uiY);
    if (uiC <= uiMax && !__idx->_deviceGetMappingIndex(sSite4, byFieldId).IsDirichlet())
    {
        if (bCalcR)
        {
            atomicAdd(&counter[uiC], 1);
        }
        
        atomicAdd(&CondR[uiC].x, CondXY[uiXY].x);
        atomicAdd(&CondR[uiC].y, CondXY[uiXY].y);
    }
}

__global__ void
_CLG_LAUNCH_BOUND
_kernelChiralAverageDist(UINT* pCount, CLGComplex* pCond)
{
    const UINT uiIdx = threadIdx.x;
    if (pCount[uiIdx] > 0)
    {
        pCond[uiIdx].x = pCond[uiIdx].x / static_cast<Real>(pCount[uiIdx]);
        pCond[uiIdx].y = pCond[uiIdx].y / static_cast<Real>(pCount[uiIdx]);
    }
}

#pragma endregion

CMeasureChiralCondensate::~CMeasureChiralCondensate()
{
    if (NULL != m_pDeviceXYBuffer[0])
    {
        for (UINT i = 0; i < _kCondMeasureCount; ++i)
        {
            checkCudaErrors(hipFree(m_pDeviceXYBuffer[i]));
        }
    }

    if (NULL != m_pHostXYBuffer)
    {
        free(m_pHostXYBuffer);
    }

    if (NULL != m_pDistributionR)
    {
        checkCudaErrors(hipFree(m_pDistributionR));
    }

    if (NULL != m_pDistribution)
    {
        checkCudaErrors(hipFree(m_pDistribution));
    }

    if (NULL != m_pHostDistributionR)
    {
        free(m_pHostDistributionR);
    }

    if (NULL != m_pHostDistribution)
    {
        free(m_pHostDistribution);
    }
}

void CMeasureChiralCondensate::Initial(CMeasurementManager* pOwner, CLatticeData* pLatticeData, const CParameters& param, BYTE byId)
{
    CMeasureStochastic::Initial(pOwner, pLatticeData, param, byId);

    for (UINT i = 0; i < _kCondMeasureCount; ++i)
    {
        checkCudaErrors(hipMalloc((void**)&m_pDeviceXYBuffer[i], sizeof(CLGComplex) * _HC_Lx * _HC_Ly));
    }    
    m_pHostXYBuffer = (CLGComplex*)malloc(sizeof(CLGComplex) * _HC_Lx * _HC_Ly);

    Reset();

    INT iValue = 1;
    param.FetchValueINT(_T("MeasureDist"), iValue);
    m_bMeasureDistribution = iValue != 0;

    if (m_bMeasureDistribution)
    {
        //assuming the center is really at center
        m_uiMaxR = ((_HC_Lx + 1) / 2 ) * ((_HC_Lx + 1) / 2 )
            + ((_HC_Ly + 1) / 2 ) * ((_HC_Ly + 1) / 2 );

        checkCudaErrors(hipMalloc((void**)&m_pDistributionR, sizeof(UINT) * (m_uiMaxR + 1)));
        checkCudaErrors(hipMalloc((void**)&m_pDistribution, sizeof(CLGComplex) * (m_uiMaxR + 1)));

        m_pHostDistributionR = (UINT*)malloc(sizeof(UINT) * (m_uiMaxR + 1));
        m_pHostDistribution = (CLGComplex*)malloc(sizeof(CLGComplex) * (m_uiMaxR + 1));
    }
}

void CMeasureChiralCondensate::OnConfigurationAcceptedZ4SingleField(
    const class CFieldGauge* pAcceptGauge, 
    const class CFieldGauge* pCorrespondingStaple, 
    const class CFieldFermion* pZ4, 
    const class CFieldFermion* pInverseZ4, 
    UBOOL bStart, 
    UBOOL bEnd)
{
    if (bStart)
    {
        for (UINT i = 0; i < _kCondMeasureCount; ++i)
        {
            _ZeroXYPlaneC(m_pDeviceXYBuffer[i]);
            m_cTmpSum[i] = _zeroc;
        }
    }

    const Real oneOuiVolume = F(1.0) / appGetLattice()->m_pIndexCache->m_uiSiteNumber[GetFermionFieldId()];
    const CFieldFermionWilsonSquareSU3 * pF1W = dynamic_cast<const CFieldFermionWilsonSquareSU3*>(pZ4);
    const CFieldFermionWilsonSquareSU3 * pF2W = dynamic_cast<const CFieldFermionWilsonSquareSU3*>(pInverseZ4);
    
#pragma region Dot

    // The results are Atomic Add to m_pDeviceXYBuffer
    preparethread;
    for (BYTE i = 0; i < _kCondMeasureCount; ++i)
    {
        _kernelDotMeasureAll << <block, threads >> > (
            i,
            CCommonData::m_fOmega,
            pF1W->m_pDeviceData,
            pF2W->m_pDeviceData,
            m_pDeviceXYBuffer[i],
            _D_ComplexThreadBuffer
            );
#if !_CLG_DOUBLEFLOAT
        const CLGComplex thisSum = _cToFloat(appGetCudaHelper()->ThreadBufferSum(_D_ComplexThreadBuffer));
#else
        const CLGComplex thisSum = appGetCudaHelper()->ThreadBufferSum(_D_ComplexThreadBuffer); 
#endif
        m_cTmpSum[i] = _cuCaddf(m_cTmpSum[i], cuCmulf_cr(thisSum, oneOuiVolume));
    }

#pragma endregion

    if (bEnd)
    {
        if (m_bMeasureDistribution)
        {
            dim3 block2(_HC_DecompX, 1, 1);
            dim3 threads2(_HC_DecompLx, 1, 1);
            dim3 block3(1, 1, 1);
            dim3 threads3(m_uiMaxR + 1, 1, 1);

            const Real fDivider = F(1.0) / (m_uiFieldCount * _HC_Lz * _HC_Lt);
            _kernelChiralCondensateInitialDistR << <block3, threads3 >> > (m_pDistributionR);
            for (UINT i = 0; i < _kCondMeasureCount; ++i)
            {
                _kernelChiralCondensateInitialDistCond << <block3, threads3 >> > (m_pDistribution);

                _kernelChiralCondensateMeasureDist << <block2, threads2 >> > (
                    m_pDeviceXYBuffer[i],
                    m_uiMaxR,
                    pF1W->m_byFieldId,
                    0 == i,
                    m_pDistributionR,
                    m_pDistribution
                    );

                _kernelChiralAverageDist << <block3, threads3 >> > (m_pDistributionR, m_pDistribution);

                if (0 == i)
                {
                    checkCudaErrors(hipMemcpy(m_pHostDistributionR, m_pDistributionR, sizeof(UINT) * (m_uiMaxR + 1), hipMemcpyDeviceToHost));
                }

                checkCudaErrors(hipMemcpy(m_pHostDistribution, m_pDistribution, sizeof(CLGComplex) * (m_uiMaxR + 1), hipMemcpyDeviceToHost));

                if (0 == m_uiConfigurationCount)
                {
                    if (0 == i)
                    {
                        assert(0 == m_lstR.Num());
                    }
                    assert(0 == m_lstCond[i].Num());
                    for (UINT uiL = 0; uiL <= m_uiMaxR; ++uiL)
                    {
                        if (m_pHostDistributionR[uiL] > 0)
                        {
                            if (0 == i)
                            {
                                m_lstR.AddItem(uiL);
                            }
                            m_lstCond[i].AddItem(cuCmulf_cr(m_pHostDistribution[uiL], fDivider));

                            if (m_bShowResult)
                            {
                                appDetailed(_T("Cond %d (r = %f)= %f + %f i\n"),
                                    i,
                                    _hostsqrt(static_cast<Real>(uiL)),
                                    m_pHostDistribution[uiL].x,
                                    m_pHostDistribution[uiL].y
                                );
                            }
                        }
                    }
                }
                else
                {
                    for (INT j = 0; j < m_lstR.Num(); ++j)
                    {
                        assert(m_pHostDistributionR[m_lstR[j]] > 0);
                        m_lstCond[i].AddItem(cuCmulf_cr(m_pHostDistribution[m_lstR[j]], fDivider));

                        if (m_bShowResult)
                        {
                            appDetailed(_T("Cond %d (r = %f)=%f + %f i\n"),
                                i,
                                _hostsqrt(static_cast<Real>(m_lstR[j])),
                                m_pHostDistribution[m_lstR[j]].x,
                                m_pHostDistribution[m_lstR[j]].y
                            );
                        }
                    }
                }
            }
        }

        const Real fDiv2 = F(1.0) / m_uiFieldCount;
        for (UINT i = 0; i < _kCondMeasureCount; ++i)
        {
            m_cTmpSum[i] = cuCmulf_cr(m_cTmpSum[i], fDiv2);
            appDetailed(_T("\n Condensate %d = %2.12f + %2.12f\n"), i, m_cTmpSum[i].x, m_cTmpSum[i].y);
            m_lstCondAll[i].AddItem(m_cTmpSum[i]);
            if (0 == i)
            {
                UpdateRealResult(m_cTmpSum[i].x, FALSE);
                UpdateComplexResult(m_cTmpSum[i], FALSE);
            }
        }

        ++m_uiConfigurationCount;
    }
}

void CMeasureChiralCondensate::Report()
{
    appPushLogDate(FALSE);
    for (UINT i = 0; i < _kCondMeasureCount; ++i)
    {
        assert(m_uiConfigurationCount == static_cast<UINT>(m_lstCondAll[i].Num()));

        appGeneral(_T("\n==========================================================================\n"));
        appGeneral(_T("==================== Condensate No %d (%d con)============================\n"), i, m_uiConfigurationCount);
        CLGComplex tmpChargeSum = _zeroc;
        if (m_uiConfigurationCount > 1)
        {
            appGeneral(_T("\n ----------- each configuration ------------- \n"));
            appGeneral(_T("{"));

            for (UINT j = 0; j < m_uiConfigurationCount; ++j)
            {
                tmpChargeSum.x += m_lstCondAll[i][j].x;
                tmpChargeSum.y += m_lstCondAll[i][j].y;
                LogGeneralComplex(m_lstCondAll[i][j]);
            }
            appGeneral(_T("}\n"));

            tmpChargeSum.x = tmpChargeSum.x / m_uiConfigurationCount;
            tmpChargeSum.y = tmpChargeSum.y / m_uiConfigurationCount;
            appGeneral(_T("\n ----------- average condensate = %2.12f + %2.12f ------------- \n"),
                tmpChargeSum.x, tmpChargeSum.y);

            //m_cAverageCondensate = tmpChargeSum;
        }
        else
        {
            appGeneral(_T("\n ----------- average condensate = %2.12f + %2.12f ------------- \n"),
                m_lstCondAll[i][0].x,
                m_lstCondAll[i][0].y);

            //m_cAverageCondensate = m_lstCondAll[i][0];
        }
    }

    appGeneral(_T("==========================================================================\n"));
    appPopLogDate();
}

void CMeasureChiralCondensate::Reset()
{
    CMeasureStochastic::Reset();

    for (UINT i = 0; i < _kCondMeasureCount; ++i)
    {
        m_lstCondAll[i].RemoveAll();
        m_lstCond[i].RemoveAll();
    }
    m_lstR.RemoveAll();
}

__END_NAMESPACE

//=============================================================================
// END OF FILE
//=============================================================================