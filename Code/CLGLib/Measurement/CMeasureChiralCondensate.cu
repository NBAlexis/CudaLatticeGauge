#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CMeasureChiralCondensate.cpp
// 
// DESCRIPTION:
//
//
// REVISION:
//  [06/13/2019 nbale]
//=============================================================================

#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

__CLGIMPLEMENT_CLASS(CMeasureChiralCondensate)

#pragma region kernels

/**
 * Psi^bar Psi
 */
__global__ void _CLG_LAUNCH_BOUND
_kernelDotAndGatherXYChiral(
    const deviceWilsonVectorSU3 * __restrict__ pMe,
    const deviceWilsonVectorSU3 * __restrict__ pOther,
    CLGComplex* resultXYPlaneChiral,
    CLGComplex * result)
{
    intokernal;

    UINT uiXY = threadIdx.x + blockIdx.x * blockDim.x;
    result[uiSiteIndex] = pMe[uiSiteIndex].ConjugateDotC(pOther[uiSiteIndex]);
    atomicAdd(&resultXYPlaneChiral[uiXY].x, result[uiSiteIndex].x);
    atomicAdd(&resultXYPlaneChiral[uiXY].y, result[uiSiteIndex].y);
}

/**
 * Psi^bar Psi
 */
__global__ void _CLG_LAUNCH_BOUND
_kernelDotAndGatherXYPion(
    const deviceWilsonVectorSU3* __restrict__ pMe,
    const deviceWilsonVectorSU3* __restrict__ pOther,
    CLGComplex* resultXYPlanePion,
    CLGComplex* result)
{
    intokernal;

    UINT uiXY = threadIdx.x + blockIdx.x * blockDim.x;
    result[uiSiteIndex] = pMe[uiSiteIndex].ConjugateDotC(pOther[uiSiteIndex]);
    atomicAdd(&resultXYPlanePion[uiXY].x, result[uiSiteIndex].x);
    atomicAdd(&resultXYPlanePion[uiXY].y, result[uiSiteIndex].y);
}

/**
 * Psi^bar Psi
 */
__global__ void _CLG_LAUNCH_BOUND
_kernelDotAndGatherXYRhon(
    const deviceWilsonVectorSU3* __restrict__ pMe,
    const deviceWilsonVectorSU3* __restrict__ pOther,
    CLGComplex* resultXYPlaneRhon,
    CLGComplex* result)
{
    intokernal;

    UINT uiXY = threadIdx.x + blockIdx.x * blockDim.x;
    result[uiSiteIndex] = pMe[uiSiteIndex].ConjugateDotC(pOther[uiSiteIndex]);
    atomicAdd(&resultXYPlaneRhon[uiXY].x, result[uiSiteIndex].x);
    atomicAdd(&resultXYPlaneRhon[uiXY].y, result[uiSiteIndex].y);
}

__global__ void
_CLG_LAUNCH_BOUND
_kernelChiralCondensateInitialDist(UINT* pCount, Real* pChiral, Real* pPion, Real* pRhon)
{
    pCount[threadIdx.x] = 0;
    pChiral[threadIdx.x] = F(0.0);
    pPion[threadIdx.x] = F(0.0);
    pRhon[threadIdx.x] = F(0.0);
}

__global__ void
_CLG_LAUNCH_BOUND
_kernelChiralCondensateMeasureDist(
    const CLGComplex* __restrict__ chiralXY,
    const CLGComplex* __restrict__ pionXY,
    const CLGComplex* __restrict__ rhonXY,
    SSmallInt4 sCenter, UINT uiMax, BYTE byFieldId,
    UINT* counter, 
    Real* chiral,
    Real* pion,
    Real* rhon
)
{
    UINT uiXY = (threadIdx.x + blockIdx.x * blockDim.x);
    SBYTE uiX = static_cast<SBYTE>(uiXY / _DC_Ly);
    SBYTE uiY = static_cast<SBYTE>(uiXY % _DC_Ly);
    UINT uiC = (sCenter.x - uiX) * (sCenter.x - uiX)
        + (sCenter.y - uiY) * (sCenter.y - uiY);

    SSmallInt4 sSite4;
    sSite4.z = sCenter.z;
    sSite4.w = sCenter.w;
    sSite4.x = uiX;
    sSite4.y = uiY;
    if (uiC <= uiMax && !__idx->_deviceGetMappingIndex(sSite4, byFieldId).IsDirichlet())
    {
        atomicAdd(&counter[uiC], 1);
        atomicAdd(&chiral[uiC], chiralXY[uiXY].x);
        atomicAdd(&pion[uiC], pionXY[uiXY].x);
        atomicAdd(&rhon[uiC], rhonXY[uiXY].x);
    }
}

__global__ void
_CLG_LAUNCH_BOUND
_kernelChiralAverageDist(UINT* pCount, Real* pChiral, Real* pPion, Real* pRhon)
{
    const UINT uiIdx = threadIdx.x;
    if (pCount[uiIdx] > 0)
    {
        pChiral[uiIdx] = pChiral[uiIdx] / static_cast<Real>(pCount[uiIdx]);
        pPion[uiIdx] = pPion[uiIdx] / static_cast<Real>(pCount[uiIdx]);
        pRhon[uiIdx] = pRhon[uiIdx] / static_cast<Real>(pCount[uiIdx]);
    }
}

#pragma endregion

CMeasureChiralCondensate::~CMeasureChiralCondensate()
{
    if (NULL != m_pDeviceXYBufferChiral)
    {
        checkCudaErrors(hipFree(m_pDeviceXYBufferChiral));
        checkCudaErrors(hipFree(m_pDeviceXYBufferPion));
        checkCudaErrors(hipFree(m_pDeviceXYBufferRhon));
        free(m_pHostXYBuffer);
    }

    if (NULL != m_pDistributionR)
    {
        checkCudaErrors(hipFree(m_pDistributionR));
    }

    if (NULL != m_pDistributionChiral)
    {
        checkCudaErrors(hipFree(m_pDistributionChiral));
        checkCudaErrors(hipFree(m_pDistributionPion));
        checkCudaErrors(hipFree(m_pDistributionRhon));
    }

    if (NULL != m_pHostDistributionR)
    {
        free(m_pHostDistributionR);
    }

    if (NULL != m_pHostDistributionChiral)
    {
        free(m_pHostDistributionChiral);
        free(m_pHostDistributionPion);
        free(m_pHostDistributionRhon);
    }
}

void CMeasureChiralCondensate::Initial(CMeasurementManager* pOwner, CLatticeData* pLatticeData, const CParameters& param, BYTE byId)
{
    CMeasureStochastic::Initial(pOwner, pLatticeData, param, byId);

    checkCudaErrors(hipMalloc((void**)&m_pDeviceXYBufferChiral, sizeof(CLGComplex) * _HC_Lx * _HC_Ly));
    checkCudaErrors(hipMalloc((void**)&m_pDeviceXYBufferPion, sizeof(CLGComplex) * _HC_Lx * _HC_Ly));
    checkCudaErrors(hipMalloc((void**)&m_pDeviceXYBufferRhon, sizeof(CLGComplex) * _HC_Lx * _HC_Ly));
    m_pHostXYBuffer = (CLGComplex*)malloc(sizeof(CLGComplex) * _HC_Lx * _HC_Ly);

    Reset();

    INT iValue = 1;
    param.FetchValueINT(_T("ShowResult"), iValue);
    m_bShowResult = iValue != 0;

    iValue = 1;
    param.FetchValueINT(_T("MeasureDist"), iValue);
    m_bMeasureDistribution = iValue != 0;

    if (m_bMeasureDistribution)
    {
        //assuming the center is really at center
        m_uiMaxR = ((_HC_Lx + 1) / 2 ) * ((_HC_Lx + 1) / 2 )
            + ((_HC_Ly + 1) / 2 ) * ((_HC_Ly + 1) / 2 );

        checkCudaErrors(hipMalloc((void**)&m_pDistributionR, sizeof(UINT) * (m_uiMaxR + 1)));
        checkCudaErrors(hipMalloc((void**)&m_pDistributionChiral, sizeof(Real) * (m_uiMaxR + 1)));
        checkCudaErrors(hipMalloc((void**)&m_pDistributionPion, sizeof(Real) * (m_uiMaxR + 1)));
        checkCudaErrors(hipMalloc((void**)&m_pDistributionRhon, sizeof(Real) * (m_uiMaxR + 1)));

        m_pHostDistributionR = (UINT*)malloc(sizeof(UINT) * (m_uiMaxR + 1));
        m_pHostDistributionChiral = (Real*)malloc(sizeof(Real) * (m_uiMaxR + 1));
        m_pHostDistributionPion = (Real*)malloc(sizeof(Real) * (m_uiMaxR + 1));
        m_pHostDistributionRhon = (Real*)malloc(sizeof(Real) * (m_uiMaxR + 1));
    }
}

void CMeasureChiralCondensate::OnConfigurationAcceptedZ4(
    const class CFieldGauge* pAcceptGauge, 
    const class CFieldGauge* pCorrespondingStaple, 
    const class CFieldFermion* pZ4, 
    const class CFieldFermion* pInverseZ4, 
    UBOOL bStart, 
    UBOOL bEnd)
{
    if (bStart)
    {
        _ZeroXYPlaneC(m_pDeviceXYBufferChiral);
        _ZeroXYPlaneC(m_pDeviceXYBufferPion);
        _ZeroXYPlaneC(m_pDeviceXYBufferRhon);
        m_cTmpSumChiral = _make_cuComplex(F(0.0), F(0.0));
        m_cTmpSumPion = _make_cuComplex(F(0.0), F(0.0));
        m_cTmpSumRhon = _make_cuComplex(F(0.0), F(0.0));
    }

    const UINT uiVolume = appGetLattice()->m_pIndexCache->m_uiSiteNumber[m_byFieldId];
    const CFieldFermionWilsonSquareSU3 * pF1W = dynamic_cast<const CFieldFermionWilsonSquareSU3*>(pInverseZ4);
    const CFieldFermionWilsonSquareSU3 * pF2W = dynamic_cast<const CFieldFermionWilsonSquareSU3*>(pZ4);   

    
#pragma region Dot

    // The results are Atomic Add to m_pDeviceXYBuffer

    preparethread;
    _kernelDotAndGatherXYChiral << <block, threads >> > (
        pF1W->m_pDeviceData,
        pF2W->m_pDeviceData,
        m_pDeviceXYBufferChiral,
        _D_ComplexThreadBuffer);

    const CLGComplex thisSumChiral = appGetCudaHelper()->ThreadBufferSum(_D_ComplexThreadBuffer);

    m_cTmpSumChiral.x = m_cTmpSumChiral.x + thisSumChiral.x / uiVolume;
    m_cTmpSumChiral.y = m_cTmpSumChiral.y + thisSumChiral.y / uiVolume;

    _kernelDotAndGatherXYPion << <block, threads >> > (
        pF1W->m_pDeviceData,
        pF2W->m_pDeviceData,
        m_pDeviceXYBufferPion,
        _D_ComplexThreadBuffer);

    const CLGComplex thisSumPion = appGetCudaHelper()->ThreadBufferSum(_D_ComplexThreadBuffer);

    m_cTmpSumPion.x = m_cTmpSumPion.x + thisSumPion.x / uiVolume;
    m_cTmpSumPion.y = m_cTmpSumPion.y + thisSumPion.y / uiVolume;

    _kernelDotAndGatherXYRhon << <block, threads >> > (
        pF1W->m_pDeviceData,
        pF2W->m_pDeviceData,
        m_pDeviceXYBufferRhon,
        _D_ComplexThreadBuffer);

    const CLGComplex thisSumRhon = appGetCudaHelper()->ThreadBufferSum(_D_ComplexThreadBuffer);

    m_cTmpSumRhon.x = m_cTmpSumRhon.x + thisSumRhon.x / uiVolume;
    m_cTmpSumRhon.y = m_cTmpSumRhon.y + thisSumRhon.y / uiVolume;

#pragma endregion

    if (bEnd)
    {
        if (m_bMeasureDistribution)
        {
            dim3 block2(_HC_DecompX, 1, 1);
            dim3 threads2(_HC_DecompLx, 1, 1);
            dim3 block3(m_uiMaxR + 1, 1, 1);
            dim3 threads3(m_uiMaxR + 1, 1, 1);

            _kernelChiralCondensateInitialDist << <block3, threads3 >> >(m_pDistributionR, 
                m_pDistributionChiral,
                m_pDistributionPion,
                m_pDistributionRhon);

            _kernelChiralCondensateMeasureDist << <block2, threads2 >> >(
                m_pDeviceXYBufferChiral,
                m_pDeviceXYBufferPion,
                m_pDeviceXYBufferRhon,
                CCommonData::m_sCenter,
                m_uiMaxR,
                m_byFieldId,
                m_pDistributionR,
                m_pDistributionChiral,
                m_pDistributionPion,
                m_pDistributionRhon
                );

            _kernelChiralAverageDist << <block3, threads3 >> >(m_pDistributionR, 
                m_pDistributionChiral,
                m_pDistributionPion,
                m_pDistributionRhon);

            //extract res
            checkCudaErrors(hipMemcpy(m_pHostDistributionR, m_pDistributionR, sizeof(UINT) * (m_uiMaxR + 1), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(m_pHostDistributionChiral, m_pDistributionChiral, sizeof(Real) * (m_uiMaxR + 1), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(m_pHostDistributionPion, m_pDistributionPion, sizeof(Real) * (m_uiMaxR + 1), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(m_pHostDistributionRhon, m_pDistributionRhon, sizeof(Real) * (m_uiMaxR + 1), hipMemcpyDeviceToHost));

            if (0 == m_uiConfigurationCount)
            {
                assert(0 == m_lstR.Num());
                assert(0 == m_lstChiral.Num());
                assert(0 == m_lstPion.Num());
                assert(0 == m_lstRhon.Num());

                for (UINT uiL = 0; uiL <= m_uiMaxR; ++uiL)
                {
                    if (m_pHostDistributionR[uiL] > 0)
                    {
                        m_lstR.AddItem(uiL);
                        m_lstChiral.AddItem(m_pHostDistributionChiral[uiL] / (m_uiFieldCount * _HC_Lz * _HC_Lt));
                        m_lstPion.AddItem(m_pHostDistributionPion[uiL] / (m_uiFieldCount * _HC_Lz * _HC_Lt));
                        m_lstRhon.AddItem(m_pHostDistributionRhon[uiL] / (m_uiFieldCount * _HC_Lz * _HC_Lt));

                        if (m_bShowResult)
                        {
                            appDetailed(_T("C(%f)=%f, %f, %f\n"),
                                _hostsqrt(static_cast<Real>(uiL)),
                                m_pHostDistributionChiral[uiL],
                                m_pHostDistributionPion[uiL],
                                m_pHostDistributionRhon[uiL]
                            );
                        }
                    }
                }
            }
            else
            {
                for (INT i = 0; i < m_lstR.Num(); ++i)
                {
                    assert(m_pHostDistributionR[m_lstR[i]] > 0);
                    m_lstChiral.AddItem(m_pHostDistributionChiral[m_lstR[i]] / (m_uiFieldCount * _HC_Lz * _HC_Lt));
                    m_lstPion.AddItem(m_pHostDistributionPion[m_lstR[i]] / (m_uiFieldCount * _HC_Lz * _HC_Lt));
                    m_lstRhon.AddItem(m_pHostDistributionRhon[m_lstR[i]] / (m_uiFieldCount * _HC_Lz * _HC_Lt));

                    if (m_bShowResult)
                    {
                        appDetailed(_T("C(%f)=%f, %f, %f\n"),
                            _hostsqrt(static_cast<Real>(m_lstR[i])),
                            m_pHostDistributionChiral[m_lstR[i]],
                            m_pHostDistributionPion[m_lstR[i]],
                            m_pHostDistributionRhon[m_lstR[i]]
                        );
                    }
                }
            }
        }

        //we in fact don't care about XY distribution now...
        checkCudaErrors(hipMemcpy(m_pHostXYBuffer, m_pDeviceXYBufferChiral, sizeof(CLGComplex) * _HC_Lx * _HC_Ly, hipMemcpyDeviceToHost));
        if (m_bShowResult)
        {
            appDetailed(_T("\n ------ Densisty -----\n"));
        }

        for (UINT i = static_cast<UINT>(CCommonData::m_sCenter.x); i < _HC_Lx; ++i)
        {
            CLGComplex cvalue = m_pHostXYBuffer[i * _HC_Ly + CCommonData::m_sCenter.y];
            cvalue.x = cvalue.x / (m_uiFieldCount * _HC_Lz * _HC_Lt);
            cvalue.y = cvalue.y / (m_uiFieldCount * _HC_Lz * _HC_Lt);
            m_lstCondensateDensity.AddItem(cvalue);
            if (m_bShowResult)
            {
                appDetailed(_T("(%d,%d)=%1.6f %s %1.6f I   "), i, CCommonData::m_sCenter.y,
                    cvalue.x,
                    cvalue.y < F(0.0) ? _T("") : _T("+"),
                    appAbs(cvalue.y));
            }
        }
        if (m_bShowResult)
        {
            appDetailed(_T("\n ------ Densisty -----\n"));
        }

        m_cTmpSumChiral.x = m_cTmpSumChiral.x / m_uiFieldCount;
        m_cTmpSumChiral.y = m_cTmpSumChiral.y / m_uiFieldCount;
        m_cTmpSumPion.x = m_cTmpSumPion.x / m_uiFieldCount;
        m_cTmpSumPion.y = m_cTmpSumPion.y / m_uiFieldCount;
        m_cTmpSumRhon.x = m_cTmpSumRhon.x / m_uiFieldCount;
        m_cTmpSumRhon.y = m_cTmpSumRhon.y / m_uiFieldCount;
        appDetailed(_T("\nChiral Condensate = %2.12f + %2.12f\n"), m_cTmpSumChiral.x, m_cTmpSumChiral.y);
        appDetailed(_T("\nPion Condensate = %2.12f + %2.12f\n"), m_cTmpSumPion.x, m_cTmpSumPion.y);
        appDetailed(_T("\nRhon Condensate = %2.12f + %2.12f\n"), m_cTmpSumRhon.x, m_cTmpSumRhon.y);
        ++m_uiConfigurationCount;
        m_lstChiralAll.AddItem(m_cTmpSumChiral);
        m_lstPionAll.AddItem(m_cTmpSumPion);
        m_lstRhonAll.AddItem(m_cTmpSumRhon);
    }
}

void CMeasureChiralCondensate::OnConfigurationAccepted(const CFieldGauge* pGauge, const CFieldGauge* pCorrespondingStaple)
{

}

void CMeasureChiralCondensate::Average(UINT )
{
    //nothing to do
}

void CMeasureChiralCondensate::Report()
{
    assert(m_uiConfigurationCount == static_cast<UINT>(m_lstChiralAll.Num()));
    assert(m_uiConfigurationCount == static_cast<UINT>(m_lstPionAll.Num()));
    assert(m_uiConfigurationCount == static_cast<UINT>(m_lstRhonAll.Num()));

    assert(static_cast<UINT>(m_uiConfigurationCount * CCommonData::m_sCenter.x)
        == static_cast<UINT>(m_lstCondensateDensity.Num()));

    appSetLogDate(FALSE);
    CLGComplex tmpChargeSum = _make_cuComplex(F(0.0), F(0.0));
    m_lstAverageCondensateDensity.RemoveAll();

    appGeneral(_T("\n==========================================================================\n"));
    appGeneral(_T("==================== Chiral Condensate (%d con)============================\n"), m_uiConfigurationCount);

    if (m_uiConfigurationCount > 1)
    {
        appGeneral(_T("\n ----------- each configuration ------------- \n"));
        appGeneral(_T("{"));

        for (UINT i = 0; i < m_uiConfigurationCount; ++i)
        {
            tmpChargeSum.x += m_lstChiralAll[i].x;
            tmpChargeSum.y += m_lstChiralAll[i].y;
            LogGeneralComplex(m_lstChiralAll[i]);
        }
        appGeneral(_T("}\n"));

        tmpChargeSum.x = tmpChargeSum.x / m_uiConfigurationCount;
        tmpChargeSum.y = tmpChargeSum.y / m_uiConfigurationCount;
        appGeneral(_T("\n ----------- average condensate = %2.12f + %2.12f ------------- \n"),
            tmpChargeSum.x, tmpChargeSum.y);

        m_cAverageCondensate = tmpChargeSum;
    }
    else
    {
        appGeneral(_T("\n ----------- average condensate = %2.12f + %2.12f ------------- \n"),
            m_lstChiralAll[0].x,
            m_lstChiralAll[0].y);

        m_cAverageCondensate = m_lstChiralAll[0];
    }

    appGeneral(_T("\n ----------- condensate density------------- \n"));
    appGeneral(_T("{\n"));
    for (UINT k = 0; k < m_uiConfigurationCount; ++k)
    {
        appGeneral(_T("{"));
        for (UINT i = 0; i < static_cast<UINT>(CCommonData::m_sCenter.x); ++i)
        {
            LogGeneralComplex(m_lstCondensateDensity[k * CCommonData::m_sCenter.x + i]);

            if (0 == k)
            {
                m_lstAverageCondensateDensity.AddItem(m_lstCondensateDensity[k * CCommonData::m_sCenter.x + i]);
            }
            else
            {
                m_lstAverageCondensateDensity[i] = _cuCaddf(m_lstAverageCondensateDensity[i], m_lstCondensateDensity[k * CCommonData::m_sCenter.x + i]);
            }

            if (k == m_uiConfigurationCount - 1)
            {
                m_lstAverageCondensateDensity[i].x = m_lstAverageCondensateDensity[i].x / m_uiConfigurationCount;
                m_lstAverageCondensateDensity[i].y = m_lstAverageCondensateDensity[i].y / m_uiConfigurationCount;
            }
        }
        appGeneral(_T("},\n"));
    }
    appGeneral(_T("}\n"));

    appGeneral(_T("==========================================================================\n"));
    appSetLogDate(TRUE);
}

void CMeasureChiralCondensate::Reset()
{
    m_uiConfigurationCount = 0;
    m_lstChiralAll.RemoveAll();
    m_lstPionAll.RemoveAll();
    m_lstRhonAll.RemoveAll();

    m_lstCondensateDensity.RemoveAll();

    m_lstR.RemoveAll();
    m_lstChiral.RemoveAll();
    m_lstPion.RemoveAll();
    m_lstRhon.RemoveAll();
}

__END_NAMESPACE

//=============================================================================
// END OF FILE
//=============================================================================