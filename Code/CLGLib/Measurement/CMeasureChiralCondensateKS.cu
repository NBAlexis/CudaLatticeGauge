#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : CMeasureChiralCondensateKS.cpp
// 
// DESCRIPTION:
// almost copy from CMeasureChiralCondensate.cpp, but with Wilson SU3 vector to SU3 vector
//
// REVISION:
//  [10/01/2020 nbale]
//=============================================================================

#include "CLGLib_Private.h"

__BEGIN_NAMESPACE

__CLGIMPLEMENT_CLASS(CMeasureChiralCondensateKS)

#pragma region kernels

/**
 * 
 */
__global__ void _CLG_LAUNCH_BOUND
_kernelDotMeasureAllKS(
    BYTE byMeasureIndex,
    Real fOmega,
    SSmallInt4 sCenter,
    const deviceSU3Vector* __restrict__ pMe,
    const deviceSU3Vector* __restrict__ pOther,
    CLGComplex* resultXYPlan,
    CLGComplex* result)
{
    intokernalInt4;

    const deviceSU3Vector& right = pOther[uiSiteIndex];
    //switch (byMeasureIndex)
    //{
    //case 1:
    //case 2:
    //case 3:
    //case 4:
    //case 5:
    //    {
    //        right = __chiralGamma[byMeasureIndex].MulWilsonC(right);
    //    }
    //    break;
    //case 6:
    //    {
    //        right = __chiralGamma[GAMMA45].MulWilsonC(right);
    //    }
    //    break;
    //case 7:
    //    {
    //        const Real fYOmega = static_cast<Real>(sSite4.y - sCenter.y)* fOmega;
    //        deviceWilsonVectorSU3 toAdd(__chiralGamma[GAMMA4].MulWilsonC(right));
    //        toAdd.MulReal(fYOmega);
    //        right = __chiralGamma[GAMMA1].MulWilsonC(right);
    //        right.Add(toAdd);
    //    }
    //    break;
    //case 8:
    //    {
    //        const Real fXOmega = static_cast<Real>(sSite4.x - sCenter.x)* fOmega;
    //        deviceWilsonVectorSU3 toAdd = __chiralGamma[GAMMA4].MulWilsonC(right);
    //        toAdd.MulReal(fXOmega);
    //        right = __chiralGamma[GAMMA2].MulWilsonC(right);
    //        right.Sub(toAdd);
    //    }
    //    break;
    //case 0:
    //default:
    //    break;
    //}

    result[uiSiteIndex] = pMe[uiSiteIndex].ConjugateDotC(right);
    atomicAdd(&resultXYPlan[_ixy].x, result[uiSiteIndex].x);
    atomicAdd(&resultXYPlan[_ixy].y, result[uiSiteIndex].y);
}

__global__ void _CLG_LAUNCH_BOUND
_kernelChiralCondensateInitialDistRKS(UINT* pCount)
{
    pCount[threadIdx.x] = 0;
}

__global__ void _CLG_LAUNCH_BOUND
_kernelChiralCondensateInitialDistCondKS(CLGComplex* pCond)
{
    pCond[threadIdx.x] = _zeroc;
}

__global__ void _CLG_LAUNCH_BOUND
_kernelChiralCondensateMeasureDistKS(
    const CLGComplex* __restrict__ CondXY,
    SSmallInt4 sCenter, UINT uiMax, BYTE byFieldId, UBOOL bCalcR,
    UINT* counter, 
    CLGComplex* CondR
)
{
    UINT uiXY = (threadIdx.x + blockIdx.x * blockDim.x);
    INT uiX = static_cast<INT>(uiXY / _DC_Ly);
    INT uiY = static_cast<INT>(uiXY % _DC_Ly);
    UINT uiC = (sCenter.x - uiX) * (sCenter.x - uiX)
        + (sCenter.y - uiY) * (sCenter.y - uiY);

    SSmallInt4 sSite4;
    sSite4.z = sCenter.z;
    sSite4.w = sCenter.w;
    sSite4.x = static_cast<SBYTE>(uiX);
    sSite4.y = static_cast<SBYTE>(uiY);
    if (uiC <= uiMax && !__idx->_deviceGetMappingIndex(sSite4, byFieldId).IsDirichlet())
    {
        if (bCalcR)
        {
            atomicAdd(&counter[uiC], 1);
        }
        
        atomicAdd(&CondR[uiC].x, CondXY[uiXY].x);
        atomicAdd(&CondR[uiC].y, CondXY[uiXY].y);
    }
}

__global__ void _CLG_LAUNCH_BOUND
_kernelChiralAverageDistKS(UINT* pCount, CLGComplex* pCond)
{
    const UINT uiIdx = threadIdx.x;
    if (pCount[uiIdx] > 0)
    {
        pCond[uiIdx].x = pCond[uiIdx].x / static_cast<Real>(pCount[uiIdx]);
        pCond[uiIdx].y = pCond[uiIdx].y / static_cast<Real>(pCount[uiIdx]);
    }
}

#pragma endregion

CMeasureChiralCondensateKS::~CMeasureChiralCondensateKS()
{
    if (NULL != m_pDeviceXYBuffer[0])
    {
        for (UINT i = 0; i < _kCondMeasureCountKS; ++i)
        {
            checkCudaErrors(hipFree(m_pDeviceXYBuffer[i]));
        }
    }

    if (NULL != m_pHostXYBuffer)
    {
        free(m_pHostXYBuffer);
    }

    if (NULL != m_pDistributionR)
    {
        checkCudaErrors(hipFree(m_pDistributionR));
    }

    if (NULL != m_pDistribution)
    {
        checkCudaErrors(hipFree(m_pDistribution));
    }

    if (NULL != m_pHostDistributionR)
    {
        free(m_pHostDistributionR);
    }

    if (NULL != m_pHostDistribution)
    {
        free(m_pHostDistribution);
    }
}

void CMeasureChiralCondensateKS::Initial(CMeasurementManager* pOwner, CLatticeData* pLatticeData, const CParameters& param, BYTE byId)
{
    CMeasureStochastic::Initial(pOwner, pLatticeData, param, byId);

    for (UINT i = 0; i < _kCondMeasureCountKS; ++i)
    {
        checkCudaErrors(hipMalloc((void**)&m_pDeviceXYBuffer[i], sizeof(CLGComplex) * _HC_Lx * _HC_Ly));
    }    
    m_pHostXYBuffer = (CLGComplex*)malloc(sizeof(CLGComplex) * _HC_Lx * _HC_Ly);

    Reset();

    INT iValue = 1;
    param.FetchValueINT(_T("ShowResult"), iValue);
    m_bShowResult = iValue != 0;

    iValue = 1;
    param.FetchValueINT(_T("MeasureDist"), iValue);
    m_bMeasureDistribution = iValue != 0;

    if (m_bMeasureDistribution)
    {
        //assuming the center is really at center
        m_uiMaxR = ((_HC_Lx + 1) / 2 ) * ((_HC_Lx + 1) / 2 )
            + ((_HC_Ly + 1) / 2 ) * ((_HC_Ly + 1) / 2 );

        checkCudaErrors(hipMalloc((void**)&m_pDistributionR, sizeof(UINT) * (m_uiMaxR + 1)));
        checkCudaErrors(hipMalloc((void**)&m_pDistribution, sizeof(CLGComplex) * (m_uiMaxR + 1)));

        m_pHostDistributionR = (UINT*)malloc(sizeof(UINT) * (m_uiMaxR + 1));
        m_pHostDistribution = (CLGComplex*)malloc(sizeof(CLGComplex) * (m_uiMaxR + 1));
    }
}

void CMeasureChiralCondensateKS::OnConfigurationAcceptedZ4(
    const class CFieldGauge* pAcceptGauge, 
    const class CFieldGauge* pCorrespondingStaple, 
    const class CFieldFermion* pZ4, 
    const class CFieldFermion* pInverseZ4, 
    UBOOL bStart, 
    UBOOL bEnd)
{
    if (bStart)
    {
        for (UINT i = 0; i < _kCondMeasureCountKS; ++i)
        {
            _ZeroXYPlaneC(m_pDeviceXYBuffer[i]);
            m_cTmpSum[i] = _zeroc;
        }
    }

    const Real oneOuiVolume = F(1.0) / appGetLattice()->m_pIndexCache->m_uiSiteNumber[m_byFieldId];
    const CFieldFermionKSSU3 * pF1W = dynamic_cast<const CFieldFermionKSSU3*>(pZ4);
    const CFieldFermionKSSU3* pF2W = dynamic_cast<const CFieldFermionKSSU3*>(pInverseZ4);
    
#pragma region Dot

    // The results are Atomic Add to m_pDeviceXYBuffer
    preparethread;
    for (BYTE i = 0; i < _kCondMeasureCountKS; ++i)
    {
        _kernelDotMeasureAllKS << <block, threads >> > (
            i,
            CCommonData::m_fOmega,
            CCommonData::m_sCenter,
            pF1W->m_pDeviceData,
            pF2W->m_pDeviceData,
            m_pDeviceXYBuffer[i],
            _D_ComplexThreadBuffer
            );
        const CLGComplex thisSum = appGetCudaHelper()->ThreadBufferSum(_D_ComplexThreadBuffer);
        m_cTmpSum[i] = _cuCaddf(m_cTmpSum[i], cuCmulf_cr(thisSum, oneOuiVolume));
    }

#pragma endregion

    if (bEnd)
    {
        if (m_bMeasureDistribution)
        {
            dim3 block2(_HC_DecompX, 1, 1);
            dim3 threads2(_HC_DecompLx, 1, 1);
            dim3 block3(1, 1, 1);
            dim3 threads3(m_uiMaxR + 1, 1, 1);

            const Real fDivider = F(1.0) / (m_uiFieldCount * _HC_Lz * _HC_Lt);
            _kernelChiralCondensateInitialDistRKS << <block3, threads3 >> > (m_pDistributionR);
            for (UINT i = 0; i < _kCondMeasureCountKS; ++i)
            {
                _kernelChiralCondensateInitialDistCondKS << <block3, threads3 >> > (m_pDistribution);

                _kernelChiralCondensateMeasureDistKS << <block2, threads2 >> > (
                    m_pDeviceXYBuffer[i],
                    CCommonData::m_sCenter,
                    m_uiMaxR,
                    m_byFieldId,
                    0 == i,
                    m_pDistributionR,
                    m_pDistribution
                    );

                _kernelChiralAverageDistKS << <block3, threads3 >> > (m_pDistributionR, m_pDistribution);

                if (0 == i)
                {
                    checkCudaErrors(hipMemcpy(m_pHostDistributionR, m_pDistributionR, sizeof(UINT) * (m_uiMaxR + 1), hipMemcpyDeviceToHost));
                }

                checkCudaErrors(hipMemcpy(m_pHostDistribution, m_pDistribution, sizeof(CLGComplex) * (m_uiMaxR + 1), hipMemcpyDeviceToHost));

                if (0 == m_uiConfigurationCount)
                {
                    if (0 == i)
                    {
                        assert(0 == m_lstR.Num());
                    }
                    assert(0 == m_lstCond[i].Num());
                    for (UINT uiL = 0; uiL <= m_uiMaxR; ++uiL)
                    {
                        if (m_pHostDistributionR[uiL] > 0)
                        {
                            if (0 == i)
                            {
                                m_lstR.AddItem(uiL);
                            }
                            m_lstCond[i].AddItem(cuCmulf_cr(m_pHostDistribution[uiL], fDivider));

                            if (m_bShowResult)
                            {
                                appDetailed(_T("Cond %d (r = %f)= %f + %f i\n"),
                                    i,
                                    _hostsqrt(static_cast<Real>(uiL)),
                                    m_pHostDistribution[uiL].x,
                                    m_pHostDistribution[uiL].y
                                );
                            }
                        }
                    }
                }
                else
                {
                    for (INT j = 0; j < m_lstR.Num(); ++j)
                    {
                        assert(m_pHostDistributionR[m_lstR[j]] > 0);
                        m_lstCond[i].AddItem(cuCmulf_cr(m_pHostDistribution[m_lstR[j]], fDivider));

                        if (m_bShowResult)
                        {
                            appDetailed(_T("Cond %d (r = %f)=%f + %f i\n"),
                                i,
                                _hostsqrt(static_cast<Real>(m_lstR[j])),
                                m_pHostDistribution[m_lstR[j]].x,
                                m_pHostDistribution[m_lstR[j]].y
                            );
                        }
                    }
                }
            }
        }

        const Real fDiv2 = F(1.0) / m_uiFieldCount;
        for (UINT i = 0; i < _kCondMeasureCountKS; ++i)
        {
            m_cTmpSum[i] = cuCmulf_cr(m_cTmpSum[i], fDiv2);
            appDetailed(_T("\n Condensate %d = %2.12f + %2.12f\n"), i, m_cTmpSum[i].x, m_cTmpSum[i].y);
            m_lstCondAll[i].AddItem(m_cTmpSum[i]);
        }

        ++m_uiConfigurationCount;
    }
}

void CMeasureChiralCondensateKS::OnConfigurationAccepted(const CFieldGauge* pGauge, const CFieldGauge* pCorrespondingStaple)
{

}

void CMeasureChiralCondensateKS::Average(UINT )
{
    //nothing to do
}

void CMeasureChiralCondensateKS::Report()
{
    for (UINT i = 0; i < _kCondMeasureCountKS; ++i)
    {
        assert(m_uiConfigurationCount == static_cast<UINT>(m_lstCondAll[i].Num()));

        appGeneral(_T("\n==========================================================================\n"));
        appGeneral(_T("==================== Condensate No %d (%d con)============================\n"), i, m_uiConfigurationCount);
        CLGComplex tmpChargeSum = _zeroc;
        if (m_uiConfigurationCount > 1)
        {
            appGeneral(_T("\n ----------- each configuration ------------- \n"));
            appGeneral(_T("{"));

            for (UINT j = 0; j < m_uiConfigurationCount; ++j)
            {
                tmpChargeSum.x += m_lstCondAll[i][j].x;
                tmpChargeSum.y += m_lstCondAll[i][j].y;
                LogGeneralComplex(m_lstCondAll[i][j]);
            }
            appGeneral(_T("}\n"));

            tmpChargeSum.x = tmpChargeSum.x / m_uiConfigurationCount;
            tmpChargeSum.y = tmpChargeSum.y / m_uiConfigurationCount;
            appGeneral(_T("\n ----------- average condensate = %2.12f + %2.12f ------------- \n"),
                tmpChargeSum.x, tmpChargeSum.y);

            //m_cAverageCondensate = tmpChargeSum;
        }
        else
        {
            appGeneral(_T("\n ----------- average condensate = %2.12f + %2.12f ------------- \n"),
                m_lstCondAll[i][0].x,
                m_lstCondAll[i][0].y);

            //m_cAverageCondensate = m_lstCondAll[i][0];
        }
    }

    appGeneral(_T("==========================================================================\n"));
    appSetLogDate(TRUE);
}

void CMeasureChiralCondensateKS::Reset()
{
    m_uiConfigurationCount = 0;
    for (UINT i = 0; i < _kCondMeasureCountKS; ++i)
    {
        m_lstCondAll[i].RemoveAll();
        m_lstCond[i].RemoveAll();
    }
    m_lstR.RemoveAll();
}

__END_NAMESPACE

//=============================================================================
// END OF FILE
//=============================================================================